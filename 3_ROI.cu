#include "hip/hip_runtime.h"
/*
 * readLaserPositions.cu
 *
 *  Created on: Apr 18, 2017
 *      Author: gabriel
 */
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include "0_Mainparameters.h"
ifstream ROIfile;
std::string ROIFILE = "lambda_488/Measure/T_0/Z_0/meas_ROIs_";
std::string endROI = ".txt";
int maxROIx = 0.0, minROIx = 1E6, maxROIy = 0.0, minROIy = 1E6;

void readstoreCroppedROI(void) {
	float ROIval;
	bool XY = FALSE;


	hipMallocManaged(&ROIx, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&ROIy, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&d_ROIx, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&d_ROIy, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&ROIxScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&ROIyScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&offsetROI, TA.Nb_LaserPositions * sizeof(int));

	int iROIpos = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		filename = resourcesdirectory + ROIFILE + std::to_string(idistrib + 1) + endROI;
		verbosefile << "filename " << filename.c_str() << " \n";
		ROIfile.open(filename);
		if (!ROIfile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}
		XY = FALSE;
		// introduce here scale and offset relative to camera origin, if needed
		while (ROIfile >> ROIval) {
			if (!XY) {
			*(ROIx + iROIpos) = (ROIval + OFSCAL.offsetROIx)* OFSCAL.scaleROIx;
			TA.maxROIx = max(TA.maxROIx, *(ROIx + iROIpos));
			TA.minROIx = min(TA.minROIx, *(ROIx + iROIpos));
			// Laser positions in x zoomed integer in the 2D scratchpad
			}
			else {
				*(ROIy + iROIpos) = (ROIval + OFSCAL.offsetROIy) * OFSCAL.scaleROIy;
				TA.maxROIy = max(TA.maxROIy, *(ROIy + iROIpos));
				TA.minROIy = min(TA.minROIy, *(ROIy + iROIpos));
				iROIpos++;
			}
			XY = !XY;
		}
		ROIfile.close();
	}

	verbosefile << "ROI \u2463 min and max x " << TA.maxROIx << " " << TA.minROIx << " y "
			<<  TA.maxROIy << " " << TA.minROIy << endl;
}

bool validateCroppedROI_control(void) {

	double Delx, Dely;
	bool testROI = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateCroppedROI_device<<<dimGrid, dimBlock, 0>>>(TA.Nb_LaserPositions);
	hipDeviceSynchronize();

	if (TA.Nb_LaserPositions < SPARSE && VERBOSE)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			if(!ival)
			verbosefile <<" ROI \u2463 ----------------------------------------------------------------------------------------------------\n";
			verbosefile << " ROI \u2463 ROI position " << ival << " ROI position x & y: ";
			verbosefile << ival << "  " << *(ROIx + ival) << "  " << *(ROIy + ival) << endl;
			verbosefile << " ROI \u2463 ROI position in scratchpad " << ival << " ROI position x & y: ";
			verbosefile << ival << "  " << *(ROIxScratch + ival) << "  " << *(ROIyScratch + ival);
			verbosefile << ival << " ROI \u2463 ***************SCRATCHPAD FULL OFFSET ";
			verbosefile << ival << *(offsetROI + ival) << "*************\n";
			verbosefile << " ROI \u2463 --------------------------------------------------------------------------------------------------\n";
		}

	for (int iROI = 0; iROI < TA.Nb_LaserPositions; iROI++) {
		Delx += ROIx[iROI] - d_ROIx[iROI];
		Dely += ROIy[iROI] - d_ROIy[iROI];
	}
	Sumdel[3] = Delx * Dely;
	if (Delx * Dely == 0.0f)
		testROI = TRUE;
	return (testROI);
}

