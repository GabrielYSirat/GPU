
#include <hip/hip_runtime.h>
#ifdef STARTDEVICE
DD.step++;
	if (!itb) timer = clock64();
		if (!iprint)  	// the condition is required to have it printed once
			printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING ** processing  %g from start  %g  total %g \n\n",
					DD.step, (float) (timer - time_start) / CLOCKS_PER_SEC,
					(float) (  time_start - time_init) / CLOCKS_PER_SEC,
					(float) (timer - time_init) / CLOCKS_PER_SEC);
		__syncthreads();
			if (!itb) time_start = clock64();
		
#endif
