#include "hip/hip_runtime.h"
/*
 * tileorganization.cu
 *
 *  Created on: Jul 3, 2017
 *      Author: gabriel
 */
#include "0_Mainparameters.h"

__managed__ int *image_to_scratchpad_offset = { 0 }, *valid_image = { 0 };
__managed__ float *image_to_scratchpad_offset_global = { 0 };
int AmaxLaserx, AmaxLasery, AminLaserx, AminLasery;
const char * MIintilefile = "results/C_microimagesintile.pgm";
const char * NIintilefile = "results/C_microimagesintile2.pgm";
float * reorganized_data;
int fullnumberoftiles,datafullsize;
__managed__ double MaxNewSimus;

bool tileorganization(void) {
	bool Lasertile = TRUE;
	int organization_x[16] = { 0, 1, 2, 3, 2, 2, 3, 3, 4, 3, 3 };
	int organization_y[16] = { 0, 1, 1, 1, 2, 2, 2, 2, 2, 3, 3 };
	int tilex, tiley, tilenumber, ilasertile;

	filename = resourcesdirectory + "rec_image.xml";
	hipMallocManaged(&image_to_scratchpad_offset, MAXNUMBERLASERTILE * MAXTILE * sizeof(int));
	hipMallocManaged(&valid_image, MAXNUMBERLASERTILE * MAXTILE * sizeof(int));

	// min and max Laser positions rounded to integer of camera pixels
	AmaxLaserx = ceil(TA.maxLaserx);
	AmaxLasery = ceil(TA.maxLasery);
	AminLaserx = floor(TA.minLaserx);
	AminLasery = floor(TA.minLasery);

	/************Tiles and aggregates*******************/
	int recdeftile0x = ceil((float) TA.Nb_Cols_reconstruction / XTile);
	int recdeftile0y = ceil((float) TA.Nb_Rows_reconstruction / YTile);
	int laserdeftile0x = CEILING_POS((float )pZOOM * (AmaxLaserx - AminLaserx) / XTile);
	int laserdeftile0y = CEILING_POS((float )pZOOM * (AmaxLasery - AminLasery) / YTile);
	tile.NbTile0x = max(laserdeftile0x, recdeftile0x);
	tile.NbTile0y = max(laserdeftile0y, recdeftile0y);
	if (((AmaxLaserx - AminLaserx) > TA.Nb_Cols_reconstruction)
			|| ((AmaxLasery - AminLasery) > TA.Nb_Rows_reconstruction))
		printf(" INIT PROG \u24FA \u26A0 tiles reconstruction too small!");

	printf(" INIT PROG \u24FA AmaxLaserx %d AmaxLasery %d AminLaserx %d AminLasery %d\n", AmaxLaserx,
			AmaxLasery, AminLaserx, AminLasery);
	printf(" INIT PROG \u24FA recdeftile0x %d recdeftile0y %d laserdeftile0x %d laserdeftile0y %d\n",
			recdeftile0x, recdeftile0y, laserdeftile0x, laserdeftile0y);
	printf(" INIT PROG \u24FA Min Number of tiles x:  %d y: %d\n", tile.NbTile0x, tile.NbTile0y);

	/*************************Aggregates organization depending on MP*******/
	TA.MP_perdistrib = TA.MP / Ndistrib;
	printf(" INIT PROG \u24FA Total number of MP per distribution %d  ", TA.MP_perdistrib);
	printf("  organized as x:%d,  y:%d \n", organization_x[TA.MP_perdistrib],
			organization_y[TA.MP_perdistrib]);

	/************************Aggregates********************************************/
	tile.NbAggregx = ceil((float) tile.NbTile0x / organization_x[TA.MP_perdistrib]);
	tile.NbAggregy = ceil((float) tile.NbTile0y / organization_y[TA.MP_perdistrib]);
	printf(" INIT PROG \u24FA Number of aggregates x:%d y:%d  \n", tile.NbAggregx, tile.NbAggregy);

	/**********************************Tiles****************************************/
	if (tile.NbAggregx == 1) {
		tile.NbTilex = tile.NbTile0x;
		tile.tileperaggregatex = tile.NbTile0x;
	} else {
		tile.NbTilex = tile.NbAggregx * organization_x[TA.MP_perdistrib];
		tile.tileperaggregatex = organization_x[TA.MP_perdistrib];
	}

	if (tile.NbAggregy == 1) {
		tile.NbTiley = tile.NbTile0y;
		tile.tileperaggregatey = tile.NbTile0y;
	} else {
		tile.NbTiley = tile.NbAggregy * organization_y[TA.MP_perdistrib];
		tile.tileperaggregatey = organization_y[TA.MP_perdistrib];
	}

	tile.NbTileXY = tile.NbTilex * tile.NbTiley;
	tile.NbTileXYD = tile.NbTilex * tile.NbTiley * Ndistrib;
	fullnumberoftiles = tile.maxLaserintile * tile.NbTileXYD;
	datafullsize = fullnumberoftiles * NThreads;

	/** FUTURE: In the real application the reconstruction
	 * is created by the program and not read from a file
	 * in this case the size data will be consistent by design
	 */
	// can be improved depending on the ratio between TILE0 and TILE: minor
	TA.Nb_Cols_reconstruction = tile.NbTilex * XTile;
	TA.Nb_Rows_reconstruction = tile.NbTiley * YTile;
	tile.startx = AminLaserx; //floor(pZOOM*((AminLaserx + AmaxLaserx)/2 - (tile.NbTilex * XTile)/2));
	tile.starty = AminLasery; //floor(pZOOM*((AminLasery + AmaxLasery)/2 - (tile.NbTiley * YTile)/2));
	TA.reconstruction_size = TA.Nb_Rows_reconstruction * TA.Nb_Cols_reconstruction;
	printf(" INIT PROG \u24FA Final number of tiles x: %d y: %d distrib %d  \n", tile.NbTilex, tile.NbTiley,
			Ndistrib);
	verbosefile << " INIT PROG \u24FA Reconstruction size x: " << TA.Nb_Cols_reconstruction << " y: " << TA.Nb_Rows_reconstruction << endl;
	verbosefile << "INIT PROG \u24FA NbTileXY " << tile.NbTileXY << " NbTileXYD " << tile.NbTileXYD;
	verbosefile << "start x " << tile.startx << " y " << tile.starty <<  " MinLaser " << AminLaserx << "  " << AminLasery;
	verbosefile << "in REC pixels x: " << tile.startx * pZOOM << " y "<< tile.starty * pZOOM << endl;

	tile.NbLaserTotal = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		for (int iLaser = tile.NbLaserTotal;
				iLaser < tile.NbLaserTotal + tile.Nblaserperdistribution[idistrib]; iLaser++) {
			// position in tiles, tilex and tiley and overall tile number (including distrib)
			tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
			tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
			tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
			float deltilex = *(PosLaserx + iLaser) * pZOOM - tilex * XTile;
			float deltiley = *(PosLasery + iLaser) * pZOOM - tiley * YTile;
			float delscratchx = deltilex + (XSCRATCH - XTile) / 2;  // XSCRATCH and XTile are odd
			float delscratchy = deltiley + (YSCRATCH - YTile) / 2;  // ySCRATCH and YTile are odd
			tile.posintile[iLaser] = tile.NbLaserpertile[tilenumber]++;
			if (VERBOSE)
				printf("TILE ORG \u2479 POS IN TILE: iLaser %d,tilenumber %d tile.posintile[iLaser]  %d\n",
						iLaser, tilenumber, tile.posintile[iLaser]);
			ilasertile = tilenumber * MAXNUMBERLASERTILE + tile.posintile[iLaser];
			valid_image[ilasertile] = 1;
			image_to_scratchpad_offset[ilasertile] = *(offsetFULL + iLaser);
			tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[tilenumber]); // acquiring the max value per tile

			// where will be this microimage in the corresponding tile if posintile is 31, and tilenumber is 8?
			//  this microimage is the microimage with index 31 (the indexes begin at 0) of tile of index 8
			// add 1 - to go to 32 - to NbLaserpertile, because we added an image

			if (VERBOSE)
				printf(
						"TILE ORG \u2479 POS IN SCRATCH: numeral %d laser pos in x %f in y: %f  tile x: %d y: %d \n"
								"TILE ORG \u24FA POS IN SCRATCH: deltile x: %f and y %f del scratch x:%f y:%f\n"
								"TILE ORG \u24FA POS IN SCRATCH: ilasertile %d SCRATCH POSITION %d\n"
								"********************ilasertile %d offset scratchpad interaction****************** %d\n",
						iLaser, *(PosLaserx + iLaser), *(PosLasery + iLaser), tilex, tiley, deltilex,
						deltiley, delscratchx, delscratchy, ilasertile,
						image_to_scratchpad_offset[ilasertile], ilasertile,
						image_to_scratchpad_offset[ilasertile]);

			if (VERBOSE)
				printf("TILE ORG \u2479 POS IN SCRATCH: image number %d tilenumber %d position in tile %d\n",
						iLaser, tilenumber, tile.posintile[iLaser]);
		}
		printf("TILE ORG \u24FA  idistrib n°%d number of laser positions in tile in distribution %d\n",
				idistrib, tile.Nblaserperdistribution[idistrib]);
		if (VERBOSE) {
			int it0 = tile.NbTilex * tile.NbTiley * idistrib;
			for (int it = it0; it < it0 + tile.NbTilex * tile.NbTiley; it++)
				printf(" \u2479 tile %d: #lasers %d ...\n", it, tile.NbLaserpertile[it]);
			printf("\n");
		}
		tile.NbLaserTotal += tile.Nblaserperdistribution[idistrib];
	}
	printf("TILE ORG \u24FA  tile.NbLaserTotal %d \n", tile.NbLaserTotal);

	for (int it1 = 0; it1 < tile.NbTileXYD; it1++) {
		if (VERBOSE)
			printf(
					"TILE ORG \u2479 Tile number %d tile in x %d tile in y %d distrib %d number of microimages %d\n",
					it1, it1 % (Ndistrib * tile.NbTiley), (it1 / tile.NbTilex) % Ndistrib,
					it1 / (tile.NbTilex * tile.NbTiley), tile.NbLaserpertile[it1]);

		tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[it1]); // acquiring the max value per tile
		tile.minLaserintile = min(tile.minLaserintile, tile.NbLaserpertile[it1]); // acquiring the min value per tile
	}

	printf("TILE ORG \u24FA Max  %d and Min %d  Laser in tile ...   \n", tile.maxLaserintile,
			tile.minLaserintile);
	tile.maxLaserintile = CEILING_POS(((float)tile.maxLaserintile)/NIMAGESPARALLEL) * NIMAGESPARALLEL;
	tile.blocks = tile.maxLaserintile / NIMAGESPARALLEL;
	// rounded to next multiple of NIMAGESPARALLEL
	printf("TILE ORG \u24FA Max Laser in tile rounded to next multiple of NIMAGESPARALLEL  .. %d\n",
			tile.maxLaserintile);

	return (Lasertile);
}

bool initializesimusData(void) {
// Initialize new simus and Data
	fullnumberoftiles = tile.maxLaserintile * tile.NbTileXYD;
	datafullsize = fullnumberoftiles * NThreads;
	verbosefile << "TILE ORG \u2466 size simus " << datafullsize << " AminLaserx " << AminLaserx << " AminLasery " << AminLasery << endl;
	hipMallocManaged(&new_simus, datafullsize * sizeof(float));
	hipMallocManaged(&Data, datafullsize * sizeof(float));
	hipMallocManaged(&Rfactor, datafullsize * sizeof(float));

	for (int ii = 0; ii < datafullsize; ii++) {
		new_simus[ii] = 0.0f;
		Data[ii] = 0.0f;
		Rfactor[ii] = 0.0f;
	}
	hipMallocManaged(&distribvalidGPU, TA.MP * PSFZOOMSQUARE * sizeof(float));
	for (int itemp = 0; itemp < Ndistrib * PSFZOOMSQUARE; itemp++) *(distribvalidGPU + itemp) = 0.0;
	return (TRUE);
}

bool microimagesintile(void) {
	float ratioMI = 1.0 / (Maxmicroimages - Minmicroimages);
	bool micimintile = FALSE;
	reorganized_data = (float *) calloc(fullnumberoftiles*PixZoomSquare, sizeof(float));

	unsigned char *i_data = (unsigned char *) calloc(PixZoomSquare * tile.NbTileXYD * tile.maxLaserintile, sizeof(unsigned char));
	unsigned char *j_data = (unsigned char *) calloc(PixZoomSquare * tile.NbTileXYD * tile.maxLaserintile, sizeof(unsigned char));
	verbosefile << "TILE ORG \u24FA Max Laser in tile rounded to multiple NIMAGESPARALLEL  .. " << tile.maxLaserintile;
	verbosefile << endl << "TILE ORG \u24FA Max and min microimages " << Maxmicroimages << " " << Minmicroimages << endl;

	float Maxdata = 0.0f;
	for (int idistrib = 0, disdelta = 0; idistrib < Ndistrib; idistrib++, disdelta += tile.Nblaserperdistribution[idistrib])
		for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
			int tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
			int tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
			int tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
			int ilasertile = tilenumber * tile.maxLaserintile + tile.posintile[iLaser];
			verbosefile << "TILE ORG \u247A idistrib " << idistrib << " iLaser " << iLaser << " iLasertile " << ilasertile << " tilenumber "
					<< tilenumber << " tilex " << tilex << " tiley " << tiley << " tileblock " << disdelta << endl;
			for (int ipix = 0; ipix < PixZoomSquare; ipix++) { // copy microimage to its position in the Data
				*(reorganized_data + ilasertile * PixZoomSquare + ipix) = *(zoomed_microimages + iLaser * PixZoomSquare + ipix);
				*(Data + ilasertile * NThreads + ipix) = *(reorganized_data + ilasertile * PixZoomSquare + ipix);
				i_data[ilasertile * PixZoomSquare + ipix] = 255.0 * (*(reorganized_data + ilasertile * PixZoomSquare + ipix) - Minmicroimages) * ratioMI;
				Maxdata = max(Maxdata, i_data[ilasertile * PixZoomSquare + ipix]) ;
			}
		}
	printf("Maxdata %f Nbtile XY %d NbTile XYD %d Laserintile %d\n", Maxdata, tile.NbTileXY, tile.NbTileXYD, tile.maxLaserintile);
	T4Dto2D( j_data, i_data, tile.NbTileXYD,tile.maxLaserintile,PixZoom, PixZoom);
	verbosefile << "HOST: \u277D DEVICE TEST in biginspect.cu: Path to calculated new simulations " << MIintilefile << " .....\n";
	sdkSavePGM(MIintilefile, i_data, PixZoom , tile.maxLaserintile * tile.NbTileXYD * PixZoom);
	sdkSavePGM(NIintilefile, j_data, tile.maxLaserintile * PixZoom, tile.NbTileXYD * PixZoom);

	return (micimintile);
}
