#include "hip/hip_runtime.h"
/*
 * readvalidatePSF_control.cu
 *
 *  Created on: Apr 19, 2017
 *      Author: gabriel
 */

#include "0_NewLoop.h"
string PSFDATA = "/lambda_488/Calib/system_PSF.bin";
double *double_PSF;			// on host
float MaxPSF=0.0f, SumPSF = 0.0f;

void PSFprepare(void) {
	char * memblock;
	int size;
//	XMLDocument doc;
//	MaxPSF = 0.; // also used as extern

	string PSFraw = resourcesdirectory + PSFDATA;
	const char * PSFImagefile = "results/PSFImagefile.pgm";

	unsigned char *i_PSF = (unsigned char *) calloc(TA.PSF_size, sizeof(unsigned char)); // on host
	double* double_PSF = (double*)std::malloc(TA.PSF_Rows*TA.Nb_Cols_PSF * sizeof(double));
	hipMallocManaged(&original_PSF, PSFZoom * PSFZoom * sizeof(float));
	hipMallocManaged(&PSFARRAY, PSFZoom * PSFZoom *  sizeof(float));

	//read pPSF bin file
	std::ifstream PSFile(PSFraw.c_str(), ios::in | ios::binary | ios::ate);
	size = (PSFile.tellg()); 	// the data is stored in doubles of 8 bytes in the file
	size -= byte_skipped;  				// removes byte_skipped
	memblock = new char[size];
	PSFile.seekg(byte_skipped, ios::beg); // 4 first bytes are offset
	PSFile.read(memblock, size);
	PSFile.close();

	double_PSF = (double*) memblock; //reinterpret the chars stored in the file as double
	for (int i = 0; i < TA.PSF_Rows*TA.Nb_Cols_PSF; i++) {
				*(original_PSF + i) = *(double_PSF + i)+0.000001;			// change to float
				SumPSF += *(original_PSF+i);
				if (MaxPSF < *(original_PSF + i))
					MaxPSF = *(original_PSF + i); // sanity check, check max
	}

	verbosefile << " PSF \u24F5  Nb_Rows: " << TA.PSF_Rows << " Nb_Cols " << TA.Nb_Cols_PSF;
	verbosefile << " size " << size << " Max: " << MaxPSF << " Sum " << SumPSF << std::endl;
	std::cout << " PSF \u24F5  Nb_Rows: " << TA.PSF_Rows << " Nb_Cols " << TA.Nb_Cols_PSF;
	std::cout << " size " << size << " Max: " << MaxPSF << " Sum " << SumPSF << std::endl;

	tile.expectedmax = MaxPSF; // to be updated later on

	// write pPSF original image to disk
	/////////////////////////////////
	for (int i = 0; i <= TA.PSF_size; i++)
		i_PSF[i] = 255.0*original_PSF[i]/MaxPSF;			// image value
	verbosefile << " PSF \u24F5 function read: Path to pPSF original" << PSFImagefile << endl;

	sdkSavePGM(PSFImagefile, i_PSF, TA.PSF_Rows, TA.Nb_Cols_PSF);
	free(i_PSF);
 }


bool PSFvalidateonhost(void) {
	bool testPSF;
	double MaxPSF;
	double Sum3PSF = 0, max3PSF =0;
		hipMallocManaged(&PSF_valid, TA.PSF_size * sizeof(float)); // representation of pPSF available in global memory
	unsigned char *i_PSF = (unsigned char *) calloc(TA.PSF_size, sizeof(unsigned char)); // on host
	const char * PSFValidationimage = "results/PSFValidationimage.pgm";

    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1,1, 1);
    // Execute the pPSF kernel
    PSFvalidateondevice<<<dimGrid, dimBlock, 0>>>( TA.PSF_Rows, TA.Nb_Cols_PSF);
    hipDeviceSynchronize();

   for(int row = 0; row < TA.PSF_Rows; row++)
    	for( int col = 0; col < TA.Nb_Cols_PSF; col++){
    		Sum3PSF += *(PSF_valid + row*TA.Nb_Cols_PSF + col);
     		max3PSF = max(*(PSF_valid + row*TA.Nb_Cols_PSF + col), max3PSF);
    		}
	verbosefile << " PSF \u24F5 Sum3PSF " << Sum3PSF << " max3PSF " << max3PSF << endl;

	// write pPSF image validation to disk
	/////////////////////////////////
	MaxPSF = 0.0f;
	for (int i = 0; i <= TA.PSF_size; i++) {
		MaxPSF = max(MaxPSF, PSF_valid[i]); // sanity check, check max
	}

	for (int i = 0; i <= TA.PSF_size; i++)
		i_PSF[i] = 255.0*PSF_valid[i]/MaxPSF;			// Validation image value

	verbosefile << " PSF \u24F5 Path to pPSF validation ..." << PSFValidationimage << endl;

	    	sdkSavePGM(PSFValidationimage, i_PSF, TA.PSF_Rows, TA.Nb_Cols_PSF);

	    	verbosefile << " PSF \u24F5  Comparing files ... \n";
	    	testPSF = compareData(PSF_valid,
	                                 original_PSF,
	                                 TA.Nb_Cols_PSF*TA.PSF_Rows,
	                                 MAX_EPSILON_ERROR/1000,
	                                 0.15f);

	        for (int jPSF = 0; jPSF < TA.PSF_size; jPSF++)
	        	Sumdel[1] += fabsf(*(PSF_valid+jPSF)- *(original_PSF+jPSF));
hipFree(PSF_valid);
return(testPSF);
}
