#include "hip/hip_runtime.h"
/*
 * readLaserPositions.cu
 *
 *  Created on: Apr 18, 2017
 *      Author: gabriel
 */
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include "NewLoop.h"
__managed__ int *posxREC = NULL;
__managed__ int *PosxScratch = NULL, *PosyScratch = NULL;
__managed__ int *offsetFULL = NULL;

void readstoreLaserPositions(void) {
	XMLDocument XMLdoc, ACQXML;
	XMLElement *pRoot, *pParm;
	string sstr;
	int index, numberofimages, offsetimages = 0;
	float maxLaserx = 0.0, minLaserx = 1E6, maxLasery = 0.0, minLasery = 1E6;

	filename = resourcesdirectory + "reconst_preprocess_results.xml";
	printf(" Laser \u2462  Laser positions:  %s \n", filename.c_str());
	int LoadOK = XMLError(XMLdoc.LoadFile(filename.c_str()));
	TA.Nb_LaserPositions = 0;

	if (!LoadOK) {
		if (verbose)
			std::cout << "reconst_preprocess_results.xml" << std::endl;
		pRoot = XMLdoc.FirstChildElement("Reconstruction_Preproc_Results");
		if (pRoot) {
			// Parse parameters
			pParm = pRoot->FirstChildElement("Laser_Positions")->FirstChildElement("Distrib");
			while (pParm) {
				index = atoi(pParm->Attribute("index"));
				numberofimages = atoi(pParm->Attribute("length"));
				Ndistrib = index + 1;
				TA.Images_perdistrib[index] = numberofimages;
				TA.Nb_LaserPositions += numberofimages;
				printf(" Laser \u2462 Distrib %d numberofimages %d ...  \n", index, numberofimages);
				pParm = pParm->NextSiblingElement("Distrib");
			}
		}
	}
	printf(" Laser \u2462 number of distrib %d ... ", Ndistrib);
	printf("TA.Nb_LaserPositions %d \n", TA.Nb_LaserPositions);

	hipMallocManaged(&PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLasery, TA.Nb_LaserPositions * sizeof(float));

	/** Allocation of memory of intermediate values - for test and validation
	 *
	 */
	hipMallocManaged(&PosxScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&PosyScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&offsetFULL, TA.Nb_LaserPositions * sizeof(int));

	if (!LoadOK) {
		if (pRoot) {
			// Parse parameters
			pParm = pRoot->FirstChildElement("Laser_Positions")->FirstChildElement("Distrib");
			while (pParm) {
				index = atoi(pParm->Attribute("index"));
				numberofimages = atoi(pParm->Attribute("length"));
				tile.Nblaserperdistribution[index] = numberofimages;
				sstr = pParm->GetText();
				for (unsigned int i = 0; i < strlen(chars); ++i)
					sstr.erase(std::remove(sstr.begin(), sstr.end(), chars[i]), sstr.end());
				stringstream stream(sstr);
				if (verbose)
					std::cout << stream.str() << std::endl;
				for (int i = offsetimages; i < numberofimages + offsetimages; i++) {
					stream.getline(buff, 10, ',');
					// introduce here scale and offset relative to camera origin, if needed
					*(PosLaserx + i) = (atof(buff) + OFSCAL.offsetLaserx) * OFSCAL.scaleLaserx;
					maxLaserx = max(maxLaserx, *(PosLaserx + i));
					minLaserx = min(minLaserx, *(PosLaserx + i));
					if (verbose && (!(i % 2048)))
						printf(" Laser \u2462 i= %d buffer x %s %f\n", i, buff, *(PosLaserx + i));
					// Laser positions in x rounded in rec pixels
					int posxREC = std::round(pZOOM * *(PosLaserx + i));
					// Laser positions in x zoomed integer in the 2D scratchpad
					*(PosxScratch + i) = 33; //(posxREC % XTile) + dxSCR/2;

					stream.getline(buff, 10, ',');
					// introduce here scale and offset relative to camera origin, if needed
					*(PosLasery + i) = (atof(buff) + OFSCAL.offsetLasery) * OFSCAL.scaleLasery;
					maxLasery = max(maxLasery, *(PosLasery + i));
					minLasery = min(minLasery, *(PosLasery + i));
					if (verbose && (!(i % 2048)))
						printf(" Laser \u2462 i= %d buffer x %s %f\n", i, buff, *(PosLasery + i));

					// Laser positions in y zoomed in integer
					int posyREC = std::round(pZOOM * *(PosLasery + i));
					// Laser positions in y zoomed integer in the scratchpad
					*(PosyScratch + i) = 47; //(posyREC % YTile) + dySCR/2;

					*(offsetFULL + i) = *(PosyScratch + i) * XSCRATCH + *(PosxScratch + i) + lostpixels;
				}
				pParm = pParm->NextSiblingElement("Distrib");
				offsetimages += numberofimages;
			}
		}
	}
	printf(" Laser \u2462 min and max x %g %g, min and max y %g %g ... ", maxLaserx, minLaserx, maxLasery, minLasery);
	TA.maxLaserx = maxLaserx;
	TA.maxLasery = maxLasery;
	TA.minLaserx = minLaserx;
	TA.minLasery = minLasery;
	printf("number of distrib %d\n", Ndistrib);

	if (TA.Nb_LaserPositions < smallnumber)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			printf(" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
			printf(" Laser \u2462 Laser position %d  Laser position x: %f  y: %f\n", ival, *(PosLaserx + ival), *(PosLasery + ival));
			printf(" Laser \u2462 Position in scratchpad x: %d y: %d \n", *(PosxScratch + ival), *(PosyScratch + ival));
			printf(" Laser \u2462 ***************SCRATCHPAD FULL OFFSET %d **************\n",*(offsetFULL+ival));
			printf(" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
		}
}

bool validateLaserPositions_control(void) {

	double Delx { 0.0 }, Dely { 0.0 };
	bool testLaserPosition = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateLaserPositions_device<<<dimGrid, dimBlock, 0>>>(TA.Nb_LaserPositions);
	hipDeviceSynchronize();

	for (int iLaser = 0; iLaser < TA.Nb_LaserPositions; iLaser++) {
		Delx += PosLaserx[iLaser] - d_PosLaserx[iLaser];
		Dely += PosLasery[iLaser] - d_PosLasery[iLaser];
	}
	Sumdel[2] = sqrt(Delx * Delx + Dely * Dely);
	printf(" Laser \u2462 delx %g dely %g Sumdel[2] %g \n", Delx, Dely, Sumdel[2]);
	if (Delx * Dely == 0.0f)
		testLaserPosition = TRUE;
	return (testLaserPosition);
}

void readstoreCroppedROI(void) {
	XMLDocument XMLdoc;
	XMLElement *pRoot, *pParm;
	string sstr;
	int numberofimages, offsetimages = 0, missedpoints = 0;
	XMLDocument doc;

	filename = resourcesdirectory + "reconst_preprocess_results.xml";
	printf(" ROI \u2463 ROI positions:  %s \n", filename.c_str());
	int LoadOK = XMLError(XMLdoc.LoadFile(filename.c_str()));

	hipMallocManaged(&ROIx, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&ROIy, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&d_ROIx, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&d_ROIy, TA.Nb_LaserPositions * sizeof(int));

	if (!LoadOK) {
		pRoot = XMLdoc.FirstChildElement("Reconstruction_Preproc_Results");
		if (pRoot) {
			// Parse parameters
			pParm = pRoot->FirstChildElement("Measurement_AOIs")->FirstChildElement("Distrib");
			while (pParm) {
				numberofimages = atoi(pParm->Attribute("length"));
				missedpoints = 0;
				printf(" ROI \u2463 number of images %d \n", numberofimages);
				sstr = pParm->GetText();
				for (unsigned int i = 0; i < strlen(chars); ++i)
					sstr.erase(std::remove(sstr.begin(), sstr.end(), chars[i]), sstr.end());
				stringstream stream(sstr);
				for (int i = offsetimages; i < offsetimages + numberofimages; i++) {
					stream.getline(buff, 10, ',');
					int temp = atoi(buff);
					*(ROIx + i) = temp;
					stream.getline(buff, 10, ',');
					temp = atoi(buff);
					*(ROIy + i) = temp;
					stream.getline(buff, 10, ','); // size of window in x constant
					temp = atoi(buff);
					stream.getline(buff, 10, ','); // size of window in y constant
					temp = atoi(buff);
					if (verbose && (!(i % 512)))
						printf(" ROI \u2463 i= %d ROI [%d,%d]\n", i, *(ROIx + i), *(ROIy + i));
					if (verbose && (*(ROIx + i) == 0) && (*(ROIy + i) == 0))
						missedpoints++;
					if (verbose && (*(ROIx + i) == 0) && (*(ROIy + i) == 0) && (!(i % 16)))
						printf(" ROI \u2463 i= %d, ", i);
				}
				if (missedpoints)
					printf(" ROI \u2463 \n missed points %d", missedpoints);

				pParm = pParm->NextSiblingElement("Distrib");
				offsetimages += numberofimages;
			}
		}
	}
	TA.maxROIx = 0;
	TA.maxROIy = 0;
	TA.minROIx = 512 * 512;
	TA.minROIy = 512 * 512;

	for (int i = 0; i < TA.Nb_LaserPositions; i++) {
		if (*(ROIx + i) > TA.maxROIx)
			TA.maxROIx = *(ROIx + i);
		if (*(ROIy + i) > TA.maxROIy)
			TA.maxROIy = *(ROIy + i);
		if (*(ROIx + i) < TA.minROIx)
			TA.minROIx = *(ROIx + i);
		if (*(ROIy + i) < TA.minROIy)
			TA.minROIy = *(ROIy + i);
	}
	printf(" ROI \u2463 min and max ROI x: min %d max %d y: min %d max %d\n", TA.minROIx, TA.maxROIx, TA.minROIy, TA.maxROIy);
	if (TA.Nb_LaserPositions < smallnumber)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			printf(" ROI \u2463 ---------------------------------------------------------------------\n");
			printf(" ROI \u2463 ROI position %d  ROI x: %d  y: %d\n", ival, *(ROIx + ival), *(ROIy + ival));
			printf(" ROI \u2463 ---------------------------------------------------------------------\n");
		}

}

bool validateCroppedROI_control(void) {

	double Delx, Dely;
	bool testROI = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateCroppedROI_device<<<dimGrid, dimBlock, 0>>>(TA.Nb_LaserPositions);
	hipDeviceSynchronize();
	for (int iLaser = 0; iLaser < TA.Nb_LaserPositions; iLaser++) {
		Delx += ROIx[iLaser] - d_ROIx[iLaser];
		Dely += ROIy[iLaser] - d_ROIy[iLaser];
	}
	Sumdel[3] = Delx * Dely;
	if (Delx * Dely == 0.0f)
		testROI = TRUE;
	return (testROI);
}

