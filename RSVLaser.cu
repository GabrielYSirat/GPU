#include "hip/hip_runtime.h"
/*
 * readLaserPositions.cu
 *
 *  Created on: Apr 18, 2017
 *      Author: gabriel
 */
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include "NewLoop.h"
ifstream inFile;
std::string LaserFILE = "lambda_488/Measure/T_0/Z_0/laser_positions_";
std::string endlaser = ".txt";
std::string ROIFILE = "lambda_488/Measure/T_0/Z_0/meas_ROIs_";
std::string endROI = ".txt";
int maxROIx = 0.0, minROIx = 1E6, maxROIy = 0.0, minROIy = 1E6;
bool XY = FALSE;

void readstoreLaserPositions(void) {
	float laserval;

	TA.Nb_LaserPositions = 0;
	tile.maxlaserperdistribution = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
	{
		filename = resourcesdirectory + LaserFILE + std::to_string(idistrib + 1) + endlaser;
		inFile.open(filename);
		if (!inFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}

		tile.Nblaserperdistribution[idistrib] = 0;
		while (inFile >> laserval) {
			if (XY) {
				TA.Nb_LaserPositions++; tile.Nblaserperdistribution[idistrib]++; }
			XY = !XY;
		} // adding one at the end because it is a number of positions

		tile.maxlaserperdistribution = max(tile.maxlaserperdistribution, tile.Nblaserperdistribution[idistrib]);
		printf(" Laser \u2462: distribution n°%d number of images %d   ", idistrib, tile.Nblaserperdistribution[idistrib]);

		inFile.close();
	}
	printf(" total number of images %d max images per distributions %d \n",  TA.Nb_LaserPositions, tile.maxlaserperdistribution);

	hipMallocManaged(&PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosxScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&PosyScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&offsetFULL, TA.Nb_LaserPositions * sizeof(int));

	int ilaserpos = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		filename = resourcesdirectory + LaserFILE + std::to_string(idistrib + 1) + endlaser;
		printf("Laser \u2462: filename %s \n", filename.c_str());
		inFile.open(filename);
		if (!inFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}
		XY = FALSE;
		// introduce here scale and offset relative to camera origin, if needed
		while (inFile >> laserval) {
			if (!XY) {
			*(PosLaserx + ilaserpos) = (laserval + OFSCAL.offsetLaserx)* OFSCAL.scaleLaserx;
			TA.maxLaserx = max(TA.maxLaserx, *(PosLaserx + ilaserpos));
			TA.minLaserx = min(TA.minLaserx, *(PosLaserx + ilaserpos));
			// Laser positions in x zoomed integer in the 2D scratchpad
			}
			else {
				*(PosLasery + ilaserpos) = (laserval + OFSCAL.offsetLasery) * OFSCAL.scaleLasery;
				TA.maxLasery = max(TA.maxLasery, *(PosLasery + ilaserpos));
				TA.minLasery = min(TA.minLasery, *(PosLasery + ilaserpos));
				ilaserpos++;
			}
			XY = !XY;
		}
		inFile.close();
	}

	printf(" Laser \u2462 min and max x %g %g, min and max y %g %g ... \n",
			TA.maxLaserx, TA.minLaserx, TA.maxLasery, TA.minLasery);
}

bool validateLaserPositions_control(void) {

	double Delx { 0.0 }, Dely { 0.0 };
	bool testLaserPosition = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateLaserPositions_device<<<dimGrid, dimBlock, 0>>> (TA.Nb_LaserPositions);
	hipDeviceSynchronize();

	if (TA.Nb_LaserPositions < smallnumber)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			if(!ival) printf(" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
			printf(" Laser \u2462 Laser position %d  Laser position x: %f  y: %f\n",
					ival, *(PosLaserx + ival), *(PosLasery + ival));
			printf(" Laser \u2462 Position in scratchpad x: %d y: %d \n",
					*(PosxScratch + ival), *(PosyScratch + ival));
			printf(" Laser \u2462 ***************SCRATCHPAD FULL OFFSET %d **************\n",
					*(offsetFULL + ival));
			printf(" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
		}
	printf(" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
	for (int iLaser = 0; iLaser < TA.Nb_LaserPositions; iLaser++) {
		Delx += PosLaserx[iLaser] - d_PosLaserx[iLaser];
		Dely += PosLasery[iLaser] - d_PosLasery[iLaser];
	}
	Sumdel[2] = sqrt(Delx * Delx + Dely * Dely);
	printf(" Laser \u2462 delx %8.6f dely %8.6f Sumdel[2] %8.6f \n", Delx, Dely, Sumdel[2]);
	if (Delx * Dely == 0.0f) testLaserPosition = TRUE;

	return (testLaserPosition);
}

void readstoreCroppedROI(void) {
	float ROIval;

	hipMallocManaged(&ROIx, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&ROIy, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&d_ROIx, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&d_ROIy, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&ROIxScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&ROIyScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&offsetROI, TA.Nb_LaserPositions * sizeof(int));

	int iROIpos = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		filename = resourcesdirectory + ROIFILE + std::to_string(idistrib + 1) + endROI;
		printf("filename %s \n", filename.c_str());
		inFile.open(filename);
		if (!inFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}
		XY = FALSE;
		// introduce here scale and offset relative to camera origin, if needed
		while (inFile >> ROIval) {
			if (!XY) {
			*(ROIx + iROIpos) = (ROIval + OFSCAL.offsetROIx)* OFSCAL.scaleROIx;
			TA.maxROIx = max(TA.maxROIx, *(ROIx + iROIpos));
			TA.minROIx = min(TA.minROIx, *(ROIx + iROIpos));
			// Laser positions in x zoomed integer in the 2D scratchpad
			}
			else {
				*(ROIy + iROIpos) = (ROIval + OFSCAL.offsetROIy) * OFSCAL.scaleROIy;
				TA.maxROIy = max(TA.maxROIy, *(ROIy + iROIpos));
				TA.minROIy = min(TA.minROIy, *(ROIy + iROIpos));
				iROIpos++;
			}
			XY = !XY;
		}
		inFile.close();
	}

	printf(" ROI \u2463 min and max x %d %d, min and max y %d %d ... ",
			TA.maxROIx, TA.minROIx, TA.maxROIy, TA.minROIy);
}

bool validateCroppedROI_control(void) {

	double Delx, Dely;
	bool testROI = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateCroppedROI_device<<<dimGrid, dimBlock, 0>>>(TA.Nb_LaserPositions);
	hipDeviceSynchronize();

	if (TA.Nb_LaserPositions < smallnumber)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			if(!ival) printf(" ROI \u2463 ----------------------------------------------------------------------------------------------------\n");
			printf(" ROI \u2463 ROI position %d  ROI position x: %d  y: %d\n",
					ival, *(ROIx + ival), *(ROIy + ival));
			printf(" ROI \u2463 Position in scratchpad x: %d y: %d \n",
					*(ROIxScratch + ival), *(ROIyScratch + ival));
			printf(" ROI \u2463 ***************SCRATCHPAD FULL OFFSET %d **************\n",
					*(offsetROI + ival));
			printf(" ROI \u2463 ----------------------------------------------------------------------------------------------------\n");
		}

	for (int iROI = 0; iROI < TA.Nb_LaserPositions; iROI++) {
		Delx += ROIx[iROI] - d_ROIx[iROI];
		Dely += ROIy[iROI] - d_ROIy[iROI];
	}
	Sumdel[3] = Delx * Dely;
	if (Delx * Dely == 0.0f)
		testROI = TRUE;
	return (testROI);
}

