#include "hip/hip_runtime.h"
/*
 * readLaserPositions.cu
 *
 *  Created on: Apr 18, 2017
 *      Author: gabriel
 */
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include "NewLoop.h"
std::string LaserFILE = "lambda_488/Measure/T_0/Z_0/laser_positions_";
std::string endlaser = ".txt";
float maxLaserx = 0.0, minLaserx = 1E6, maxLasery = 0.0, minLasery = 1E6;


void readstoreLaserPositions(void) {
	XMLDocument XMLdoc, ACQXML;
	ifstream inFile;
	string sstr;
	float laserval;
	bool XY = FALSE;
	int numberofimages;
	int xREC, yREC;

	TA.Nb_LaserPositions = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		filename = resourcesdirectory + LaserFILE + std::to_string(idistrib + 1) + endlaser;
		printf("Laser \u2462 idistrib %d filename %s \n", idistrib, filename.c_str());
		inFile.open(filename);
		if (!inFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}
		numberofimages = 0;
		while (inFile >> laserval) {
			if (XY) {
				TA.Nb_LaserPositions++; numberofimages++; }
			if (!verbose)
			XY = !XY;
		} // adding one at the end because it is a number of positions
		tile.Nblaserperdistribution[idistrib] = numberofimages;
		printf(" Laser \u2462: distribution n°%d number of images %d\n", idistrib, tile.Nblaserperdistribution[idistrib]);

		inFile.close();
	}
	hipMallocManaged(&PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosxScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&PosyScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&offsetFULL, TA.Nb_LaserPositions * sizeof(int));

	int ilaserpos = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		filename = resourcesdirectory + LaserFILE + std::to_string(idistrib + 1) + endlaser;
		printf("filename %s \n", filename.c_str());
		inFile.open(filename);
		if (!inFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}
		XY = FALSE;
		// introduce here scale and offset relative to camera origin, if needed
		while (inFile >> laserval) {
//			printf("laserval %g, XY %d, il %d\n", laserval, XY, ilaserpos);
			if (!XY) {
			*(PosLaserx + ilaserpos) = (laserval + OFSCAL.offsetLaserx)* OFSCAL.scaleLaserx;
			maxLaserx = max(maxLaserx, *(PosLaserx + ilaserpos));
			minLaserx = min(minLaserx, *(PosLaserx + ilaserpos));
			xREC = std::round(pZOOM * *(PosLaserx + ilaserpos));
			// Laser positions in x zoomed integer in the 2D scratchpad
			*(PosxScratch + ilaserpos) = (xREC % XTile) + dxSCR/2;
// 		printf("xREC %d scratch %d\n", xREC, *(PosxScratch + ilaserpos));
			}
			else {
				*(PosLasery + ilaserpos) = (laserval + OFSCAL.offsetLasery) * OFSCAL.scaleLasery;
				maxLasery = max(maxLasery, *(PosLasery + ilaserpos));
				minLasery = min(minLasery, *(PosLasery + ilaserpos));
				// Laser positions in y zoomed in integer
				yREC = std::round(pZOOM * *(PosLasery + ilaserpos));
				// Laser positions in y zoomed integer in the scratchpad
				*(PosyScratch + ilaserpos) = (yREC % YTile) + dySCR/2;
				*(offsetFULL + ilaserpos) = *(PosyScratch + ilaserpos) * XSCRATCH + *(PosxScratch + ilaserpos) + lostpixels;
//				printf("yREC %d y scratch %d offsetFull %d\n", yREC, *(PosyScratch + ilaserpos), *(offsetFULL + ilaserpos));
//				printf("laserval %g, XY %d, il %d rec x:%d y:%d val %g scratch %d\n",
//						laserval, XY, ilaserpos, xREC, yREC, *(PosLaserx + ilaserpos),
//						*(PosxScratch + ilaserpos));
				ilaserpos++;
			}
			XY = !XY;
		}
		inFile.close();
	}
	printf(" Laser \u2462: TA.Nb_LaserPositions %d \n", TA.Nb_LaserPositions);

	/** Allocation of memory of intermediate values - for test and validation
	 *
	 */

	printf(" Laser \u2462 min and max x %g %g, min and max y %g %g ... ",
			maxLaserx, minLaserx, maxLasery, minLasery);
	TA.maxLaserx = maxLaserx;
	TA.maxLasery = maxLasery;
	TA.minLaserx = minLaserx;
	TA.minLasery = minLasery;
	printf("number of distrib %d\n", Ndistrib);

	if (TA.Nb_LaserPositions < smallnumber)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			printf(
					" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
			printf(
					" Laser \u2462 Laser position %d  Laser position x: %f  y: %f\n",
					ival, *(PosLaserx + ival), *(PosLasery + ival));
			printf(" Laser \u2462 Position in scratchpad x: %d y: %d \n",
					*(PosxScratch + ival), *(PosyScratch + ival));
			printf(
					" Laser \u2462 ***************SCRATCHPAD FULL OFFSET %d **************\n",
					*(offsetFULL + ival));
			printf(
					" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
		}
}

bool validateLaserPositions_control(void) {

	double Delx { 0.0 }, Dely { 0.0 };
	bool testLaserPosition = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateLaserPositions_device<<<dimGrid, dimBlock, 0>>>(
			TA.Nb_LaserPositions);
	hipDeviceSynchronize();

	for (int iLaser = 0; iLaser < TA.Nb_LaserPositions; iLaser++) {
		Delx += PosLaserx[iLaser] - d_PosLaserx[iLaser];
		Dely += PosLasery[iLaser] - d_PosLasery[iLaser];
	}
	Sumdel[2] = sqrt(Delx * Delx + Dely * Dely);
	printf(" Laser \u2462 delx %g dely %g Sumdel[2] %g \n", Delx, Dely,
			Sumdel[2]);
	if (Delx * Dely == 0.0f)
		testLaserPosition = TRUE;
	return (testLaserPosition);
}

void readstoreCroppedROI(void) {
	XMLDocument XMLdoc;
	XMLElement *pRoot, *pParm;
	string sstr;
	int numberofimages, offsetimages = 0, missedpoints = 0;
	XMLDocument doc;

	filename = resourcesdirectory + "reconst_preprocess_results.xml";
	printf(" ROI \u2463 ROI positions:  %s \n", filename.c_str());
	int LoadOK = XMLError(XMLdoc.LoadFile(filename.c_str()));

	hipMallocManaged(&ROIx, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&ROIy, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&d_ROIx, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&d_ROIy, TA.Nb_LaserPositions * sizeof(int));

	if (!LoadOK) {
		pRoot = XMLdoc.FirstChildElement("Reconstruction_Preproc_Results");
		if (pRoot) {
			// Parse parameters
			pParm =
					pRoot->FirstChildElement("Measurement_AOIs")->FirstChildElement(
							"Distrib");
			while (pParm) {
				numberofimages = atoi(pParm->Attribute("length"));
				missedpoints = 0;
				printf(" ROI \u2463 number of images %d \n", numberofimages);
				sstr = pParm->GetText();
				for (unsigned int i = 0; i < strlen(chars); ++i)
					sstr.erase(std::remove(sstr.begin(), sstr.end(), chars[i]),
							sstr.end());
				stringstream stream(sstr);
				for (int i = offsetimages; i < offsetimages + numberofimages;
						i++) {
					stream.getline(buff, 10, ',');
					int temp = atoi(buff);
					*(ROIx + i) = temp;
					stream.getline(buff, 10, ',');
					temp = atoi(buff);
					*(ROIy + i) = temp;
					stream.getline(buff, 10, ','); // size of window in x constant
					temp = atoi(buff);
					stream.getline(buff, 10, ','); // size of window in y constant
					temp = atoi(buff);
					if (verbose && (!(i % 512)))
						printf(" ROI \u2463 i= %d ROI [%d,%d]\n", i,
								*(ROIx + i), *(ROIy + i));
					if (verbose && (*(ROIx + i) == 0) && (*(ROIy + i) == 0))
						missedpoints++;
					if (verbose && (*(ROIx + i) == 0) && (*(ROIy + i) == 0)
							&& (!(i % 16)))
						printf(" ROI \u2463 i= %d, ", i);
				}
				if (missedpoints)
					printf(" ROI \u2463 \n missed points %d", missedpoints);

				pParm = pParm->NextSiblingElement("Distrib");
				offsetimages += numberofimages;
			}
		}
	}
	TA.maxROIx = 0;
	TA.maxROIy = 0;
	TA.minROIx = 512 * 512;
	TA.minROIy = 512 * 512;

	for (int i = 0; i < TA.Nb_LaserPositions; i++) {
		if (*(ROIx + i) > TA.maxROIx)
			TA.maxROIx = *(ROIx + i);
		if (*(ROIy + i) > TA.maxROIy)
			TA.maxROIy = *(ROIy + i);
		if (*(ROIx + i) < TA.minROIx)
			TA.minROIx = *(ROIx + i);
		if (*(ROIy + i) < TA.minROIy)
			TA.minROIy = *(ROIy + i);
	}
	printf(" ROI \u2463 min and max ROI x: min %d max %d y: min %d max %d\n",
			TA.minROIx, TA.maxROIx, TA.minROIy, TA.maxROIy);
	if (TA.Nb_LaserPositions < smallnumber)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			printf(
					" ROI \u2463 ---------------------------------------------------------------------\n");
			printf(" ROI \u2463 ROI position %d  ROI x: %d  y: %d\n", ival,
					*(ROIx + ival), *(ROIy + ival));
			printf(
					" ROI \u2463 ---------------------------------------------------------------------\n");
		}

}

bool validateCroppedROI_control(void) {

	double Delx, Dely;
	bool testROI = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateCroppedROI_device<<<dimGrid, dimBlock, 0>>>(TA.Nb_LaserPositions);
	hipDeviceSynchronize();
	for (int iLaser = 0; iLaser < TA.Nb_LaserPositions; iLaser++) {
		Delx += ROIx[iLaser] - d_ROIx[iLaser];
		Dely += ROIy[iLaser] - d_ROIy[iLaser];
	}
	Sumdel[3] = Delx * Dely;
	if (Delx * Dely == 0.0f)
		testROI = TRUE;
	return (testROI);
}

