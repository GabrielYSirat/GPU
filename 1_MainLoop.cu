#include "hip/hip_runtime.h"
/*
 * MainLoop.cu
 *
 *  Created on: November 23, 2017
 *      Author: Gabriel Y Sirat */
/** Contains  the next simulation of microimages, in the full measured surface
 *  with optionally number of laser positions below 16 the value of NIMAGESPARALLEL!!
 *  **/
#include "NewLoop.h"
#define VERBOSELOOP 1
#include "include.tst"
__managed__ float *new_simus , *Data , *Rfactor, *distribvalidGPU;

__managed__ float EnergyGlobal;
__global__ void BigLoop(devicedata DD) {

	extern __shared__ int shared[]; /***************semi-global variables stored in shared memory ***************/
	int *image_to_scratchpad_offset_tile = (int *) shared;// Offset of each image in NIMAGESPARALLEL block
	float *Scratchpad = (float *) &image_to_scratchpad_offset_tile[NIMAGESPARALLEL]; // ASCRATCH floats for Scratchpad
	float *shared_distrib = (float*) &Scratchpad[ASCRATCH]; // XDISTRIB*YDISTRIB floats for distrib

	int MemoryOffsetscratch = 0; // to be redefine with aggregates
	float MaxNewSimus = 0.0f;
	float * scrglobal;

	/*****************constant values & auxiliary variables stored in registers *****************/
	register float PSFDISVAL[MAXTHRRATIO] = { 0.0f };// multiplication of pPSF and distribution
	register int tmpi[MAXTHRRATIO], ipixel[MAXTHRRATIO], jpixel[MAXTHRRATIO],
			valid_pixel[MAXTHRRATIO], distribpos0[MAXTHRRATIO], distribpos[MAXTHRRATIO];

	/****************Larger segmented areas to be stored in registers **************************/
	// new simus values kept in registers for speed issues
	register float new_simu_inregister_float_0[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_1[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_2[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_3[NIMAGESPARALLEL] = { 0.0f };
	// Running position on the scratchpad, different for:

	/***** INITIALIZATION *****************/

	DD.step = 0;
	int ithreads = threadIdx.x;
	int itb = blockIdx.x + blockIdx.y + blockIdx.z;
	int itc = ithreads + blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
	int distrib_number = blockIdx.z;
	int iprint = !VERBOSELOOP + itc;

	int center_distrib = ((YDistrib / 2) * XDistrib) + XDistrib / 2;
	int center_microimage = (PixZoomo2) * PixZoom + PixZoomo2;
	DD.step++; time_init = clock64();  time_start = time_init;

#ifdef STARTDEVICE
	if (!iprint) { 	// the condition is required to have it printed once
		printf("\n\u2460********************************** START *****************************\n");
		printf("DEVICE: \u2460****************PARAMETERS OF MEASUREMENT *******************\n");
		printf("DEVICE: \u2460 PARAMETERS  NThreads %d Npixel %d pZOOM %d, pPSF %d\n", NThreads, Npixel, pZOOM, pPSF);
		printf("DEVICE: \u2460 PARAMETERS dimBlock  x: %d y: %d z: %d   ...   ", blockDim.x, blockDim.y, blockDim.z);
		printf("dimGrid  x: %d y: %d z: %d\n", gridDim.x, gridDim.y, gridDim.z);
		printf("DEVICE: \u2460 PARAMETERS pPSF %d XDistrib %d YDistrib %d ADistrib %d\n", pPSF, XDistrib, YDistrib, ADistrib);
		printf("DEVICE: \u2460 PARAMETERS XSCRATCH %d YSCRATCH %d XTILE %d YTILE %d\n", XSCRATCH, YSCRATCH, XTile, YTile);
		printf("DEVICE: \u2460 PARAMETERS Number of pixels calculated in parallel %d Number of threads used"
				" %d loop on threads %d\n", NThreads, THREADSVAL, THreadsRatio);
		printf("DEVICE: \u2460  TILES: XSCRATCH %d, YSCRATCH %d  iprint %d", XSCRATCH, YSCRATCH,iprint);
		printf("XTILE %d, YTILE %d\n", XTile, YTile);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Aggregates in x: %d in y:%d\n", DD.NbAggregx,
				DD.NbAggregy);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Tiles per aggregates in x: %d in y:%d\n",
				DD.tileperaggregatex, DD.tileperaggregatey);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Tiles in x: %d in y:%d\n", DD.NbTilex, DD.NbTiley);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Max number of laser position in Tile: %d min value:%d Number of blocks %d\n",
				DD.maxLaserintile, DD.minLaserintile, DD.blocks);
		printf("\u2460*******************************PARAMETERS OF MEASUREMENT ***************\n");
	}
	__syncthreads();
	if(!ithreads && VERBOSELOOP) printf("TEST: block x %d y %d z %d distrib number %d itc %d itb %d\n",
			blockIdx.x, blockIdx.y, blockIdx.z, distrib_number, itc, itb);
	if (!itc) time_start = clock64(); if (!itc) timer = clock64();
		if (!iprint)  	// the condition is required to have it printed once
			printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING ** processing  %g from start  %g  total %g \n\n",
					DD.step, (float) (timer - time_start) / CLOCKS_PER_SEC,
					(float) (  time_start - time_init) / CLOCKS_PER_SEC,
					(float) (timer - time_init) / CLOCKS_PER_SEC);
		__syncthreads();


#endif

	/***************************Basic parameters **************************************************/
	/*************************Threads and pixels related parameters *******************************/

	for (int apix = 0; apix < THreadsRatio; apix++) { // ipixel, jpixel have 0 values too often
		tmpi[apix] = (ithreads + apix * THREADSVAL);
		ipixel[apix] = tmpi[apix] % PixZoom - PixZoomo2; // centered on the center of the zoomed microimage
		jpixel[apix] = tmpi[apix] / PixZoom - PixZoomo2; // centered on the center of the zoomed microimage
		valid_pixel[apix] = tmpi[apix] < PixZoomSquare;
		distribpos0[apix] = center_distrib + ipixel[apix] - PSFZoomo2
				+ (jpixel[apix] - PSFZoomo2) * XDistrib;
	}
#ifdef TESTTHREADS
if (!iprint) printf("\n\u2461*******************************DEVICE:  THREADS *********************\n");

	if (!itb)
		for (int apix = 0; apix < THreadsRatio; apix++){
			if (!ipixel[apix] && !jpixel[apix])
				printf(
						"DEVICE CENTER: \u2461 THREAD3 : apix %d itc %d tmpi %d ipixel %d, jpixel %d  valid %d distribpos0 %d center %d\n",
						apix, itc, tmpi[apix], ipixel[apix], jpixel[apix], valid_pixel[apix],
						distribpos0[apix], center_distrib);
			int tmpi = (ithreads + apix * THREADSVAL);
				if ((ithreads == 0)||(ithreads == (THREADSVAL-1)) ||(tmpi == PixZoomSquare-1) ||(tmpi == PixZoomSquare))
				printf("DEVICE: \u2461 THREAD1 : apix %d  itc %d ipixel %d, jpixel %d  valid %d distribpos0 %d\n",
						apix, itc, ipixel[apix], jpixel[apix], valid_pixel[apix], distribpos0[apix]);
		}
	if (!iprint) printf("\u2461 **********************************DEVICE:  THREADS  ********************\n\n");
__syncthreads();
#endif
/*{
		if(tmpi == (PixZoomSquare-1) || tmpi == PixZoomSquare)
		printf("DEVICE: \u2461 THREAD2 : apix %d itc %d ipixel %d, jpixel %d  valid %d distribpos0 %d\n",
				apix, itc, ipixel[apix], jpixel[apix], valid_pixel[apix], distribpos0[apix]);
		for (int apix = 0; apix < THreadsRatio; apix++)
	}	__syncthreads(); */


}

