#include "hip/hip_runtime.h"
/*
 * MainLoop.cu
 *
 *  Created on: November 23, 2017
 *      Author: Gabriel Y Sirat */
/** Contains  the next simulation of microimages, in the full measured surface
 *  with optionally number of laser positions below 16 the value of NIMAGESPARALLEL!!
 *  **/
#include "0_NewLoop.h"
#define VERBOSELOOP 1
#include "include.tst"
__managed__ float *new_simus, *Data, *Rfactor, *distribvalidGPU;
__managed__ float EnergyGlobal;

__global__ void BigLoop(devicedata DD) {

	extern __shared__ int shared[]; /***************semi-global variables stored in shared memory ***************/
	int *image_to_scratchpad_offset_tile = (int *) shared; // Offset of each image in NIMAGESPARALLEL block
	float *Scratchpad = (float *) &image_to_scratchpad_offset_tile[NIMAGESPARALLEL]; // ASCRATCH floats for Scratchpad
	float *shared_distrib = (float*) &Scratchpad[ASCRATCH]; // ASCRATCH floats for distrib

	int MemoryOffsetscratch = 0; // to be redefine with aggregates
	float MaxNewSimus = 0.0f;
	float * scrglobal;

	/*****************constant values & auxiliary variables stored in registers *****************/
	register float PSFDISVAL[MAXTHRRATIO] = { 0.0f }; // multiplication of pPSF and distribution
	register int tmpi[MAXTHRRATIO], ipixel[MAXTHRRATIO], jpixel[MAXTHRRATIO], valid_pixel[MAXTHRRATIO],
			distribpos0[MAXTHRRATIO], distribpos[MAXTHRRATIO];

	/****************Larger segmented areas to be stored in registers **************************/
	// new simus values kept in registers for speed issues
	register float new_simu_inregister_float_0[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_1[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_2[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_3[NIMAGESPARALLEL] = { 0.0f };
	// Running position on the scratchpad, different for:

	/***** INITIALIZATION *****************/

	int ithreads = threadIdx.x; int distrib_number = blockIdx.z;
	int itb = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y; int itc = ithreads + itb;
	int iprint = !VERBOSELOOP + itc; int jprint = !VERBOSELOOP + itb;

	int center_distrib = ((YDistrib / 2) * XDistrib) + XDistrib / 2;
	int center_microimage = (PixZoomo2) * PixZoom + PixZoomo2;
	DD.step = 1; time_init = clock64(); time_start = time_init;

#include "8_startdevice.cu"

	/***************************Basic parameters **************************************************/
	/*************************Threads and pixels related parameters *******************************/

	for (int apix = 0; apix < THreadsRatio; apix++) { // ipixel, jpixel have 0 values too often
		tmpi[apix] = (ithreads + apix * THREADSVAL);
		ipixel[apix] = tmpi[apix] % PixZoom - PixZoomo2; // centered on the center of the zoomed microimage
		jpixel[apix] = tmpi[apix] / PixZoom - PixZoomo2; // centered on the center of the zoomed microimage
		valid_pixel[apix] = tmpi[apix] < PixZoomSquare;
		distribpos0[apix] = center_distrib + ipixel[apix] - PSFZoomo2 + (jpixel[apix] - PSFZoomo2) * XDistrib;
	}
#include "8_testthreads.cu"

	/*************************************************************************************************/
	/**O. Initialize zoomed distrib as calculated  by the preprocessing                               /
	 /** the mosaic has to be prepared before hand on the host and copied in global memory            /
	 /************************************************************************************************/
#pragma unroll
	for (int idistrub = ithreads; idistrub < ADistrib; idistrub += THREADSVAL)
			*(shared_distrib + idistrub) = *(original_distrib + idistrub + distrib_number * ADistrib);

#include "8_testdistrib.cu"
}

