#include "hip/hip_runtime.h"
/*
 * gpu_initialization.cu
 *
 *  Created on: Jun 23, 2017
 *      Author: gabriel
 */

#include"0_Mainparameters.h"

double Timestep[16];
string stepname[] = { "initialization  ", "PSF  ", " distrib  ", " Laser positions", " Measurement ROI  ",
		" microimages  ", " laser in tile  ", " microimages in tile  ", " reconstruction  ", " scratchpad    ",
		" bigLoop  ", " end bigloop  ", " bigloop results  "};
double Sumdel[16] = { 0 };
string Stepdiag[16] = NULL;

void report_gpu_mem() {
	size_t free, total;
	float freeMB, totalMB;
	hipMemGetInfo(&free, &total);
	freeMB = (float) free / (1024 * 1024);
	totalMB = (float) total / (1024 * 1024);
	verbosefile << endl << "******************Completion of GPU initialization ***************" << endl;
	verbosefile << "******************************************************************" << endl;
	verbosefile << "used MB =  " << totalMB - freeMB << "   Free MB = " << freeMB << " Total MB = " << totalMB
			<< std::endl;
	verbosefile << "MAIN PROGRAM  \u2776 End of data preparation in device memory ...\n";
}

void GPU_init::start(void) {
	/* pPSF */
	Nb_Rows_PSF = pPSF * pZOOM;
	Nb_Cols_PSF = pPSF * pZOOM;
	PSF_size = (pPSF * pZOOM) * (pPSF * pZOOM);
	/* Reconstruction */
	Nb_Cols_reconstruction = 0;
	Nb_Rows_reconstruction = 0;

	/*Laser positions and MicroImages*/
	Nb_Rows_microimages = Npixel;
	Nb_Cols_microimages = Npixel;
	maxLaserx = 0.0;
	maxLasery = 0.0;
	minLaserx = 1.E6;
	minLasery = 1.E6;

}

void COS::start(void) {
	offsetLaserx = 0.0;
	offsetLasery = 0.0;
	offsetROIx = 0.0;
	offsetROIy = 0.0;
	offsetmicroimagesx = 0.0;
	offsetmicroimagesy = 0.0;
	offsetPSFx = 0.0;
	offsetPSFy = 0.0;
	offsetdistribx = 0.0;
	offsetdistriby = 0.0;
	scaleLaserx = 1.0;
	scaleLasery = 1.0;
	scaleROIx = 1.0;
	scaleROIy = 1.0;
	scalemicroimagesx = 1.0;
	scalemicroimagesy = 1.0;
	scalePSFx = 1.0;
	scalePSFy = 1.0;
	scaledistribx = 1.0;
	scaledistriby = 1.0;
}

void Ctile::print() const {
	verbosefile << "previous calculation: Number of Aggregates in x:" << NbAggregx << " in y:" << NbAggregy;
	verbosefile << " Number of Tiles per aggregates in x:" << tileperaggregatex << " in y:"
			<< tileperaggregatey << endl;
	verbosefile << "Number of Tiles in x:" << NbTilex << " in y:" << NbTiley << endl;
	verbosefile << "Max number of laser position in Tile:" << maxLaserintile << " min value" << minLaserintile
			<< endl << endl;
}

void stepinit(int test, int& stepval) {
	Timestep[stepval] = ((float) (timer - time_start)) / clockRate;
	float Timetotal = ((float) (timer - time_init)) / clockRate;
	if (Sumdel[stepval] == 0)
		Stepdiag[stepval] = "PASS";
	else
		Stepdiag[stepval] = Sumdel[stepval];

	if (test)
		verbosefile << "+++" << stepname[stepval] << " Test validated++++ " << Stepdiag[stepval];
	else
		verbosefile << "---" << stepname[stepval] << " Test not validated++++  Sumdel =  " << Sumdel[stepval];
	if (stepval != 0)
		verbosefile << std::fixed << " \u23F1 msec " << " device  " << Timestep[stepval] << "  total "
				<< Timetotal << endl;
	verbosefile << "END STEP	*******end of step  " << stepval << "  " << stepname[stepval]
			<< "**********************************" << endl << endl;
	stepval++;
		verbosefile << "START STEP	*************  step " << stepval << "  " << stepname[stepval]
				<< "*************" << endl;

}

int retrieveargv(string argvdata) {
	string name, value;
	stringstream ss(argvdata);
	getline(ss, name, '=');
	getline(ss, value);
	int result = atoi(value.c_str());
	return (result);
}

bool T4Dto2D(unsigned char *matrix4D, unsigned char *matrix2D, int dimension4, int dimension3, int dimension2,
		int dimension1) {
	int max4D = 0, max2D = 0;
	for (int i1 = 0; i1 < dimension1; i1++)
		for (int i2 = 0; i2 < dimension2; i2++)
			for (int i3 = 0; i3 < dimension3; i3++)
				for (int i4 = 0; i4 < dimension4; i4++) {
					int xvalue = (i4 * dimension1 + i1);
					int yvalue = (i3 * dimension2 + i2);
					*(matrix4D + xvalue + yvalue * dimension1 * dimension3) = *(matrix2D
							+ i4 * dimension3 * dimension2 * dimension1 + i3 * dimension2 * dimension1
							+ i2 * dimension1 + i1);
				}
	for (int i1 = 0; i1 < dimension4 * dimension3 * dimension2 * dimension1; i1++) {
		max4D = max(max4D, *(matrix4D + i1));
		max2D = max(max2D, *(matrix2D + i1));

	}
	return (TRUE);
}

float displaydata(float * datavalues, int stepval) {
	float MaxData = 0.0f;
	int n_colintern = PixZoom * tile.blocks * tile.NbTilex;
	int n_rowintern = PixZoom * NIMAGESPARALLEL * tile.NbTiley;
	string stepnumber, dataliteral, callprogram, filebase;

	if (stepval == 12)
		stepnumber.append("\u24EF");
	if (stepval == 12)
		dataliteral.append("SimusA1");
	if (stepval == 12)
		callprogram.append("biginspect.cu");
	if (stepval == 12)
		filebase.append("results/F_simusA1.pgm");

	if (stepval == 13)
		stepnumber.append("\u24F0");
	if (stepval == 13)
		dataliteral.append("RFactorA1");
	if (stepval == 13)
		callprogram.append("biginspect.cu");
	if (stepval == 13)
		filebase.append("results/G_RFactorA1.pgm");

	if (stepval == 7)
		stepnumber.append("\u24FB");
	if (stepval == 7)
		dataliteral.append("MicroimagesA1");
	if (stepval == 7)
		callprogram.append("tileorganization.cu");
	if (stepval == 7)
		filebase.append("results/C_microimagesdeviceloop.pgm");

	unsigned char *i_data = (unsigned char *) calloc(n_colintern * n_rowintern, sizeof(unsigned char)); // on host

	for (int i = 0; i < tile.maxLaserintile * NThreads; i++)
		MaxData = max(MaxData, *(datavalues + i));
	verbosefile << "HOST: " << stepnumber.c_str() << "  " << stepval << "parameters " << " n_rowintern "
			<< n_rowintern;
	verbosefile << "n_colintern " << n_colintern << "MaxData " << MaxData;
	verbosefile << " dataliteral.c_str() " << dataliteral.c_str() << " callprogram.c_str() "
			<< callprogram.c_str() << endl;

	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {

		const char * DataFile = filebase.c_str();

		for (int idistrib = 0, disdelta = 0; idistrib < Ndistrib;
				idistrib++, disdelta += tile.Nblaserperdistribution[idistrib])
			for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
				int tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startxdomain) / XTile;
				int tiley = pZOOM * (*(PosLasery + iLaser) - tile.startydomain) / YTile;
				int tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
				int ilasertile = tilenumber * tile.maxLaserintile + tile.posintile[iLaser];
				verbosefile << "TILE ORG \u24FA idistrib " << idistrib << "  " << iLaser << " iLaser "
						<< iLaser;
				verbosefile << " tilenumber " << tilenumber << " ilasertile " << ilasertile << endl;
				for (int ipix = 0; ipix < PixZoomSquare; ipix++) { // copy microimage to its position in the Data
					*(Data + ilasertile * PixZoomSquare + ipix) = *(zoomed_microimages
							+ iLaser * PixZoomSquare + ipix);
					int xpix = ipix % PixZoom;
					int ypix = ipix / PixZoom;
					i_data[tilenumber * PixZoom + tile.posintile[iLaser] * PixZoomSquare * tile.maxLaserintile
							+ xpix + PixZoom * tile.maxLaserintile * ypix] = 255.0
							* (*(datavalues + ilasertile * PixZoomSquare + ipix) - Minmicroimages)
							/ (Maxmicroimages - Minmicroimages);
				}
			}

		sdkSavePGM(DataFile, i_data, tile.maxLaserintile * PixZoom, tile.NbTileXYD * PixZoom);
		verbosefile << "HOST: " << stepnumber.c_str() << "  " << stepval
				<< " ******************************************\n\n";
	}
	return (MaxData);
}
int sizesimus = tile.maxLaserintile * tile.NbTileXY * NThreads;

float displaySimus(float * simusvalues) {
/*	float MaxSimusD = 0.0f, MinSimusD = 1.E6;
	unsigned char *i_simus = (unsigned char *) calloc(sizesimus, sizeof(unsigned char)); // on host
	string filebase, file;
	int n_colintern = PixZoom * tile.NbTileXY;
	int n_rowintern = PixZoom * tile.maxLaserintile;

	filebase.append("results/F_simus");

	for (int i = 0; i < datafullsize; i++) MaxSimusD = max(MaxSimusD, *(simusvalues + i)); // all distributions!!
	for (int i = 0; i < datafullsize; i++) MinSimusD = min(MinSimusD, *(simusvalues + i));
	float ratio = 255. / (MaxSimusD - MinSimusD);*/

/*	verbosefile << "HOST: \u24EF parameters: n_rowintern " << n_rowintern << "n_colintern " << n_colintern
			<< "MaxSimusD " << MaxSimusD << " MinSimusD " << MinSimusD << " Simulations call program biginspect.cu " << endl;

/*	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		file = filebase + to_string(idistrib) + ".pgm";
		verbosefile << "file " << file << endl;

		for (int isimus = 0; isimus < sizesimus; isimus++) {
			int imicro = isimus / NThreads; // number of microimage
			int ipixel = isimus % NThreads; // pixel number in microimage
			if (ipixel < PixZoomSquare) {
				int ix = ipixel % PixZoom + PixZoom * imicro % tile.maxLaserintile;
				int iy = ipixel / PixZoom + imicro/tile.maxLaserintile *PixZoomSquare;
				i_simus[isimus] = ratio * (simusvalues[ix + iy * PixZoom] - MinSimusD);
			}
		}
		sdkSavePGM(file.c_str(), i_simus, PixZoom, PixZoom*fullnumberoflasers);
	}*/

	return (TRUE);
}

float scratchreaddisplay(float * reconstructiondata, float * scratchdata, const char * filename, bool readtile) {
	unsigned char *i_scratchpad = (unsigned char *) calloc(tile.NbTileXY * XSCRATCH * YSCRATCH, sizeof(unsigned char));
	float MaxScratchlocal = 0.0f;
	for (int iy = 0; iy < tile.NbTiley; iy++)
		for (int ix = 0; ix < tile.NbTilex; ix++)
			for (int iix = 0; iix < XTile; iix++)
				for (int iiy = 0; iiy < YTile; iiy++) {

					int iscratch = lostpixels + iix + dxSCRo2; 		// contribution of x in the 1D SCRATCH
					iscratch += ix * XSCRATCH; 					// contribution of previous tiles in x
					iscratch += (iiy + dySCRo2) * XSCRATCH * tile.NbTilex; 	// contribution of y in 1D SCRATCH
					iscratch += iy * ASCRATCH * tile.NbTilex; // contribution of previous tiles in y

					int itile = iix;  // contribution of x in the TILE
					itile += ix * XTile; // contribution of previous tile in x
					itile += iiy * XTile * tile.NbTilex; // contribution of y in the TILE
					itile += iy * ATile * tile.NbTilex; // contribution of previous tiles in y

					int iscratch2Dx = iix + dxSCRo2 + ix * XSCRATCH; // contribution of x in the 1D SCRATCH + contribution of previous tiles in x
					int iscratch2Dy = iiy + dySCRo2 + iy * YSCRATCH; // contribution of y in 1D SCRATCH +contribution of previous tiles in y
					int iscratch2D = iscratch2Dx + iscratch2Dy * XSCRATCH * tile.NbTilex;
					if (readtile)
						scratchdata[iscratch] = reconstructiondata[itile];
					MaxScratchlocal = max(MaxScratchlocal, scratchdata[iscratch]);
					i_scratchpad[iscratch2D] = 255.0 * scratchdata[iscratch] / Maxscratch;
					if (!(i_scratchpad[iscratch2D] == 0) && NOVERBOSE) {
						printf(
								"SCRATCHPAD \u24FC itile %d, iscratch %d iscratch2Dx %d, iscratch2Dy %d iscratch2D %d\n",
								itile, iscratch, iscratch2Dx, iscratch2Dy, iscratch2D);
						printf(
								"SCRATCHPAD \u24FC itile %d, i_scratchpad[iscratch2D] %d val_scratchpad[arg1D] %f\n",
								itile, i_scratchpad[iscratch2D], scratchdata[iscratch]);

					}
				}
	sdkSavePGM(filename, i_scratchpad, XSCRATCH * tile.NbTilex, YSCRATCH * tile.NbTiley);
	return (MaxScratchlocal);
}

float scratch2D2tile(float * fscratch2D, float * ftile, int fxtile, int fytile, int fXscratch, int fYscratch)
{
	float maxtile = 0.0f;

	int del = fXscratch - fxtile + (fYscratch - fytile)*fXscratch;

	for (int itile=0; itile < fxtile*fytile; itile++) {
			ftile[itile] = fscratch2D[itile+del];
			maxtile = max(maxtile, fscratch2D[itile+del]);
		}

	return maxtile;
}

float tile2scratch2D(float * fscratch2D, float * ftile, int fxtile, int fytile, int fXscratch, int fYscratch)
{
	float maxscratch = 0.0f;

	for(int iscratch=0; iscratch < fXscratch*fYscratch; iscratch++)
		fscratch2D[iscratch]=0.0f;

	int del = fXscratch - fxtile + (fYscratch - fytile)*fXscratch;

	for (int itile=0; itile < fxtile*fytile; itile++) {
		fscratch2D[itile+del] = ftile[itile];
		maxscratch = max(maxscratch, fscratch2D[itile+del]);
		}

	return maxscratch;
}

float scratch2D2scratch1D(float * fscratch2D, float * fscratch1D, int fXscratch, int fYscratch, int fAscratch, int flostpixels)
{
	float maxscratch1D = 0.0f;
/* first lost pixels to zero
 *
 */
	for (int iscratch = 0; iscratch < flostpixels; iscratch++) fscratch1D[iscratch] = 0.0f;
/* fXscratch * fYscratch real pixels
 *
 */
	for (int iscratch2D=0; iscratch2D < fXscratch*fYscratch; iscratch2D++) {
			fscratch1D[flostpixels+iscratch2D] = fscratch2D[iscratch2D];
			maxscratch1D = max(maxscratch1D, fscratch2D[iscratch2D]);
		}
	/* last pixels
	 *
	 */
	for (int iscratch = fXscratch*fYscratch+flostpixels; iscratch < fAscratch; iscratch++)
		fscratch1D[iscratch] = 0.0f;

	return maxscratch1D;
}

float scratch1D2scratch2D(float * fscratch2D, float * fscratch1D, const char * filename, int fXscratch,
		int fYscratch, int flostpixels) {
	float maxscratch2D = 0.0f;
	unsigned char *i_scratch2D = (unsigned char *) calloc(fXscratch * fYscratch, sizeof(unsigned char));
	/* fXscratch * fYscratch real pixels
	 *
	 */
	for (int iscratch2D = 0; iscratch2D < fXscratch * fYscratch; iscratch2D++) {
		fscratch2D[iscratch2D] = fscratch1D[flostpixels + iscratch2D];
		maxscratch2D = max(maxscratch2D, *(fscratch2D + iscratch2D));
	}
	float ratio = 255. / maxscratch2D;

	for (int iscratch2D = 0; iscratch2D < fXscratch * fYscratch; iscratch2D++)
		i_scratch2D[iscratch2D] = ratio * fscratch2D[iscratch2D];

	if (filename != NULL)
		sdkSavePGM(filename, i_scratch2D, fXscratch, fYscratch);
	return maxscratch2D;
}
