#include "hip/hip_runtime.h"
/*
 * gpu_initialization.cu
 *
 *  Created on: Jun 23, 2017
 *      Author: gabriel
 */

#include"0_NewLoop.h"
string stepname[] = {"initialization  ", "PSF   ", "distrib  ",
		"Laser positions", "Measurement ROI  ", "microimages", " laser in tile ", "microimages in tile",  "reconstruction  ",
		"scratchpad    ", "bigLoop" , "end bigloop", "bigloop results"};
int smallnumber =20;
double Sumdel[16] = { 0 };
string Stepdiag[16] = NULL;

void report_gpu_mem()
{
    size_t free, total;
    float freeMB, totalMB;
    hipMemGetInfo(&free, &total);
    freeMB =(float)free/(1024*1024);
    totalMB = (float)total/(1024*1024);
    std::cout << endl << "******************Completion of GPU initialization ***************"<< endl;
    std::cout  << "******************************************************************"<< endl;
    std::cout << "used MB =  " << totalMB - freeMB << "   Free MB = " << freeMB << " Total MB = " << totalMB <<std::endl;
	printf("MAIN PROGRAM  \u2776 End of data preparation in device memory ...\n");
}


void GPU_init::start(void) {
/* pPSF */
	  PSF_Rows = pPSF*pZOOM;
	  Nb_Cols_PSF = pPSF*pZOOM;
	  PSF_size = (pPSF*pZOOM)*(pPSF*pZOOM);
	/* Reconstruction */
	  Nb_Cols_reconstruction = 0;
	  Nb_Rows_reconstruction = 0;

/*Laser positions and MicroImages*/
	  Nb_Rows_microimages = Npixel;
	  Nb_Cols_microimages = Npixel;

}

void COS::start(void) {
	offsetLaserx = 0.0;
	offsetLasery =0.0;
	offsetROIx = 0.0;
	offsetROIy = 0.0;
	offsetmicroimagesx = 0.0;
	offsetmicroimagesy = 0.0;
	offsetPSFx = 0.0;
	offsetPSFy = 0.0;
	offsetdistribx = 0.0;
	offsetdistriby =0.0;
	scaleLaserx = 1.0;
	scaleLasery =1.0;
	scaleROIx = 1.0;
	scaleROIy = 1.0;
	scalemicroimagesx = 1.0;
	scalemicroimagesy = 1.0;
	scalePSFx = 1.0;
	scalePSFy = 1.0;
	scaledistribx = 1.0;
	scaledistriby =1.0;
}

void Ctile::print() const
{
	  cout << "previous calculation: Number of Aggregates in x:" << NbAggregx  << " in y:" << NbAggregy;
	  cout << " Number of Tiles per aggregates in x:" << tileperaggregatex  << " in y:" << tileperaggregatey << endl;
	  cout << "Number of Tiles in x:" << NbTilex  << " in y:" << NbTiley <<endl ;
	  cout << "Max number of laser position in Tile:" << maxLaserintile  << " min value" << minLaserintile <<endl<<endl ;
}

void stepinit(int test, int& stepval)
{
Timestep[stepval] = ((float) (timer - time_start)) / clockRate;
	float Timetotal = ((float) (timer - time_init)) / clockRate;
	if(Sumdel[stepval] == 0)
		Stepdiag[stepval] = "PASS";
	else
		Stepdiag[stepval] = Sumdel[stepval];

	if (test)
		cout << "+++" << stepname[stepval]<< " Test validated++++ " << Stepdiag[stepval];
	else
		cout << "---" << stepname[stepval]<< " Test not validated++++  Sumdel =  " << Sumdel[stepval];
	if(stepval != 0) std::cout << std::fixed << " \u23F1 msec " <<" device  "  << Timestep[stepval]  << "  total " << Timetotal << endl;
	cout << "END STEP	*******end of step  " << stepval << "  " << stepname[stepval] << "**********************************" << endl << endl;
	stepval++;
	if(stepval != 9)
	cout << "START STEP	*************  step " << stepval << "  " << stepname[stepval] << "*************" << endl;

}

int retrieveargv(string argvdata) {
	string name, value;
	stringstream ss(argvdata);
	getline(ss, name, '=');
	getline(ss, value);
	int result = atoi(value.c_str());
	return (result);
}

float displaydata( float * datavalues, int stepval)
{
	float MaxData = 0.0f;
	int n_colintern = PixZoom * tile.blocks *tile.NbTilex;
	int n_rowintern = PixZoom * NIMAGESPARALLEL*tile.NbTiley;
	string stepnumber, dataliteral, callprogram,  filebase;

	if (stepval == 12) stepnumber.append("\u24EF");
	if (stepval == 12) dataliteral.append("SimusA1");
	if (stepval == 12) callprogram.append("biginspect.cu");
	if (stepval == 12) filebase.append("results/simusA1.pgm");

	if (stepval == 13) stepnumber.append("\u24F0");
	if (stepval == 13) dataliteral.append("RFactorA1");
	if (stepval == 13) callprogram.append("biginspect.cu");
	if (stepval == 13) filebase.append("results/RFactorA1.pgm");

	if (stepval == 7) stepnumber.append("\u24EF");
	if (stepval == 7) dataliteral.append("MicroimagesA1");
	if (stepval == 7) callprogram.append("tileorganization.cu");
	if (stepval == 7) filebase.append("results/microimagesB.pgm");

	unsigned char *i_data = (unsigned char *) calloc(n_colintern*n_rowintern, sizeof(unsigned char)); // on host

	for (int i = 0; i < tile.maxLaserintile * NThreads; i++)
		MaxData = max(MaxData, *(datavalues + i));
	printf("HOST: %s %d parameters %s in %s:  n_rowintern %d n_colintern %d, total %d MaxData %g\n",
			stepnumber.c_str(), stepval, dataliteral.c_str(), callprogram.c_str(),
			n_rowintern, n_colintern, tile.maxLaserintile * NThreads, MaxData);

	for (int idistrib = 0; idistrib < Ndistrib; idistrib++){

		const char * DataFile = filebase.c_str ();

		for (int idistrib = 0, disdelta = 0; idistrib < Ndistrib; idistrib++, disdelta += tile.Nblaserperdistribution[idistrib])
			for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
				int tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
				int tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
				int tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
				int ilasertile = tilenumber * tile.maxLaserintile + tile.posintile[iLaser];
				if (VERBOSE) printf("TILE ORG \u247A idistrib %d, iLaser %d tilenumber %d ilasertile %d\n", idistrib, iLaser,
						tilenumber, ilasertile);
				for (int ipix = 0; ipix < PixZoomSquare; ipix++) { // copy microimage to its position in the Data
					*(Data + ilasertile * PixZoomSquare + ipix) = *(zoomed_microimages + iLaser * PixZoomSquare + ipix);
					int xpix = ipix % PixZoom;	int ypix = ipix / PixZoom;
					i_data[tilenumber * PixZoom + tile.posintile[iLaser] * PixZoomSquare * tile.maxLaserintile + xpix + PixZoom * tile.maxLaserintile * ypix]
					       = 255.0 * (*(datavalues + ilasertile * PixZoomSquare + ipix) - Minmicroimages) /(Maxmicroimages - Minmicroimages);
			}
		}

		printf("HOST: %s %d results %s in %s:: %s .....\n",
			 stepnumber.c_str(), stepval, dataliteral.c_str(), callprogram.c_str(),DataFile);
	sdkSavePGM(DataFile, i_data,tile.maxLaserintile *PixZoom , tile.NbTileXYD * PixZoom);
	printf("HOST: %s %d ******************************************\n\n",
			 stepnumber.c_str(), stepval);
}
return (MaxData);
}
