#include "hip/hip_runtime.h"
/*
 * gpu_initialization.cu
 *
 *  Created on: Jun 23, 2017
 *      Author: gabriel
 */

#include"0_Mainparameters.h"
string stepname[] = {"initialization  ", "PSF   ", "distrib  ",
		"Laser positions", "Measurement ROI  ", "microimages", " laser in tile ", "microimages in tile",  "reconstruction  ",
		"scratchpad    ", "bigLoop" , "end bigloop", "bigloop results"};
int smallnumber =20;
double Sumdel[16] = { 0 };
string Stepdiag[16] = NULL;

void report_gpu_mem()
{
    size_t free, total;
    float freeMB, totalMB;
    hipMemGetInfo(&free, &total);
    freeMB =(float)free/(1024*1024);
    totalMB = (float)total/(1024*1024);
    verbosefile << endl << "******************Completion of GPU initialization ***************"<< endl;
    verbosefile  << "******************************************************************"<< endl;
    verbosefile << "used MB =  " << totalMB - freeMB << "   Free MB = " << freeMB << " Total MB = " << totalMB <<std::endl;
    verbosefile << "MAIN PROGRAM  \u2776 End of data preparation in device memory ...\n";
}


void GPU_init::start(void) {
/* pPSF */
	  PSF_Rows = pPSF*pZOOM;
	  Nb_Cols_PSF = pPSF*pZOOM;
	  PSF_size = (pPSF*pZOOM)*(pPSF*pZOOM);
	/* Reconstruction */
	  Nb_Cols_reconstruction = 0;
	  Nb_Rows_reconstruction = 0;

/*Laser positions and MicroImages*/
	  Nb_Rows_microimages = Npixel;
	  Nb_Cols_microimages = Npixel;

}

void COS::start(void) {
	offsetLaserx = 0.0;
	offsetLasery =0.0;
	offsetROIx = 0.0;
	offsetROIy = 0.0;
	offsetmicroimagesx = 0.0;
	offsetmicroimagesy = 0.0;
	offsetPSFx = 0.0;
	offsetPSFy = 0.0;
	offsetdistribx = 0.0;
	offsetdistriby =0.0;
	scaleLaserx = 1.0;
	scaleLasery =1.0;
	scaleROIx = 1.0;
	scaleROIy = 1.0;
	scalemicroimagesx = 1.0;
	scalemicroimagesy = 1.0;
	scalePSFx = 1.0;
	scalePSFy = 1.0;
	scaledistribx = 1.0;
	scaledistriby =1.0;
}

void Ctile::print() const
{
	  verbosefile << "previous calculation: Number of Aggregates in x:" << NbAggregx  << " in y:" << NbAggregy;
	  verbosefile << " Number of Tiles per aggregates in x:" << tileperaggregatex  << " in y:" << tileperaggregatey << endl;
	  verbosefile << "Number of Tiles in x:" << NbTilex  << " in y:" << NbTiley <<endl ;
	  verbosefile << "Max number of laser position in Tile:" << maxLaserintile  << " min value" << minLaserintile <<endl<<endl ;
}

void stepinit(int test, int& stepval)
{
Timestep[stepval] = ((float) (timer - time_start)) / clockRate;
	float Timetotal = ((float) (timer - time_init)) / clockRate;
	if(Sumdel[stepval] == 0)
		Stepdiag[stepval] = "PASS";
	else
		Stepdiag[stepval] = Sumdel[stepval];

	if (test)
		verbosefile << "+++" << stepname[stepval]<< " Test validated++++ " << Stepdiag[stepval];
	else
		verbosefile << "---" << stepname[stepval]<< " Test not validated++++  Sumdel =  " << Sumdel[stepval];
	if(stepval != 0) verbosefile << std::fixed << " \u23F1 msec " <<" device  "  << Timestep[stepval]  << "  total " << Timetotal << endl;
	verbosefile << "END STEP	*******end of step  " << stepval << "  " << stepname[stepval] << "**********************************" << endl << endl;
	stepval++;
	if(stepval != 9)
	verbosefile << "START STEP	*************  step " << stepval << "  " << stepname[stepval] << "*************" << endl;

}

int retrieveargv(string argvdata) {
	string name, value;
	stringstream ss(argvdata);
	getline(ss, name, '=');
	getline(ss, value);
	int result = atoi(value.c_str());
	return (result);
}

bool T4Dto2D( float *matrix4D, float *matrix2D,  int dimension1, int dimension2, int dimension3, int dimension4)
{
for(int i1 =0 ; i1 < dimension1; i1++)
		for(int i2 =0 ; i2 < dimension2; i2++)
			for(int i3 =0 ; i3 < dimension3; i3++)
				for(int i4 =0 ; i4 < dimension4; i4++)
					*(matrix4D + (i4*dimension2 + i2) * dimension3 * dimension1 + (i3*dimension1 + i1))
					= *(matrix2D + i4*dimension3*dimension2*dimension1 + i3*dimension2*dimension1 + i2*dimension1 + i1);

	return(TRUE);
}

bool T4Dto2Di( int *matrix4D, int *matrix2D,  int dimension1, int dimension2, int dimension3, int dimension4)
{
for(int i1 =0 ; i1 < dimension1; i1++)
		for(int i2 =0 ; i2 < dimension2; i2++)
			for(int i3 =0 ; i3 < dimension3; i3++)
				for(int i4 =0 ; i4 < dimension4; i4++)
					*(matrix4D + (i4*dimension2 + i2) * dimension3 * dimension1 + (i3*dimension1 + i1))
					= *(matrix2D + i4*dimension3*dimension2*dimension1 + i3*dimension2*dimension1 + i2*dimension1 + i1);

	return(TRUE);
}

float displaydata( float * datavalues, int stepval)
{
	float MaxData = 0.0f;
	int n_colintern = PixZoom * tile.blocks *tile.NbTilex;
	int n_rowintern = PixZoom * NIMAGESPARALLEL*tile.NbTiley;
	string stepnumber, dataliteral, callprogram,  filebase;

	if (stepval == 12) stepnumber.append("\u24EF");
	if (stepval == 12) dataliteral.append("SimusA1");
	if (stepval == 12) callprogram.append("biginspect.cu");
	if (stepval == 12) filebase.append("results/F_simusA1.pgm");

	if (stepval == 13) stepnumber.append("\u24F0");
	if (stepval == 13) dataliteral.append("RFactorA1");
	if (stepval == 13) callprogram.append("biginspect.cu");
	if (stepval == 13) filebase.append("results/G_RFactorA1.pgm");

	if (stepval == 7) stepnumber.append("\u24EF");
	if (stepval == 7) dataliteral.append("MicroimagesA1");
	if (stepval == 7) callprogram.append("tileorganization.cu");
	if (stepval == 7) filebase.append("results/C_microimagesdeviceloop.pgm");

	unsigned char *i_data = (unsigned char *) calloc(n_colintern*n_rowintern, sizeof(unsigned char)); // on host

	for (int i = 0; i < tile.maxLaserintile * NThreads; i++)
		MaxData = max(MaxData, *(datavalues + i));
	verbosefile << "HOST: " << stepnumber.c_str() << "  " <<  stepval << "parameters " << " n_rowintern " << n_rowintern;
	verbosefile << "n_colintern " << n_colintern << "MaxData " << MaxData;
	verbosefile << " dataliteral.c_str() " << dataliteral.c_str() << " callprogram.c_str() " << callprogram.c_str() << endl;

	for (int idistrib = 0; idistrib < Ndistrib; idistrib++){

		const char * DataFile = filebase.c_str ();

		for (int idistrib = 0, disdelta = 0; idistrib < Ndistrib; idistrib++, disdelta += tile.Nblaserperdistribution[idistrib])
			for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
				int tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
				int tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
				int tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
				int ilasertile = tilenumber * tile.maxLaserintile + tile.posintile[iLaser];
				verbosefile << "TILE ORG \u247A idistrib " << idistrib << "  " << iLaser << " iLaser " << iLaser;
				verbosefile << " tilenumber " << tilenumber << " ilasertile " << ilasertile << endl;
				for (int ipix = 0; ipix < PixZoomSquare; ipix++) { // copy microimage to its position in the Data
					*(Data + ilasertile * PixZoomSquare + ipix) = *(zoomed_microimages + iLaser * PixZoomSquare + ipix);
					int xpix = ipix % PixZoom;	int ypix = ipix / PixZoom;
					i_data[tilenumber * PixZoom + tile.posintile[iLaser] * PixZoomSquare * tile.maxLaserintile + xpix + PixZoom * tile.maxLaserintile * ypix]
					       = 255.0 * (*(datavalues + ilasertile * PixZoomSquare + ipix) - Minmicroimages) /(Maxmicroimages - Minmicroimages);
			}
		}

		sdkSavePGM(DataFile, i_data,tile.maxLaserintile *PixZoom , tile.NbTileXYD * PixZoom);
	verbosefile << "HOST: " << stepnumber.c_str() << "  " <<  stepval << " ******************************************\n\n";
	}
	return (MaxData);
}
