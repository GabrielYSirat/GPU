#include "hip/hip_runtime.h"
/*
 * gpu_initialization.cu
 *
 *  Created on: Jun 23, 2017
 *      Author: gabriel
 */

#include"0_Mainparameters.h"
string stepname[] = {"initialization  ", "PSF   ", "distrib  ",
		"Laser positions", "Measurement ROI  ", "microimages", " laser in tile ", "microimages in tile",  "reconstruction  ",
		"scratchpad    ", "bigLoop" , "end bigloop", "bigloop results"};
int smallnumber =20;
double Sumdel[16] = { 0 };
string Stepdiag[16] = NULL;

void report_gpu_mem()
{
    size_t free, total;
    float freeMB, totalMB;
    hipMemGetInfo(&free, &total);
    freeMB =(float)free/(1024*1024);
    totalMB = (float)total/(1024*1024);
    verbosefile << endl << "******************Completion of GPU initialization ***************"<< endl;
    verbosefile  << "******************************************************************"<< endl;
    verbosefile << "used MB =  " << totalMB - freeMB << "   Free MB = " << freeMB << " Total MB = " << totalMB <<std::endl;
    verbosefile << "MAIN PROGRAM  \u2776 End of data preparation in device memory ...\n";
}


void GPU_init::start(void) {
/* pPSF */
	  PSF_Rows = pPSF*pZOOM;
	  Nb_Cols_PSF = pPSF*pZOOM;
	  PSF_size = (pPSF*pZOOM)*(pPSF*pZOOM);
	/* Reconstruction */
	  Nb_Cols_reconstruction = 0;
	  Nb_Rows_reconstruction = 0;

/*Laser positions and MicroImages*/
	  Nb_Rows_microimages = Npixel;
	  Nb_Cols_microimages = Npixel;

}

void COS::start(void) {
	offsetLaserx = 0.0;
	offsetLasery =0.0;
	offsetROIx = 0.0;
	offsetROIy = 0.0;
	offsetmicroimagesx = 0.0;
	offsetmicroimagesy = 0.0;
	offsetPSFx = 0.0;
	offsetPSFy = 0.0;
	offsetdistribx = 0.0;
	offsetdistriby =0.0;
	scaleLaserx = 1.0;
	scaleLasery =1.0;
	scaleROIx = 1.0;
	scaleROIy = 1.0;
	scalemicroimagesx = 1.0;
	scalemicroimagesy = 1.0;
	scalePSFx = 1.0;
	scalePSFy = 1.0;
	scaledistribx = 1.0;
	scaledistriby =1.0;
}

void Ctile::print() const
{
	  verbosefile << "previous calculation: Number of Aggregates in x:" << NbAggregx  << " in y:" << NbAggregy;
	  verbosefile << " Number of Tiles per aggregates in x:" << tileperaggregatex  << " in y:" << tileperaggregatey << endl;
	  verbosefile << "Number of Tiles in x:" << NbTilex  << " in y:" << NbTiley <<endl ;
	  verbosefile << "Max number of laser position in Tile:" << maxLaserintile  << " min value" << minLaserintile <<endl<<endl ;
}

void stepinit(int test, int& stepval)
{
Timestep[stepval] = ((float) (timer - time_start)) / clockRate;
	float Timetotal = ((float) (timer - time_init)) / clockRate;
	if(Sumdel[stepval] == 0)
		Stepdiag[stepval] = "PASS";
	else
		Stepdiag[stepval] = Sumdel[stepval];

	if (test)
		verbosefile << "+++" << stepname[stepval]<< " Test validated++++ " << Stepdiag[stepval];
	else
		verbosefile << "---" << stepname[stepval]<< " Test not validated++++  Sumdel =  " << Sumdel[stepval];
	if(stepval != 0) verbosefile << std::fixed << " \u23F1 msec " <<" device  "  << Timestep[stepval]  << "  total " << Timetotal << endl;
	verbosefile << "END STEP	*******end of step  " << stepval << "  " << stepname[stepval] << "**********************************" << endl << endl;
	stepval++;
	if(stepval != 9)
	verbosefile << "START STEP	*************  step " << stepval << "  " << stepname[stepval] << "*************" << endl;

}

int retrieveargv(string argvdata) {
	string name, value;
	stringstream ss(argvdata);
	getline(ss, name, '=');
	getline(ss, value);
	int result = atoi(value.c_str());
	return (result);
}

bool T4Dto2D(unsigned char *matrix4D, unsigned char *matrix2D, int dimension4, int dimension3,
		int dimension2, int dimension1) {
int max4D = 0, max2D = 0;
	for (int i1 = 0; i1 < dimension1; i1++)
		for (int i2 = 0; i2 < dimension2; i2++)
			for (int i3 = 0; i3 < dimension3; i3++)
				for (int i4 = 0; i4 < dimension4; i4++) {
					int xvalue = (i4 * dimension1 + i1);
					int yvalue = (i3 * dimension2 + i2);
					*(matrix4D + xvalue + yvalue * dimension1 * dimension3) = *(matrix2D
							+ i4 * dimension3 * dimension2 * dimension1 + i3 * dimension2 * dimension1
							+ i2 * dimension1 + i1);
				}
	for (int i1 = 0; i1 < dimension4 * dimension3 * dimension2 * dimension1; i1++) {
		max4D = max(max4D, *(matrix4D + i1));
		max2D = max(max2D, *(matrix2D + i1));

	}
	printf("Max4D %d, max2D %d \n\n", max4D, max2D);
	return (TRUE);
}

float displaydata( float * datavalues, int stepval)
{
	float MaxData = 0.0f;
	int n_colintern = PixZoom * tile.blocks *tile.NbTilex;
	int n_rowintern = PixZoom * NIMAGESPARALLEL*tile.NbTiley;
	string stepnumber, dataliteral, callprogram,  filebase;

	if (stepval == 12) stepnumber.append("\u24EF");
	if (stepval == 12) dataliteral.append("SimusA1");
	if (stepval == 12) callprogram.append("biginspect.cu");
	if (stepval == 12) filebase.append("results/F_simusA1.pgm");

	if (stepval == 13) stepnumber.append("\u24F0");
	if (stepval == 13) dataliteral.append("RFactorA1");
	if (stepval == 13) callprogram.append("biginspect.cu");
	if (stepval == 13) filebase.append("results/G_RFactorA1.pgm");

	if (stepval == 7) stepnumber.append("\u24EF");
	if (stepval == 7) dataliteral.append("MicroimagesA1");
	if (stepval == 7) callprogram.append("tileorganization.cu");
	if (stepval == 7) filebase.append("results/C_microimagesdeviceloop.pgm");

	unsigned char *i_data = (unsigned char *) calloc(n_colintern*n_rowintern, sizeof(unsigned char)); // on host

	for (int i = 0; i < tile.maxLaserintile * NThreads; i++)
		MaxData = max(MaxData, *(datavalues + i));
	verbosefile << "HOST: " << stepnumber.c_str() << "  " <<  stepval << "parameters " << " n_rowintern " << n_rowintern;
	verbosefile << "n_colintern " << n_colintern << "MaxData " << MaxData;
	verbosefile << " dataliteral.c_str() " << dataliteral.c_str() << " callprogram.c_str() " << callprogram.c_str() << endl;

	for (int idistrib = 0; idistrib < Ndistrib; idistrib++){

		const char * DataFile = filebase.c_str ();

		for (int idistrib = 0, disdelta = 0; idistrib < Ndistrib; idistrib++, disdelta += tile.Nblaserperdistribution[idistrib])
			for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
				int tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
				int tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
				int tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
				int ilasertile = tilenumber * tile.maxLaserintile + tile.posintile[iLaser];
				verbosefile << "TILE ORG \u247A idistrib " << idistrib << "  " << iLaser << " iLaser " << iLaser;
				verbosefile << " tilenumber " << tilenumber << " ilasertile " << ilasertile << endl;
				for (int ipix = 0; ipix < PixZoomSquare; ipix++) { // copy microimage to its position in the Data
					*(Data + ilasertile * PixZoomSquare + ipix) = *(zoomed_microimages + iLaser * PixZoomSquare + ipix);
					int xpix = ipix % PixZoom;	int ypix = ipix / PixZoom;
					i_data[tilenumber * PixZoom + tile.posintile[iLaser] * PixZoomSquare * tile.maxLaserintile + xpix + PixZoom * tile.maxLaserintile * ypix]
					       = 255.0 * (*(datavalues + ilasertile * PixZoomSquare + ipix) - Minmicroimages) /(Maxmicroimages - Minmicroimages);
			}
		}

		sdkSavePGM(DataFile, i_data,tile.maxLaserintile *PixZoom , tile.NbTileXYD * PixZoom);
	verbosefile << "HOST: " << stepnumber.c_str() << "  " <<  stepval << " ******************************************\n\n";
	}
	return (MaxData);
}

bool scratchdisplay (float * scratchdata, char * filename)
{
	unsigned char *i_scratchpad = (unsigned char *) calloc(tile.NbTileXY * XSCRATCH * YSCRATCH, sizeof(unsigned char)); // on host
	for (int iy = 0; iy < tile.NbTiley; iy++)
		for (int ix = 0; ix < tile.NbTilex; ix++)
			for (int iix = 0; iix < XTile; iix++)
				for (int iiy = 0; iiy < YTile; iiy++) {

					int iscratch = lostpixels + iix + dxSCRo2; 		// contribution of x in the 1D SCRATCH
					iscratch += ix * XSCRATCH; 					// contribution of previous tiles in x
					iscratch += (iiy + dySCRo2) * XSCRATCH * tile.NbTilex; 		// contribution of y in 1D SCRATCH
					iscratch += iy * YSCRATCH * XSCRATCH  * tile.NbTilex; 	// contribution of previous tiles in y

					int itile = iix;  // contribution of x in the TILE
					itile += ix * XTile; // contribution of previous tile in x
					itile += iiy * XTile * tile.NbTilex; // contribution of y in the TILE
					itile += iy * ATile * tile.NbTilex ; // contribution of previous tiles in y

					int iscratch2Dx = iix + dxSCRo2 + ix * XSCRATCH; 	// contribution of x in the 1D SCRATCH + contribution of previous tiles in x
					int iscratch2Dy = iiy + dySCRo2 + iy * YSCRATCH; 		// contribution of y in 1D SCRATCH +contribution of previous tiles in y
					int iscratch2D = iscratch2Dx + iscratch2Dy * XSCRATCH * tile.NbTilex;
					i_scratchpad[iscratch2D] = 255.0 * scratchdata[iscratch] / Maxscratch;
					if(!(i_scratchpad[iscratch2D] ==0) && VERBOSE){
					printf("SCRATCHPAD \u24FC itile %d, iscratch %d iscratch2Dx %d, iscratch2Dy %d iscratch2D %d\n",
							itile, iscratch, iscratch2Dx, iscratch2Dy, iscratch2D);
					printf("SCRATCHPAD \u24FC itile %d, i_scratchpad[iscratch2D] %d val_scratchpad[arg1D] %f\n",
							itile, i_scratchpad[iscratch2D], scratchdata[iscratch]);
					}
				}
return(TRUE);
}
