
#include <hip/hip_runtime.h>
#ifdef STARTDEVICE
	if(!ithreads && VERBOSELOOP) printf("TEST: block x %d y %d z %d distrib number %d itc %d itb %d\n",
			blockIdx.x, blockIdx.y, blockIdx.z, distrib_number, itc, itb);
		if (!iprint)  	// the condition is required to have it printed once

		if (!iprint) { 	// the condition is required to have it printed once
		printf("\n\u2460********************************** START *****************************\n");
		printf("DEVICE: \u2460****************PARAMETERS OF MEASUREMENT *******************\n");
		printf("DEVICE: \u2460 PARAMETERS  NThreads %d Npixel %d pZOOM %d, pPSF %d\n", NThreads, Npixel, pZOOM, pPSF);
		printf("DEVICE: \u2460 PARAMETERS dimBlock  x: %d y: %d z: %d   ...   ", blockDim.x, blockDim.y, blockDim.z);
		printf("dimGrid  x: %d y: %d z: %d\n", gridDim.x, gridDim.y, gridDim.z);
		printf("DEVICE: \u2460 PARAMETERS pPSF %d XDistrib %d YDistrib %d ADistrib %d\n", pPSF, XDistrib, YDistrib, ADistrib);
		printf("DEVICE: \u2460 PARAMETERS XSCRATCH %d YSCRATCH %d XTILE %d YTILE %d\n", XSCRATCH, YSCRATCH, XTile, YTile);
		printf("DEVICE: \u2460 PARAMETERS Number of pixels calculated in parallel %d Number of threads used"
				" %d loop on threads %d\n", NThreads, THREADSVAL, THreadsRatio);
		printf("DEVICE: \u2460  TILES: XSCRATCH %d, YSCRATCH %d  iprint %d", XSCRATCH, YSCRATCH,iprint);
		printf("XTILE %d, YTILE %d\n", XTile, YTile);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Aggregates in x: %d in y:%d\n", DD.NbAggregx,
				DD.NbAggregy);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Tiles per aggregates in x: %d in y:%d\n",
				DD.tileperaggregatex, DD.tileperaggregatey);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Tiles in x: %d in y:%d\n", DD.NbTilex, DD.NbTiley);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Max number of laser position in Tile: %d min value:%d Number of blocks %d\n",
				DD.maxLaserintile, DD.minLaserintile, DD.blocks);
		printf("\u2460*******************************PARAMETERS OF MEASUREMENT ***************\n");
	}
	__syncthreads();  // to be replaced for group synchronization of CUDA 9.0

	if (!iprint) timer = clock64();
	time_start = timer; time_init = timer;
	if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %g this step  %g  total %g \n\n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);

	#endif
