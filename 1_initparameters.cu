#include "hip/hip_runtime.h"
/*
 * initparameters.cu
 *
 *  Created on: May 7, 2017
 *      Author: gabriel
 */
/*
 *  Created on: April 18, 2017
 *      Author: gabriel
 */

#include "0_Mainparameters.h"

/*******************PARAMETERS**************/
char buff[BUFFSIZE]; // a buffer to temporarily park the data
double Timestep[16];
char chars[] = "[]()", delimeter('=');
__managed__ int pPSF, Npixel, RDISTRIB, pZOOM, Ndistrib;

__managed__ double  Energy_global =0.0f;
__managed__ clock_t timer, time_init, time_start; // in KHz

std::string resourcesdirectory, filename, name, value, MIFILE, PSFFILE, DISTRIBFILE;

__managed__ int XTile, YTile, ATile;
__managed__ int THreadsRatio, NThreads;
__managed__ int XDistrib, YDistrib, YDistrib_extended, lostlines, ADistrib;

hipDeviceProp_t deviceProps;

bool initparameters( int argc, char **argv) {
	XMLDocument XMLdoc, ACQXML, doc;
	XMLElement *pRoot, *pParm;
	string sstr, filenamexml;
	bool dimfit = TRUE;

	// acquire information on the CUDA device: name and number of multiprocessors
	devID = gpuDeviceInit(devID);
	verbosefile << "MAIN PROGRAM  \u24EA NewLoop start ...\n";
	if (devID < 0) {
		printf("exiting...\n");
		exit(EXIT_FAILURE);
	}

	/********************NVIDIA Card assesment and parameters ************************/
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID)); 	// get device name & properties
	TA.MP = deviceProps.multiProcessorCount;
	TA.sharedmemory = deviceProps.sharedMemPerBlock;
	clockRate = deviceProps.clockRate;
	verbosefile << " INIT PROG \u24EA Number of Multiprocessors (MP) " << TA.MP << " clock rate (KHz) " << clockRate
			<< " SharedMemory " << (float) TA.sharedmemory/1024. << " in KBytes\n\n";

	/***************************** command line management*****************************/
	resourcesdirectory = argv[1]; 	// Directory with all preprocessing files and data
	pPSF = retrieveargv(argv[2]);	// PSF Size, without zoom
	Npixel = retrieveargv(argv[3]);	// Pixel number without zoom
	RDISTRIB = retrieveargv(argv[4]);
	pZOOM = retrieveargv(argv[5]);
	Ndistrib = retrieveargv(argv[6]);
	MIFILE = argv[7];
	// to read the values in the program and to add tests

	verbosefile << "MAIN PROGRAM  \u24EA ARG: EXE arguments number argc: " << argc << endl;
	verbosefile << "MAIN PROGRAM  \u24EA command line parameters: " << "******************************************"
			<< endl;
	verbosefile << "MAIN PROGRAM  \u24EA ARG: argv[1]: working directory: " << resourcesdirectory << endl;
	verbosefile << "MAIN PROGRAM  \u24EA ARG: argv[2]: pPSF: " << pPSF << endl;
	verbosefile << "MAIN PROGRAM  \u24EA ARG: argv[3]: Npixel: " << Npixel << endl;
	verbosefile << "MAIN PROGRAM  \u24EA ARG: argv[4]: RDISTRIB: " << RDISTRIB << endl;
	verbosefile << "MAIN PROGRAM  \u24EA ARG: argv[5]: pZOOM: " << pZOOM << endl;
	verbosefile << "MAIN PROGRAM  \u24EA ARG: argv[6]: NDISTRIB: " << Ndistrib << endl;
	verbosefile << "MAIN PROGRAM  \u24EA ARG: argv[7]: MI directory: " << resourcesdirectory + MIFILE << endl;
	verbosefile << "MAIN PROGRAM  \u24EA command line parameters: " << "******************************************"
			<< endl <<endl;

	/** initialize the general parameters and the offset & scale parameters
	 */
	TA.start();
	OFSCAL.start();

	/** Threads parameters
		 *
		 */
		THreadsRatio = CEILING_POS((1.0*PixZoomSquare)/THREADSVAL);
		NThreads = THreadsRatio * THREADSVAL;

		/** distrib parameters: cover all possible values of the distribution up
		 *  to the corner of the microimage included dummy pixels incremented by the pPSF size -1 to avoid counting twice the pixel
		 */
		XDistrib = (PixZoomo2+PSFZoomo2)*2+1;
		YDistrib = (PixZoomo2+PSFZoomo2)*2+1;
		YDistrib_extended = NThreads/PixZoom+PSFZoom;
		lostlines = NThreads/PixZoom - PixZoom +1;

		float tempe = XDistrib * YDistrib_extended;
		ADistrib = CEILING_POS(tempe/THREADSVAL)*THREADSVAL;


	/** SCRATCH and TILE parameters
	 *
	 */
	XTile = XSCRATCH - dxSCR;	// we do not care on x because the distribution will be zero
								// We add lostpixels  at start and end of the scratchpad
								// for "spillover" of the first and last line
	YTile = YSCRATCH - dySCR - lostlines; 	// in y we need the full size
	if((YTile%2)==0) YTile--;				// We insure that YTile is odd
	ATile = XTile * YTile;										// Total size in pixels


	verbosefile << "************** DATA: PARAMETERS OF MEASUREMENT *************************************";
	verbosefile << " INIT PROG \u24EA BASIC  : THreadsRatio "<<  THreadsRatio << " NThreads " << NThreads;
	verbosefile << " Npixel " << Npixel <<" pZOOM " << pZOOM << " pPSF " << pPSF << " RDISTRIB " << RDISTRIB << endl;
	verbosefile << " INIT PROG \u24EA BASIC  : YTile " << YTile << " YSCRATCH " << YSCRATCH << " dySCR " << dySCR << endl;
	verbosefile << "INIT PROG \u24EA PIXEL  : Npixel " << Npixel << " PixZoom " << PixZoom << " PixZoomo2 " << PixZoomo2 << endl;
	verbosefile << "INIT PROG \u24EA PIXEL  : lost lines " << lostlines << " additional lines at the end of microimage\n" << endl;
	verbosefile << "INIT PROG \u24EA pPSF   : pPSF " << pPSF << " PSFZoom " << PSFZoom << " PSFZoomo2 " << PSFZoomo2 << endl;
	verbosefile << " INIT PROG \u24EA DISTRIB: XDistrib " << XDistrib << " YDistrib " << YDistrib <<  "extended "
			 << YDistrib_extended << " Size in KBytes " << ADistrib/1024. <<
			 "ADistrib " << ADistrib << " RDISTRIB " << RDISTRIB << endl;
	verbosefile << " INIT PROG \u24EA SCRATCH X&Y: " << XSCRATCH << " " << YSCRATCH << " dxSCR "
			<< dxSCR << " dySCR " << dySCR << endl;
	verbosefile << " INIT PROG \u24EA SCRATCH: DEL SCRATCH " << lostpixels << " Additional pixels at start and end of SCRATCH\n";
	verbosefile << " INIT PROG \u24EA PARAMS :  Number of threads " << NThreads << " Threads per batch "
			<< THREADSVAL <<" number of batch "  << THreadsRatio << endl;
	verbosefile << "************** DATA: PARAMETERS OF MEASUREMENT *************************************\n\n";

	verbosefile << " INIT PROG \u23f3 Data parameters in device memory ...\n";


	/********************************Reconstruction parameters *************************/
	filenamexml = resourcesdirectory + "reconstruction.xml";
	verbosefile << " INIT PROG \u24EA reconstruction xml: " << filenamexml.c_str() << endl;
	doc.LoadFile(filenamexml.c_str());

	TA.Nb_Rows_reconstruction = atoi(doc.FirstChildElement("Image_Contents")
			->FirstChildElement("Nb_Rows")->GetText());
	TA.Nb_Cols_reconstruction = atoi(doc.FirstChildElement("Image_Contents")
			->FirstChildElement("Nb_Cols")->GetText());
	TA.reconstruction_size = TA.Nb_Cols_reconstruction*TA.Nb_Rows_reconstruction;
	verbosefile << " INIT PROG \u24EA reconstruction from tiles: Cols " << TA.Nb_Cols_reconstruction;
	verbosefile << " size " << TA.Nb_Rows_reconstruction << " size " << TA.reconstruction_size;

	/***********************Sizes in nm *************************************************/
	filename = resourcesdirectory + "ACQ.xml";
	int LoadACQOK = XMLError(ACQXML.LoadFile(filename.c_str()));
	pRoot = ACQXML.FirstChildElement("BioAxialAcquisitionRequest");
	pParm = pRoot->FirstChildElement("Camera_parameters")->FirstChildElement("PixelSize_nm");
	sstr = pParm->GetText();
	for (unsigned int i = 0; i < strlen(chars); ++i)
		sstr.erase(std::remove(sstr.begin(), sstr.end(), chars[i]), sstr.end());
	stringstream stream_p(sstr);
	stream_p.getline(buff, 10, ',');
	TA.Pixel_size_nm = atoi(buff);
	verbosefile << " INIT PROG \u24EA PARAMS :  original µimage pixel size" << TA.Pixel_size_nm;
	verbosefile << " nm pixel size reconstruction ",TA.Pixel_size_nm/pZOOM;
	TA.XTileSize = (XTile * TA.Pixel_size_nm)/(1000.*pZOOM); 	// Tile size in nm
	TA.YTileSize = (YTile * TA.Pixel_size_nm)/(1000.*pZOOM);	// Tile size in nm
	verbosefile << " INIT PROG \u24EA TILE   : XTILE " << XTile << " YTILE " << YTile
			<< " size : XTILE: " << TA.XTileSize << " YTILE " << TA.YTileSize;
	verbosefile << " INIT PROG \u24EA RECONSTRUCTION in nm   : X " <<  TA.Nb_Cols_reconstruction*TA.Pixel_size_nm/1000.;
	verbosefile << " Y " << TA.Nb_Rows_reconstruction*TA.Pixel_size_nm/1000. << " µm\n";

	return (dimfit);
}

