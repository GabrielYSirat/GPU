
#include <hip/hip_runtime.h>
#ifdef TESTSCRATCH
DD.step++;
float Sumscratchval=0.0f, Maxscratchval=0.0f;

	for (int jscratch = 0; jscratch < ASCRATCH; jscratch ++) {
		val2_scratchpad[jscratch + scrglobal - scratchpad_matrix] = Scratchpad[jscratch]; // scratchpad image validation
		int delta = jscratch + scrglobal - scratchpad_matrix - lostpixels;
//		int x = delta%XSCRATCH; int y = delta/YSCRATCH;
		Sumscratchval += val2_scratchpad[jscratch + scrglobal - scratchpad_matrix];
		Maxscratchval = max(Scratchpad[jscratch], Maxscratchval);
		if(*(Scratchpad + jscratch) != 0.0f && !ithreads)
			printf("DEVICE: \u2463 SCRATCHPAD distrib_number %d itb %d delta %d position (x*y) (%d*%d) position in scratchpad %d value %f Sum %f max %f\n",
					distrib_number, itb, delta, (delta-DD.lostpixelsdevice)%(XSCRATCH*tilexdevice), (delta - DD.lostpixelsdevice)/(XSCRATCH*tilexdevice), jscratch, *(Scratchpad + jscratch), Sumscratchval, Maxscratchval);
	}
	__syncthreads();

	if (!iprint) printf("end \u2463**********DEVICE:  SCRATCHPAD *aggregate  (%d*%d) *****************\n\n", aggregx, aggregy);
__syncthreads();

if(((aggregx+1) == DD.NbAggregx) && ((aggregy+1) == DD.NbAggregy)) {
	if (!iprint) printf("DEVICE: \u2464 SUM SCRATCHPAD: Sum of scratchpad %5.1f Max of Scratchpad %5.1f \n", Sumscratchval, Maxscratchval);
	if (!iprint) timer = clock64();

	if (!iprint)
	printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) ( time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	if (!iprint) printf("end \u2464****************DEVICE:  SCRATCHPAD & AGGREGATES & TILES ********************\n\n");
}
__syncthreads();

#endif
