
#include <hip/hip_runtime.h>
#ifdef TESTSCRATCH
DD.step++;
float Sumscratchval=0.0f, Maxscratchval=0.0f;

	for (int jscratch = 0; jscratch < ASCRATCH; jscratch ++) {
		val2_scratchpad[jscratch + MemoryOffsetscratch] = Scratchpad[jscratch]; // scratchpad image validation
		Sumscratchval += val2_scratchpad[jscratch + MemoryOffsetscratch];
		Maxscratchval = max(Scratchpad[jscratch], Maxscratchval);
		if(*(Scratchpad + jscratch) != 0.0f && !ithreads)
			printf("DEVICE: \u2463 SCRATCHPAD distrib_number %d itb %d position in scratchpad %d value %f Sum %f max %f\n",
					distrib_number, itb, jscratch, *(Scratchpad + jscratch), Sumscratchval, Maxscratchval);
	}
	__syncthreads();
	if (!iprint) printf("end \u2463****************DEVICE:  SCRATCHPAD ********************\n\n");
__syncthreads();

if(((aggregx+1) == DD.NbAggregx) && ((aggregy+1) == DD.NbAggregy)) {
	if (!iprint) printf("DEVICE: \u2464 SUM SCRATCHPAD: Sum of scratchpad %5.1f Max of Scratchpad %5.1f \n", Sumscratchval, Maxscratchval);
	if (!iprint) timer = clock64();
	if (!iprint)
	printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) ( time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	if (!iprint) printf("end \u2464****************DEVICE:  SCRATCHPAD & AGGREGATES & TILES ********************\n\n");
}
__syncthreads();

#endif
