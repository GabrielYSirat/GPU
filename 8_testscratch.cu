
#include <hip/hip_runtime.h>
#ifdef TESTSCRATCH
DD.step++;
float Sumscratchval, Maxscratchval;
if(!ithreads)
for (int jscratch = 0; jscratch < ASCRATCH; jscratch ++) {
	val2_scratchpad[jscratch + MemoryOffsetscratch] = Scratchpad[jscratch]; // scratchpad image validation
	Sumscratchval += val2_scratchpad[jscratch + MemoryOffsetscratch];
	Maxscratchval = max(Scratchpad[jscratch], Maxscratchval);
	if(*(Scratchpad + jscratch) != 0.0f && !ithreads) printf("DEVICE: \u2464 SCRATCHPAD ithreads %d itb %d position in scratchpad %d value %f Sum %f max %f\n",
			ithreads, itc, jscratch, *(Scratchpad + jscratch), Sumscratchval, Maxscratchval);
}
if(((aggregx+1) == DD.NbAggregx) && ((aggregy+1) == DD.NbAggregy)) {
	if (!iprint) printf("DEVICE: \u2464 SUM SCRATCHPAD: Sum of scratchpad %5.1f Max of Scratchpad %5.1f \n", Sumscratchval, Maxscratchval);
	if (!iprint) timer = clock64();
	if (!iprint)
	printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) ( time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	if (!iprint) printf("end \u2464****************DEVICE:  SCRATCHPAD & AGGREGATES & TILES ********************\n\n");
}
__syncthreads();

#endif
