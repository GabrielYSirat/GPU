
#include <hip/hip_runtime.h>
#ifdef TESTSCRATCH
DD.step++;
if(!ithreads)
for (int jscratch = 0; jscratch < ASCRATCH; jscratch ++) {
	val2_scratchpad[jscratch + MemoryOffsetscratch + distrib_number * ASCRATCH * DD.NbTileXY] = Scratchpad[jscratch]; // scratchpad image validation
	Sumscratch += val2_scratchpad[jscratch + MemoryOffsetscratch + distrib_number * ASCRATCH * DD.NbTileXY];
	maxscratch = max(Scratchpad[jscratch], maxscratch);
	if(*(Scratchpad + jscratch) != 0.0f && !ithreads) printf("DEVICE: \u2464 SCRATCHPAD ithreads %d itb %d position in scratchpad %d value %f Sum %f max %f\n",
			ithreads, itc, jscratch, *(Scratchpad + jscratch), Sumscratch, maxscratch);
}
if(((aggregx+1) == DD.NbAggregx) && ((aggregy+1) == DD.NbAggregy)) {
	if (!iprint) printf("DEVICE: \u2464 SUM SCRATCHPAD: Sum of scratchpad %5.1f Max of Scratchpad %5.1f \n", Sumscratch, maxscratch);
	if (!iprint) timer = clock64();
	if (!iprint)
	printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) ( time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	if (!iprint) printf("end \u2464****************DEVICE:  SCRATCHPAD & AGGREGATES & TILES ********************\n\n");
}
__syncthreads();

#endif