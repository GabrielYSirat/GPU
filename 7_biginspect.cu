#include "hip/hip_runtime.h"
/*
 * biginspect.cu
 *
 *  Created on: Sep 16, 2017
 *      Author: gabriel
 */

#include "0_Mainparameters.h"
const char * ScratchpadVal2Imagefile = "results/E_Scratchpadloop.pgm";
const char * DistribPosImage = "results/B_DistributionsLoopintern.pgm";
const char * PSFLoopImage = "results/A_PSFloop.pgm";
const char * DistribTestImage = "results/B_DistributionsLoop.pgm";

bool biginspect(int stepval) {
	bool boolinspect;
	float MaxPSFLoop = 0.0f, MaxDistribvalid = 0.0f;



	onhost.MaxSimus = displaydata(new_simus, stepval); 	stepval++;

	onhost.MaxRfactor = displaydata(Rfactor, stepval); 	stepval++;

	verbosefile << "HOST: \u24F3 *************************BigLoop terminated ***************************" << endl;
	verbosefile << "HOST: \u24F3 ***********************************************************************" << endl << endl;

	scratchreaddisplay(val2_scratchpad, val2_scratchpad, ScratchpadVal2Imagefile, FALSE);

	boolinspect = ((onhost.MaxSimus == 0.0f) && (onhost.MaxRfactor == 0.0f));

	unsigned char *i_test2PSF = (unsigned char *) calloc(PSFZOOMSQUARE, sizeof(unsigned char)); // on host
	unsigned char *i_distribpos = (unsigned char *) calloc(TA.MP * PSFZOOMSQUARE, sizeof(unsigned char)); // on host
	unsigned char *j_distribpos = (unsigned char *) calloc(XDistrib * TA.MP * YDistrib_extended,
			sizeof(unsigned char)); // on host
	verbosefile << "SCRATCHPAD \u24EC Path to DistribPos validation ....." << DistribPosImage << endl;

	for (int i = 0; i < PSFZOOMSQUARE; i++)
		MaxPSFLoop = max(MaxPSFLoop, test2_psf[i]);
	for (int i = 0; i < PSFZOOMSQUARE; i++)
		i_test2PSF[i] = 255.0 * test2_psf[i] / MaxPSFLoop;			// Validation image value
	verbosefile << "Max PSF Loop " << MaxPSFLoop << endl;
	sdkSavePGM(PSFLoopImage, i_test2PSF, PSFZoom, PSFZoom);

	for (int i = 0; i < TA.MP * PSFZOOMSQUARE; i++)
		MaxDistribvalid = max(MaxDistribvalid, distribvalidGPU[i]);
	for (int i = 0; i < TA.MP * PSFZOOMSQUARE; i++)
		i_distribpos[i] = 255.0 * distribvalidGPU[i] / Maxdistrib;			// Validation image value
	verbosefile << "Max Distrib Validation " << MaxDistribvalid << endl;
	sdkSavePGM(DistribPosImage, i_distribpos, PSFZoom, TA.MP * PSFZoom);

	float MaxDistribtest = 0.0f;
	for (int i = 0; i < TA.MP * ADistrib; i++)
		MaxDistribtest = max(MaxDistribtest, test2_distrib[i]);
	verbosefile << "Max Distrib Test " << MaxDistribtest << endl;
	for (int i = 0; i < TA.MP * ADistrib; i++) {
		int tempa = i % ADistrib; int tempb = i / ADistrib;
		if (tempa < XDistrib * YDistrib_extended)
			j_distribpos[tempa + tempb * XDistrib * YDistrib_extended] = 255.0 * test2_distrib[i] / Maxdistrib;			// Validation image value
	}
	sdkSavePGM(DistribTestImage, j_distribpos, XDistrib, TA.MP * YDistrib_extended);

	return (boolinspect);
}

