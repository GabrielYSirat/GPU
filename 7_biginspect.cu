#include "hip/hip_runtime.h"
/*
 * biginspect.cu
 *
 *  Created on: Sep 16, 2017
 *      Author: gabriel
 */

#include "0_Mainparameters.h"
const char * ScratchpadVal2Imagefile = "results/E_Scratchpadloop.pgm";
const char * DistribPosImage = "results/B_DistributionsLoopintern.pgm";
const char * PSFLoopImage = "results/A_PSFloop.pgm";
const char * DistribTestImage = "results/B_DistributionsTest.pgm";

bool biginspect(int stepval) {
	bool boolinspect;
	float MaxPSFLoop = 0.0f, MaxDistribvalid = 0.0f, SumDistribvalid = 0.0f;



	onhost.MaxSimus = displaydata(new_simus, stepval); 	stepval++;

	onhost.MaxRfactor = displaydata(Rfactor, stepval); 	stepval++;

	cout << "HOST: \u24F3 *************************BigLoop terminated ***************************" << endl;
	cout << "HOST: \u24F3 ***********************************************************************" << endl << endl;

	scratchreaddisplay(val2_scratchpad, val2_scratchpad, ScratchpadVal2Imagefile, FALSE);

	boolinspect = ((onhost.MaxSimus == 0.0f) && (onhost.MaxRfactor == 0.0f));

	unsigned char *i_test2PSF = (unsigned char *) calloc(PSFZOOMSQUARE, sizeof(unsigned char));
	unsigned char *i_distribpos = (unsigned char *) calloc(TA.MP * PSFZOOMSQUARE, sizeof(unsigned char));
	unsigned char *j_distribpos = (unsigned char *) calloc(XDistrib * TA.MP * YDistrib_extended, sizeof(unsigned char));
	verbosefile << "SCRATCHPAD \u24EC Path to DistribPos validation ....." << DistribPosImage << endl;

	for (int i = 0; i < PSFZOOMSQUARE; i++)
		MaxPSFLoop = max(MaxPSFLoop, test2_psf[i]);
	for (int i = 0; i < PSFZOOMSQUARE; i++)
		i_test2PSF[i] = 255.0 * test2_psf[i] / MaxPSFLoop;			// Validation image value
	verbosefile << "Max PSF Loop " << MaxPSFLoop << endl;
	sdkSavePGM(PSFLoopImage, i_test2PSF, PSFZoom, PSFZoom);

	for (int i = 0; i < TA.MP * PSFZOOMSQUARE; i++){
		MaxDistribvalid = max(MaxDistribvalid, distribvalidGPU[i]);
		SumDistribvalid += distribvalidGPU[i];
	}
	printf("MaxDistribvalid %f SumDistribvalid %f\n", MaxDistribvalid, SumDistribvalid);
	for (int i = 0; i < TA.MP * PSFZOOMSQUARE; i++){
//		if(i < PSFZOOMSQUARE) printf("i %d value %f\n", i, distribvalidGPU[i]);
		i_distribpos[i] = 255.0 * distribvalidGPU[i] / Maxdistrib;			// Validation image value
	}
	verbosefile << "Max Distrib Validation " << MaxDistribvalid << endl;
	sdkSavePGM(DistribPosImage, i_distribpos, PSFZoom, TA.MP * PSFZoom);

	float MaxDistribtest = 0.0f;
	for (int i = 0; i < TA.MP * ADistrib; i++)
		MaxDistribtest = max(MaxDistribtest, test2_distrib[i]);
	float ratio = 255./MaxDistribtest;
	verbosefile << "Max Distrib Test " << MaxDistribtest << " ratio " << ratio << endl;
	for (int i = 0; i < TA.MP * ADistrib; i++) {
		int tempa = i % ADistrib; int tempb = i / ADistrib;
		if (tempa < XDistrib * YDistrib_extended)
			j_distribpos[tempa + tempb * XDistrib * YDistrib_extended] = ratio * test2_distrib[i];// Validation image value
	}
	sdkSavePGM(DistribTestImage, j_distribpos, XDistrib, TA.MP * YDistrib_extended);

	return (boolinspect);
}

