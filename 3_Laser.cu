#include "hip/hip_runtime.h"
/*
 * readLaserPositions.cu
 *
 *  Created on: Apr 18, 2017
 *      Author: gabriel
 */
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include "0_Mainparameters.h"
ifstream LaserFile;
std::string LaserFILE = "lambda_488/Measure/T_0/Z_0/laser_positions_";
std::string endlaser = ".txt";

void readstoreLaserPositions(void) {
	float laserval;
	bool XY = FALSE;

	TA.Nb_LaserPositions = 0;
	tile.maxlaserperdistribution = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
	{
		filename = resourcesdirectory + LaserFILE + std::to_string(idistrib + 1) + endlaser;
		LaserFile.open(filename);
		if (!LaserFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}

		tile.Nblaserperdistribution[idistrib] = 0;
		while (LaserFile >> laserval) {
			if (XY) {
				TA.Nb_LaserPositions++; tile.Nblaserperdistribution[idistrib]++; }
			XY = !XY;
		} // adding one at the end because it is a number of positions

		tile.maxlaserperdistribution = max(tile.maxlaserperdistribution, tile.Nblaserperdistribution[idistrib]);
		printf(" Laser \u2462: distribution n°%d number of images %d   ", idistrib, tile.Nblaserperdistribution[idistrib]);

		LaserFile.close();
	}
	printf(" total number of images %d max images per distributions %d \n",  TA.Nb_LaserPositions, tile.maxlaserperdistribution);

	hipMallocManaged(&PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosxScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&PosyScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&offsetFULL, TA.Nb_LaserPositions * sizeof(int));

	int ilaserpos = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		filename = resourcesdirectory + LaserFILE + std::to_string(idistrib + 1) + endlaser;
		printf(" Laser \u2462: filename %s \n", filename.c_str());
		LaserFile.open(filename);
		if (!LaserFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}
		XY = FALSE;
		// introduce here scale and offset relative to camera origin, if needed
		while (LaserFile >> laserval) {
			if (!XY) {
			*(PosLaserx + ilaserpos) = (laserval + OFSCAL.offsetLaserx)* OFSCAL.scaleLaserx;
			TA.maxLaserx = max(TA.maxLaserx, *(PosLaserx + ilaserpos));
			TA.minLaserx = min(TA.minLaserx, *(PosLaserx + ilaserpos));
			// Laser positions in x zoomed integer in the 2D scratchpad
			}
			else {
				*(PosLasery + ilaserpos) = (laserval + OFSCAL.offsetLasery) * OFSCAL.scaleLasery;
				TA.maxLasery = max(TA.maxLasery, *(PosLasery + ilaserpos));
				TA.minLasery = min(TA.minLasery, *(PosLasery + ilaserpos));
				ilaserpos++;
			}
			XY = !XY;
		}
		LaserFile.close();
	}

	printf(" Laser \u2462 min and max x %g %g, min and max y %g %g ... \n",
			TA.maxLaserx, TA.minLaserx, TA.maxLasery, TA.minLasery);
}

bool validateLaserPositions_control(void) {

	double Delx { 0.0 }, Dely { 0.0 };
	bool testLaserPosition = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateLaserPositions_device<<<dimGrid, dimBlock, 0>>> (TA.Nb_LaserPositions);
	hipDeviceSynchronize();

	if (TA.Nb_LaserPositions < smallnumber)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			if(!ival && VERBOSE)
				verbosefile << " Laser \u2462 ----------------------------------------------------------------------------------------------------\n";
			verbosefile << " Laser \u2462 Laser position %d  Laser position x: %f  y: %f\n";
			verbosefile << ival << "  " << *(PosLaserx + ival) << "  " << *(PosLasery + ival) << endl;
			verbosefile << "Laser \u2462 Position in scratchpad",
			verbosefile << *(PosxScratch + ival) << "  " << *(PosyScratch + ival) << endl;
			verbosefile << " Laser \u2462 ***************SCRATCHPAD FULL OFFSET ";
			verbosefile << *(offsetFULL + ival) << " **************\n";
			verbosefile << " Laser \u2462 ----------------------------------------------------------------------------------------------------\n";
		}
	if (VERBOSE) printf(" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
	for (int iLaser = 0; iLaser < TA.Nb_LaserPositions; iLaser++) {
		Delx += PosLaserx[iLaser] - d_PosLaserx[iLaser];
		Dely += PosLasery[iLaser] - d_PosLasery[iLaser];
	}
	Sumdel[2] = sqrt(Delx * Delx + Dely * Dely);
	printf(" Laser \u2462 delx %8.6f dely %8.6f Sumdel[2] %8.6f \n", Delx, Dely, Sumdel[2]);
	if (Delx * Dely == 0.0f) testLaserPosition = TRUE;

	return (testLaserPosition);
}


