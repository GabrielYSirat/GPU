#include "hip/hip_runtime.h"
/*
 * readLaserPositions.cu
 *
 *  Created on: Apr 18, 2017
 *      Author: gabriel
 */
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include "0_Mainparameters.h"
ifstream LaserFile;
std::string LaserFILE = "lambda_488/Measure/T_0/Z_0/laser_positions_";
std::string endlaser = ".txt";

void readstoreLaserPositions(void) {
	float laserval;
	bool XY = FALSE;

	TA.Nb_LaserPositions = 0;
	tile.maxlaserperdistribution = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
	{
		filename = resourcesdirectory + LaserFILE + std::to_string(idistrib + 1) + endlaser;
		LaserFile.open(filename);
		if (!LaserFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}

		tile.Nblaserperdistribution[idistrib] = 0;
		while (LaserFile >> laserval) {
			if (XY) {
				TA.Nb_LaserPositions++; tile.Nblaserperdistribution[idistrib]++; }
			XY = !XY;
		} // adding one at the end because it is a number of positions

		tile.maxlaserperdistribution = max(tile.maxlaserperdistribution, tile.Nblaserperdistribution[idistrib]);
		verbosefile << " Laser \u2462: distribution n°" << idistrib << " number of images " << tile.Nblaserperdistribution[idistrib] << endl;

		LaserFile.close();
	}
	verbosefile << " Laser \u2462:  total number of images "<< TA.Nb_LaserPositions << " max images per distributions "
			<< tile.maxlaserperdistribution << endl;

	hipMallocManaged(&PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLaserx, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&d_PosLasery, TA.Nb_LaserPositions * sizeof(float));
	hipMallocManaged(&PosxScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&PosyScratch, TA.Nb_LaserPositions * sizeof(int));
	hipMallocManaged(&offsetFULL, TA.Nb_LaserPositions * sizeof(int));

	int ilaserpos = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		filename = resourcesdirectory + LaserFILE + std::to_string(idistrib + 1) + endlaser;
		verbosefile << "Laser \u2462: filename " << filename.c_str() << " \n";
		LaserFile.open(filename);
		if (!LaserFile) {
			printf("unable to open filename %s\n\n", filename.c_str());
			exit(1);   // call system to stop
		}
		XY = FALSE;
		// introduce here scale and offset relative to camera origin, if needed
		while (LaserFile >> laserval) {
			if (!XY) {
			*(PosLaserx + ilaserpos) = (laserval + OFSCAL.offsetLaserx)* OFSCAL.scaleLaserx;
			TA.maxLaserx = max(TA.maxLaserx, *(PosLaserx + ilaserpos));
			TA.minLaserx = min(TA.minLaserx, *(PosLaserx + ilaserpos));
			// Laser positions in x zoomed integer in the 2D scratchpad
			}
			else {
				*(PosLasery + ilaserpos) = (laserval + OFSCAL.offsetLasery) * OFSCAL.scaleLasery;
				TA.maxLasery = max(TA.maxLasery, *(PosLasery + ilaserpos));
				TA.minLasery = min(TA.minLasery, *(PosLasery + ilaserpos));
				ilaserpos++;
			}
			XY = !XY;
		}
		LaserFile.close();
	}

	printf("\n Laser \u2462 HOST : min and max x %g %g, min and max y %g %g ... \n",
			TA.maxLaserx, TA.minLaserx, TA.maxLasery, TA.minLasery);
}

bool validateLaserPositions_control(void) {

	double Delx { 0.0 }, Dely { 0.0 };
	bool testLaserPosition = FALSE;
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Laser positions kernel
	validateLaserPositions_device<<<dimGrid, dimBlock, 0>>> (TA.Nb_LaserPositions);
	hipDeviceSynchronize();

	if (TA.Nb_LaserPositions < smallnumber)
		for (int ival = 0; ival < TA.Nb_LaserPositions; ival++) {
			if(!ival && VERBOSE)
				verbosefile << " Laser \u2462 ----------------------------------------------------------------------------------------------------\n";
			verbosefile << " Laser \u2462 Laser position n°" << ival << " x " << *(PosLaserx + ival)
					<< " y " << *(PosLasery + ival) << endl;
			verbosefile << "Laser \u2462 Position in scratchpad",
			verbosefile << *(PosxScratch + ival) << "  " << *(PosyScratch + ival) << endl;
			verbosefile << " Laser \u2462 ***************SCRATCHPAD FULL OFFSET ";
			verbosefile << *(offsetFULL + ival) << " **************\n";
			verbosefile << " Laser \u2462 ----------------------------------------------------------------------------------------------------\n";
		}
	if (VERBOSE) printf(" Laser \u2462 ----------------------------------------------------------------------------------------------------\n");
	for (int iLaser = 0; iLaser < TA.Nb_LaserPositions; iLaser++) {
		Delx += PosLaserx[iLaser] - d_PosLaserx[iLaser];
		Dely += PosLasery[iLaser] - d_PosLasery[iLaser];
	}
	Sumdel[2] = sqrt(Delx * Delx + Dely * Dely);
	verbosefile << " Laser \u2462 delx " << Delx << " dely " << Dely << " Sumdel[2] "<<  Sumdel[2] << endl;
	if (Delx * Dely == 0.0f) testLaserPosition = TRUE;

	return (testLaserPosition);
}


