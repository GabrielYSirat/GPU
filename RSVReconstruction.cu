#include "hip/hip_runtime.h"
/*
 * RSVReconstruction.cu
 *
 *  Created on: Jul 8, 2017
 *      Author: gabriel
 */
#include "0_NewLoop.h"
std::string filenameimage;
std::string RECFILE = "image_iteration_0__63x114_4em";
std::string endREC = ".raw";



void Recprepare(void) {
	float MaxRec = 0.0f, SumRec = 0.0f;
	hipMallocManaged(&original_rec, TA.reconstruction_size * sizeof(float)); // on device with a shadow on host
	hipMallocManaged(&double_rec, TA.reconstruction_size * sizeof(double)); // on device with a shadow on host

	char * memblock;
	int size;

	filenameimage = resourcesdirectory + RECFILE + endREC;
	printf("REC \u24FC reconstruction image:  %s \n", filenameimage.c_str());

	/** *****************************data arrays allocation*********************************/
	/** original_reconstruction data in float stored on device with a shadow copy on host
	 *  double_rec data in double read from the file
	 *  i_reconstruction normalized data in char on host for image display
	 */


	//read reconstruction raw file
	std::ifstream RecFile(filenameimage.c_str(), ios::in | ios::binary | ios::ate);
	size = (RecFile.tellg()); 	// the data is stored in float of 4 bytes in the file
	size -= byte_skipped; 	// WE REMOVE byte_skipped BYTES
	std::cout << "REC \u24FC ************file read: size reconstruction in bytes = " << size << endl;
	memblock = new char[size];
	RecFile.seekg(byte_skipped, ios::beg); // byte_skipped first bytes are skipped
	RecFile.read(memblock, size);
	RecFile.close();
	std::cout << "REC \u24FC *******complete size  " << TA.reconstruction_size << "  Size in Bytes " << TA.reconstruction_size * sizeof(double) << endl;

	/** read reconstruction data from file in double, transfer to float on the global memory of the device and create a normalized image
	 *
	 */
	double_rec = (double*) memblock; //reinterpret the chars stored in the file as float
	for (int i = 0; i < TA.reconstruction_size; i++) {
		*(original_rec + i) = *(double_rec + i);
		SumRec += *(original_rec + i);
		MaxRec = max(*(original_rec + i), MaxRec);
	}	// sanity check, check max and sum


	std::cout << "REC \u24FC ***  max =" << MaxRec << "  Sum =" << SumRec << endl;
	const char * reconstructionImagefile = "results/reconstruction.pgm";
	unsigned char *i_reconstruction = (unsigned char *) calloc(TA.reconstruction_size, sizeof(unsigned char)); // on host
	double* double_rec = (double*) std::malloc(TA.reconstruction_size * sizeof(double)); // on host
	// write reconstruction image to disk /////////////////////////////////
	for (int i = 0; i < TA.reconstruction_size; i++) i_reconstruction[i] = 255.0 * original_rec[i] / MaxRec;			// image value

	printf("REC \u24FC Path to reconstruction original %s .....\n", reconstructionImagefile);
	sdkSavePGM(reconstructionImagefile, i_reconstruction, TA.Nb_Cols_reconstruction, TA.Nb_Rows_reconstruction);
	free(i_reconstruction);
	free(double_rec);
}

bool Recvalidate_host(void) {
	bool testrec;
	float MaXTile = 0.0f, Sum3rec = 0.0f, max3rec = 0.0f;

	// write rec in memory and validate
	unsigned char *i_rec = (unsigned char *) calloc(TA.reconstruction_size, sizeof(unsigned char)); // on host
	hipMallocManaged(&val_rec, TA.reconstruction_size * sizeof(float));
	const char * recValImagefile = "results/recValImage.pgm";

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the rec kernel
	Recvalidate_device<<<dimGrid, dimBlock, 0>>>(TA.Nb_Rows_reconstruction, TA.Nb_Cols_reconstruction);
	hipDeviceSynchronize();

	for (int row = 0; row < TA.Nb_Rows_reconstruction; row++)
		for (int col = 0; col < TA.Nb_Cols_reconstruction; col++) {
			int tempr = row * TA.Nb_Cols_reconstruction + col;
			Sum3rec += *(val_rec + tempr);
			max3rec = max(max3rec, *(val_rec + tempr));
		}
	printf("on host: Sum3rec  %f max3rec %f   ", Sum3rec, max3rec);

	// write rec image validation to disk
	/////////////////////////////////
	MaXTile = 0.0f;

	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
		for (int i = 0; i < TA.reconstruction_size; i++) {
			MaXTile = max(MaXTile, val_rec[i]); // sanity check, check max
		}
	std::cout << "max device =" << MaXTile << "\n";
	for (int i = 0; i < TA.reconstruction_size; i++){
		i_rec[i] = 255.0 * val_rec[i] / MaXTile;			// Validation image value
	if(VERBOSE)
		if(i_rec[i] > 1)
	printf("i %d, col %d x %d y %d\n", i, TA.Nb_Cols_reconstruction, i % TA.Nb_Cols_reconstruction, i / TA.Nb_Cols_reconstruction);
	}
	printf("REC \u24FC Path to rec validation %s .....\n", recValImagefile);

	sdkSavePGM(recValImagefile, i_rec, TA.Nb_Cols_reconstruction, TA.Nb_Rows_reconstruction);

	printf("REC \u24FC Comparing files ... ");
	testrec = compareData(val_rec, original_rec, TA.reconstruction_size, MAX_EPSILON_ERROR, 0.15f);

	for (int jrec = 0; jrec < TA.reconstruction_size; jrec++) {
		Sumdel[1] += fabsf(*(val_rec + jrec) - *(original_rec + jrec));
	}
	printf("Sumdel[1] %f  ", Sumdel[1]);
	std::cout << "testrec = " << testrec << "\n";


	hipFree(val_rec);
	return (testrec);
}

void Scratchprepare(void) {
	float Maxscratch = 0.0f, Sumscratch = 0.0f, maxTile = 0.0f, SumTile = 0.0f;

	const char * rectilereconstructionfile = "results/tilerec.pgm";
	const char * scratchpadImagefile = "results/scratchpad.pgm";

	float *tile_rec = (float*) std::calloc(ATile * tile.NbTileXY , sizeof(float)); 					// on host
	unsigned char *i_tilerec = (unsigned char *) calloc(ATile * tile.NbTileXY, sizeof(unsigned char));// on host
	hipMallocManaged(&scratchpad_matrix, tile.NbTileXY * ASCRATCH * sizeof(float));
	unsigned char *i_scratchpad = (unsigned char *) calloc(tile.NbTileXY * XSCRATCH * YSCRATCH, sizeof(unsigned char)); // on host

/***********************************TILE RECONSTRUCTION ************************/
	printf("TILE \u24FC Path to tile reconstruction  %s .....", rectilereconstructionfile);

	int deltilex = tile.NbTilex * XTile - TA.Nb_Cols_reconstruction;
	int deltiley = tile.NbTiley * YTile - TA.Nb_Rows_reconstruction;
	printf(" offset = - del /2 !! x %d  y  %d\n", deltilex/2,deltiley/2);

	for (int arg = 0; arg < TA.reconstruction_size; arg++) {
		maxTile = max(maxTile,*(original_rec + arg));
		SumTile += *(original_rec + arg);
	}
	Maxscratch = maxTile; Sumscratch = SumTile;
	printf("TILE \u24FC maxTile %f SumTile %f\n",maxTile, SumTile);

	for (int row = 0; row < TA.Nb_Rows_reconstruction; row++)
		for (int col = 0; col < TA.Nb_Cols_reconstruction; col++) {
			int itemp = col + deltilex / 2 + (row + deltiley / 2) * tile.NbTilex * XTile;
			int itemp2 = col +  row  * TA.Nb_Cols_reconstruction;
			*(tile_rec + itemp) = *(original_rec + itemp2);
			i_tilerec[itemp] = 255. * *(tile_rec + itemp) / maxTile;
			if(VERBOSE)
				if(i_tilerec[itemp] > 1)
			printf("itemp %d, col %d x %d y %d\n", itemp, XTile*tile.NbTilex, itemp % (XTile*tile.NbTilex), itemp / (XTile*tile.NbTilex));

		}

	sdkSavePGM(rectilereconstructionfile, i_tilerec, XTile*tile.NbTilex, YTile * tile.NbTiley);

	// write scratchpad matrix to disk

	for (int iy = 0; iy < tile.NbTiley; iy++)
		for (int ix = 0; ix < tile.NbTilex; ix++)
			for (int iix = 0; iix < XTile; iix++)
				for (int iiy = 0; iiy < YTile; iiy++) {

					int iscratch = lostpixels + iix + dxSCRo2; 		// contribution of x in the 1D SCRATCH
					iscratch += ix * XSCRATCH; 					// contribution of previous tiles in x
					iscratch += (iiy + dySCRo2) * XSCRATCH * tile.NbTilex; 		// contribution of y in 1D SCRATCH
					iscratch += iy * YSCRATCH * XSCRATCH  * tile.NbTilex; 	// contribution of previous tiles in y

					int itile = iix;  // contribution of x in the TILE
					itile += ix  * XTile; // contribution of previous tile in x
					itile += iiy * XTile * tile.NbTilex; // contribution of y in the TILE
					itile += iy  * ATile * tile.NbTilex ; // contribution of previous tiles in y

					int iscratch2Dx = iix + dxSCRo2 + ix * XSCRATCH; 	// contribution of x in the 1D SCRATCH + contribution of previous tiles in x
					int iscratch2Dy = iiy + dySCRo2 + iy * YSCRATCH; 		// contribution of y in 1D SCRATCH +contribution of previous tiles in y
					int iscratch2D = iscratch2Dx + iscratch2Dy * XSCRATCH * tile.NbTilex;
					scratchpad_matrix[iscratch] = tile_rec[itile];
					i_scratchpad[iscratch2D] = 255.0 * tile_rec[itile] / Maxscratch;
					if(!(i_scratchpad[iscratch2D] ==0) && VERBOSE){
					printf("SCRATCHPAD \u24FC itile %d, iscratch %d iscratch2Dx %d, iscratch2Dy %d iscratch2D %d\n",
							itile, iscratch, iscratch2Dx, iscratch2Dy, iscratch2D);
					printf("SCRATCHPAD \u24FC itile %d, i_scratchpad[iscratch2D] %d scratchpad_matrix[iscratch] %f tile_rec[itile] %f\n",
							itile, i_scratchpad[iscratch2D], scratchpad_matrix[iscratch], tile_rec[itile]);
					}
				}

	printf("SCRATCHPAD \u24FC :Image of scratchpad matrix  %s .....\n", scratchpadImagefile);
	printf("SCRATCHPAD \u24FC : Max Scratchpad %f Sum scratchpad %f \n", Maxscratch, Sumscratch);
	printf("SCRATCHPAD \u24FC : size of one SCRATCHPAD 2D %d of full SCRATCHPAD 2D %d\n", XSCRATCH * YSCRATCH,
			XSCRATCH * YSCRATCH * tile.NbTileXY);
	sdkSavePGM(scratchpadImagefile, i_scratchpad, XSCRATCH * tile.NbTilex, YSCRATCH * tile.NbTiley);
	free(i_scratchpad);

}

bool Scratchvalidate_host(void) {
	bool testScratchpad;
	float MaxScratchpad = 0.0f, Sum3Scratchpad = 0.0f, max3Scratchpad = 0.0f;

	// write Scratchpad in memory and validate
	unsigned char *i_Scratchpad = (unsigned char *) calloc(tile.NbTileXY * XSCRATCH * YSCRATCH, sizeof(unsigned char)); // on host
	hipMallocManaged(&val_scratchpad, tile.NbTileXY * ASCRATCH * sizeof(float));
	hipMallocManaged(&val2_scratchpad, tile.NbTileXY * ASCRATCH * sizeof(float));
	const char * ScratchpadValImagefile = "results/ScratchpadValImagefile.pgm";

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Scratchpad kernel
	Scratchvalidate_device<<<dimGrid, dimBlock, 0>>>(tile.NbTilex,tile.NbTiley, lostpixels);
	hipDeviceSynchronize();

	for (int arg = 0; arg < ASCRATCH*tile.NbTileXY; arg++) {
			Sum3Scratchpad += *(val_scratchpad + arg);
			max3Scratchpad  = max(max3Scratchpad, *(val_scratchpad + arg));
		}
	printf("SCRATCHPAD \u24FC Sum3Scratchpad  %f max3Scratchpad %f   ", Sum3Scratchpad, max3Scratchpad);

	// write Scratchpad image validation to disk
	/////////////////////////////////
	MaxScratchpad = 0.0f;

	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
		for (int i = 0; i < ASCRATCH *tile.NbTileXY; i++) {
			MaxScratchpad = max(MaxScratchpad, val_scratchpad[i]); // sanity check, check max
		}
	std::cout << "max device =" << MaxScratchpad << "\n";

	for (int ity = 0; ity < tile.NbTiley; ity ++)
		for (int itx = 0; itx < tile.NbTilex; itx ++){
			int it = itx + ity*tile.NbTilex;
			for(int arg = lostpixels; arg < XSCRATCH*YSCRATCH + lostpixels; arg++){
				int arg1D = arg + it*ASCRATCH;
				int argy = (arg - lostpixels)/ XSCRATCH;
				int argx = (arg - lostpixels)% XSCRATCH;
				int arg2D = argx + itx * XSCRATCH + argy*XSCRATCH* tile.NbTilex+ ity * YSCRATCH * XSCRATCH* tile.NbTilex;
				i_Scratchpad[arg2D] = 255.0 * val_scratchpad[arg1D] / MaxScratchpad;			// Validation image value
			}
		}

	printf("SCRATCHPAD \u24FC Path to Scratchpad validation %s .....\n", ScratchpadValImagefile);
	sdkSavePGM(ScratchpadValImagefile, i_Scratchpad, XSCRATCH*tile.NbTilex, YSCRATCH*tile.NbTiley);

	printf("SCRATCHPAD \u24FC Comparing files ... ");
	testScratchpad = compareData(val_scratchpad, scratchpad_matrix, ASCRATCH *tile.NbTileXY,MAX_EPSILON_ERROR, 0.15f);

	for (int jScratchpad = 0; jScratchpad < ASCRATCH *tile.NbTileXY; jScratchpad++) {
		Sumdel[8] += fabsf(*(val_scratchpad + jScratchpad) - *(scratchpad_matrix + jScratchpad));
	}
	printf("Sumdel[8] %f  ", Sumdel[8]);
	std::cout << "testScratchpad = " << testScratchpad << "\n";
	hipFree(val_scratchpad);
	return (testScratchpad);
}

