#include "hip/hip_runtime.h"
#include "NewLoop.h"

using namespace tinyxml2;
using namespace std;

#define TEST 1
#define verboseNewLoop 1

//////////////pPSF parameters
//__device__ float PSF_array[pPSF*pZOOM*pPSF*pZOOM]; // pPSF in constant memory

__managed__ float *PSFvalidationdata_managed = NULL;
__managed__ float *distrib = NULL, *v_distrib = NULL;			 // on device original and for validation
__managed__ float *MicroImages = NULL, *v_MicroImages = NULL;				 // on device original and for validation
__managed__ float *LaserPositions = NULL, *v_LaserPositions = NULL;		 // on device original and for validation
__managed__ float *PosLaserx = NULL, *PosLasery = NULL;
__managed__ float *d_PosLaserx, *d_PosLasery;

__managed__ int *ROIx, *ROIy;
__managed__ int *d_ROIx, *d_ROIy;
__managed__ float *microimages, *d_microimages;
__managed__ float *original_distrib, *val_distrib, *test_distrib;
__managed__ double *double_distrib;
__managed__ float *original_microimages, *valmicroimages, *MIintile, *zoomed_microimages;
__managed__ double *double_microimages;
__managed__ float *original_rec, *val_rec;
__managed__ double *double_rec;
__managed__ float *scratchpad_matrix, *val_scratchpad, *val2_scratchpad;

__managed__ float *PSFARRAY;
__managed__ float *Sumdevmicroimages, *Maxdevmicroimages, *Sumdevzoommicroimages, *Maxdevzoommicroimages;

__global__ void PSFvalidateondevice(int Nb_Rows_PSF, int Nb_Cols_PSF) {
	double SumPSF = 0, maxPSF = 0, Sum2PSF = 0, max2PSF = 0; // in pPSF the Sum of all pixels is normalize to 1.00000
	float tempv;
	time_start = clock64();
	time_init = clock64();
	printf(" PSF \u2776 test: Rows: %d cols: %d \n", Nb_Rows_PSF, Nb_Cols_PSF);
// calculate pPSF on device Sum and max
	for (int row = 0; row < Nb_Rows_PSF; row++)
		for (int col = 0; col < Nb_Cols_PSF; col++) {
			int tempp = row * Nb_Cols_PSF + col;
			tempv = *(original_PSF+tempp);
			PSFARRAY[tempp] = *(original_PSF+tempp)+0.002;
			if (verboseNewLoop)
				if ((row == (Nb_Rows_PSF / 2)) && !(col % 8))
					printf(" PSF \u2776 device: tempv,%g row %d column %d, tempp %d\n", tempv, row, col, tempp);
			*(PSFvalidationdata_managed + tempp) = tempv;
			Sum2PSF += *(PSFvalidationdata_managed + tempp);
			SumPSF += PSFARRAY[tempp];
			if (maxPSF < PSFARRAY[tempp])
				maxPSF = PSFARRAY[tempp];
			if (max2PSF < *(PSFvalidationdata_managed + row * Nb_Cols_PSF + col))
				max2PSF = *(PSFvalidationdata_managed + row * Nb_Cols_PSF + col);
		}
	if ((threadIdx.x == 0) && (threadIdx.y == 0))
		printf(" PSF \u2776 device: SumPSF %f Sum2PSF %f maxPSF %f max2PSF %f ...  \n", SumPSF, Sum2PSF, maxPSF, max2PSF);
	timer = clock64();
}

__global__ void validate_distrib(int Nb_Rows_distrib, int Nb_Cols_distrib, int Nb_Distrib) {
	double Sumdistrib = 0, maxdistrib = 0, Sum2distrib = 0, max2distrib = 0; // in distrib the Sum of all pixels is normalize to 1.00000
	float tempv;
	int tempp;
	time_start = clock64();
	printf(" DISTRIBUTIONS \u2777  device: Nb_Row %d, Nb_col %d, Nb_distrib %d\n", Nb_Rows_distrib, Nb_Cols_distrib, Nb_Distrib);
// calculate distrib Sum and max
	for (int idistrib = 0; idistrib < Nb_Distrib; idistrib++)
		for (int row = 0; row < Nb_Rows_distrib; row++)
			for (int col = 0; col < Nb_Cols_distrib; col++) {
				tempp = (idistrib * Nb_Rows_distrib + row) * Nb_Cols_distrib + col;
				tempv = *(original_distrib + tempp);
				if (verboseNewLoop)
					if (!(row%21))
						if (!(col % 25))
							printf(" DISTRIBUTIONS \u2777  device: tempv,%g idistrib %d row %d"
									" column %d, tempp %d\n", tempv, idistrib, row, col, tempp);
				*(val_distrib + tempp) = tempv;
				Sum2distrib += *(val_distrib + row * Nb_Cols_distrib + col);
				Sumdistrib += *(original_distrib + row * Nb_Cols_distrib + col);
				if (maxdistrib < *(original_distrib + row * Nb_Cols_distrib + col))
					maxdistrib = *(original_distrib + row * Nb_Cols_distrib + col);
				if (max2distrib < *(val_distrib + row * Nb_Cols_distrib + col))
					max2distrib = *(val_distrib + row * Nb_Cols_distrib + col);
			}
		printf(" DISTRIBUTIONS \u2777 device: Sum distrib %f Sum2distrib %f \n"
				" DISTRIBUTIONS \u2777           max distrib %f max2distrib %f ...  \n", Sumdistrib, Sum2distrib, maxdistrib, max2distrib);
	timer = clock64();

}

__global__ void validateLaserPositions_device(int Nb_LaserPositions) {
	double maxLaserPositionx = 0, maxLaserPositiony = 0; // in LaserPosition the max in x is xmax
	double minLaserPositionx = 1E6, minLaserPositiony = 1E6; // in LaserPosition the max in x is xmax
// calculate LaserPositions Sum and max
	time_start = clock64();

	for (int ipos = 0; ipos < Nb_LaserPositions; ipos++) {
		d_PosLaserx[ipos] = PosLaserx[ipos];
		d_PosLasery[ipos] = PosLasery[ipos];
		if (verboseNewLoop && (ipos < 10)) {
			printf(" Laser \u2778 DEVICE: laser position n° %d original position x:%f , y: %f ....  \n",
					ipos, PosLaserx[ipos],PosLasery[ipos]);
			printf(" Laser \u2778 DEVICE: copy position: %f y: %f\n", d_PosLaserx[ipos], d_PosLasery[ipos]);
		}

		if (minLaserPositionx > PosLaserx[ipos]) minLaserPositionx = PosLaserx[ipos];
		if (maxLaserPositionx < PosLaserx[ipos]) maxLaserPositionx = PosLaserx[ipos];
		if (maxLaserPositiony < PosLasery[ipos]) maxLaserPositiony = PosLasery[ipos];
		if (minLaserPositiony > PosLasery[ipos]) minLaserPositiony = PosLasery[ipos];
	}
		printf(
				" Laser \u2778  DEVICE: LaserPosition x max %f min %f ... LaserPositiony max %f min %f \n",
				maxLaserPositionx, minLaserPositionx, maxLaserPositiony, minLaserPositiony);
	timer = clock64();

}

__global__ void validateCroppedROI_device(int Nb_ROI) {
	int maxROIx = 0, max2ROIx = 0; // in ROI the max in x is xmax
	int maxROIy = 0, max2ROIy = 0; // in ROI the max in y is ymax
// calculate pPSF Sum and max
	time_start = clock64();
	for (uint row = 0; row < Nb_ROI; row++) {
		d_ROIx[row] = ROIx[row];
		d_ROIy[row] = ROIy[row];
		if ((verboseNewLoop) && ((row < 10) || !(row % 512))) {
				printf("ROI \u2779 DEVICE:  original ROIx,%d row %d, ROIy %d ....  ", ROIx[row], row, ROIy[row]);
				printf(" copy d_ROIx,%d row %d, d_ROIy %d\n", d_ROIx[row], row, d_ROIy[row]);
			}

		maxROIx = max(maxROIx, d_ROIx[row]);
		max2ROIx = max(max2ROIx, ROIx[row]);
		maxROIy = max(maxROIy, d_ROIy[row]);
		max2ROIy = max(max2ROIy, ROIy[row]);
	}
		printf("ROI \u2779 DEVICE: \u21C8 ROI maxROI %d max2ROI %d ...  maxROI %d max2ROI %d .....Nb_ROI %d\n",
				maxROIx, max2ROIx, maxROIy,max2ROIy, Nb_ROI);
	timer = clock64();
}
__managed__ float * SumMI;

__global__ void validate_microimages(int Nb_LaserPositions) {
	float tempv;
	int tempp, tempz;
	int row, col, rowz, colz;
	int iprint = threadIdx.x + threadIdx.y;
	int iblock = blockIdx.x + blockIdx.y;
	col = threadIdx.x;
	colz =  threadIdx.x * pZOOM + blockIdx.x;
	row = threadIdx.y;
	rowz = threadIdx.y * pZOOM+ blockIdx.y;
	if(!(iprint+iblock)) 	time_start = clock64();
	__syncthreads();

	for (int ilaser = 0; ilaser < Nb_LaserPositions; ilaser++) {

		tempp = (ilaser * Npixel + row) * Npixel + col;
		tempz = (ilaser * Npixel*pZOOM + rowz) * Npixel*pZOOM + colz;
		tempv = *(original_microimages + tempp);

		*(valmicroimages + tempp) = tempv;
		*(zoomed_microimages + tempz) = tempv;
		__syncthreads();
	}

}

__global__ void microimages_intiles(int Nb_tiles, int nbLintile) {
}
__global__ void Recvalidate_device(int Nb_Rows_reconstruction, int Nb_Cols_reconstruction) {
	double Sumreconstruction = 0.0f;
	double maxreconstruction = 1.0f;

	float tempv;
	int tempp;
// calculate reconstruction Sum and max
	time_start = clock64();
	for (int row = 0; row < Nb_Rows_reconstruction; row++)
		for (int col = 0; col < Nb_Cols_reconstruction; col++) {
			tempp = row * Nb_Cols_reconstruction + col;
			tempv = *(original_rec + tempp);
			*(val_rec + tempp) = tempv;
			if ((tempv != 0.0f) && (TEST)){
				printf("REC \u277C DEVICE ----------------------------------------------------------------------------------------------------\n");
				printf("REC \u277C DEVICE position %d position x: %d y: %d value %f\n", tempp, tempp % Nb_Cols_reconstruction,
						tempp / Nb_Cols_reconstruction, tempv);
				printf("REC \u277C DEVICE ----------------------------------------------------------------------------------------------------\n");
			}
			Sumreconstruction += *(original_rec + row * Nb_Cols_reconstruction + col);
			if (maxreconstruction < *(original_rec + row * Nb_Cols_reconstruction + col))
				maxreconstruction = *(original_rec + row * Nb_Cols_reconstruction + col);
		}
	printf("REC \u277C DEVICE:  Sum reconstruction %f max reconstruction %f ...  ", Sumreconstruction, maxreconstruction);
	__syncthreads();
	if ((threadIdx.x == 0) && (threadIdx.y == 0))
		timer = clock64();
	__syncthreads();

}

__global__ void Scratchvalidate_device(int NbTilex, int NbTiley, int dels) {
	float Sumscratchpad = 0.0f, maxscratchpad = 0.0f;
	float tempv;
	int NbTile = NbTilex * NbTiley;
// calculate scratchpad Sum and max
	time_start = clock64();
	for (int tempp = 0; tempp < ASCRATCH * NbTile; tempp++) {
		tempv = *(scratchpad_matrix + tempp);
		*(val_scratchpad + tempp) = tempv;
		Sumscratchpad += *(val_scratchpad + tempp);
		if (maxscratchpad < *(val_scratchpad + tempp))
			maxscratchpad = *(val_scratchpad + tempp);

		if ((*(val_scratchpad + tempp) != 0.0f) && (TEST)) {
			int positionx = (tempp - dels) % (XSCRATCH * NbTilex);
			int positiony = (tempp - dels) / (XSCRATCH * NbTilex);

			printf("SCRATCHPAD \u24EC DEVICE TEST:  position %d position x: %d y: %d value %f\n",
					tempp, positionx, positiony, tempv);
		}

	}
	printf("SCRATCHPAD \u24EC DEVICE:  Sum scratchpad %f max scratchpad %f ... \n", Sumscratchpad, maxscratchpad);
	timer = clock64();
	__syncthreads();

}


