#include "hip/hip_runtime.h"
#include "0_NewLoop.h"

using namespace tinyxml2;
using namespace std;

#define TEST 1
#define VERBOSEINITLOOP 0

const char * Laserfile = "results/LaserDevice.txt";


//////////////pPSF parameters
//__device__ float PSF_array[pPSF*pZOOM*pPSF*pZOOM]; // pPSF in constant memory

__managed__ float *PSF_valid = NULL, *PSFARRAY = NULL, *original_PSF=NULL;
__managed__ float *distrib = NULL;			 // on device original and for validation
__managed__ float *MicroImages = NULL;				 // on device original and for validation
__managed__ float *LaserPositions = NULL, *v_LaserPositions = NULL;		 // on device original and for validation
__managed__ float *PosLaserx = NULL, *PosLasery = NULL;
__managed__ float *d_PosLaserx, *d_PosLasery;
__managed__ int *PosxScratch = NULL, *PosyScratch = NULL;
__managed__ int *posxREC, *posyREC, *offsetFULL;
__managed__ int *d_PosxScratch = NULL, *d_PosyScratch = NULL;
__managed__ int *d_posxREC, *d_posyREC, *d_offsetFULL;

__managed__ int *ROIx, *ROIy;
__managed__ int *d_ROIx, *d_ROIy;
__managed__ int *ROIxScratch, *ROIyScratch, *offsetROI;
__managed__ float *microimages, *d_microimages;
__managed__ float *original_distrib, *val_distrib, *test_distrib, *test2_distrib;
__managed__ double *double_distrib;
__managed__ float *original_microimages, *valmicroimages, *MIintile, *zoomed_microimages;
__managed__ float *original_rec, *val_rec;
__managed__ double *double_rec;
__managed__ float *scratchpad_matrix, *val_scratchpad, *val2_scratchpad;

__managed__ float *Sumdevmicroimages, *Maxdevmicroimages, *Sumdevzoommicroimages, *Maxdevzoommicroimages;

__global__ void PSFvalidateondevice(int Nb_Rows_PSF, int Nb_Cols_PSF) {
	double SumPSF = 0, maxPSF = 0, Sum2PSF = 0, max2PSF = 0; // in pPSF the Sum of all pixels is normalize to 1.00000
	float tempv;
	time_start = clock64();
	time_init = clock64();
	printf(" PSF \u2776 device: Rows: %d cols: %d ...", Nb_Rows_PSF, Nb_Cols_PSF);
// calculate pPSF on device Sum and max
	for (int row = 0; row < Nb_Rows_PSF; row++)
		for (int col = 0; col < Nb_Cols_PSF; col++) {
			int tempp = row * Nb_Cols_PSF + col;
			tempv = *(original_PSF + tempp);
			PSFARRAY[tempp] = *(original_PSF + tempp);
			if (VERBOSEINITLOOP)
				if ((row == (Nb_Rows_PSF / 4)) && !(col % 8))
					printf(" PSF \u2776  VERBOSE tempv,%g row %d column %d, tempp %d\n", tempv, row, col, tempp);
			*(PSF_valid + tempp) = tempv;
			Sum2PSF += *(PSF_valid + tempp);
			SumPSF += PSFARRAY[tempp];
			if (maxPSF < PSFARRAY[tempp])
				maxPSF = PSFARRAY[tempp];
			if (max2PSF < *(PSF_valid + row * Nb_Cols_PSF + col))
				max2PSF = *(PSF_valid + row * Nb_Cols_PSF + col);
		}
	if ((threadIdx.x == 0) && (threadIdx.y == 0))
		printf("  SumPSF %f Sum2PSF %f maxPSF %f max2PSF %f ...  \n", SumPSF, Sum2PSF, maxPSF,
				max2PSF);
	timer = clock64();
}

__global__ void validate_distrib(int Nb_Rows_distrib, int Nb_Cols_distrib, int Nb_Distrib) {
	double Sumdistrib = 0, maxdistrib = 0, Sum2distrib = 0, max2distrib = 0; // in distrib the Sum of all pixels is normalize to 1.00000
	float tempv;
	int tempp;
	time_start = clock64();
	printf("DISTRIBUTIONS \u2777  device: Nb_Row %d, Nb_col %d, Nb_distrib %d\n", Nb_Rows_distrib, Nb_Cols_distrib,
			Nb_Distrib);
// calculate distrib Sum and max
	for (int idistrib = 0; idistrib < Nb_Distrib; idistrib++)
		for (int row = 0; row < Nb_Rows_distrib; row++)
			for (int col = 0; col < Nb_Cols_distrib; col++) {
				tempp = (idistrib * Nb_Rows_distrib + row) * Nb_Cols_distrib + col;
				tempv = *(original_distrib + tempp);
				if (VERBOSEINITLOOP)
					if (!(row % 31))
						if (!(col % 25))
							printf("DISTRIBUTIONS \u2777  device: tempv,%g idistrib %d row %d"
									" column %d, tempp %d\n", tempv, idistrib, row, col, tempp);
				*(val_distrib + tempp) = tempv;
				Sum2distrib += *(val_distrib + row * Nb_Cols_distrib + col);
				Sumdistrib += *(original_distrib + row * Nb_Cols_distrib + col);
				if (maxdistrib < *(original_distrib + row * Nb_Cols_distrib + col))
					maxdistrib = *(original_distrib + row * Nb_Cols_distrib + col);
				if (max2distrib < *(val_distrib + row * Nb_Cols_distrib + col))
					max2distrib = *(val_distrib + row * Nb_Cols_distrib + col);
			}
	printf("DISTRIBUTIONS \u2777 device: Sum distrib %f Sum2distrib %f \n"
			"DISTRIBUTIONS \u2777           max distrib %f max2distrib %f ...  \n", Sumdistrib, Sum2distrib,
			maxdistrib, max2distrib);
	timer = clock64();

}

__global__ void validateLaserPositions_device(int Nb_LaserPositions) {
	double maxLaserPositionx = 0, maxLaserPositiony = 0; // in LaserPosition the max in x is xmax
	double minLaserPositionx = 1E6, minLaserPositiony = 1E6; // in LaserPosition the max in x is xmax
	int xREC, yREC, tilex, tiley;
// calculate LaserPositions Sum and max
	time_start = clock64();

	for (int ipos = 0; ipos < Nb_LaserPositions; ipos++) {
		d_PosLaserx[ipos] = PosLaserx[ipos];
		d_PosLasery[ipos] = PosLasery[ipos];
		// Laser positions in y zoomed in integer
		xREC = nearbyintf(pZOOM * *(PosLaserx + ipos));
		tilex = xREC / XTile;
		*(PosxScratch + ipos) = (xREC % XTile) + dxSCR / 2; // Laser positions in x zoomed integer in the scratchpad of tile
		yREC = nearbyintf(pZOOM * *(PosLasery + ipos));
		tiley = yREC / YTile;
		*(PosyScratch + ipos) = (yREC % YTile) + dySCR / 2; // Laser positions in x zoomed integer in the scratchpad of tile
		*(offsetFULL + ipos) = *(PosxScratch + ipos) + *(PosyScratch + ipos) * XSCRATCH;
		if (VERBOSEINITLOOP && (ipos < 20)) {
			printf(" Laser \u2778 DEVICE 1: laser position n° %d original position x:%f , y: %f ....\n", ipos,
					PosLaserx[ipos], PosLasery[ipos]);
			printf(" Laser \u2778 DEVICE 2:  ....xREC %d yREC %d tilex %d tiley %d Scratchx %d Scratchy %d \n", xREC,
					yREC, tilex, tiley, PosxScratch[ipos], PosyScratch[ipos]);
			printf(" Laser \u2778 DEVICE 3: copy position: %f y: %f\n\n", d_PosLaserx[ipos], d_PosLasery[ipos]);
		}

		if (minLaserPositionx > PosLaserx[ipos])
			minLaserPositionx = PosLaserx[ipos];
		if (maxLaserPositionx < PosLaserx[ipos])
			maxLaserPositionx = PosLaserx[ipos];
		if (maxLaserPositiony < PosLasery[ipos])
			maxLaserPositiony = PosLasery[ipos];
		if (minLaserPositiony > PosLasery[ipos])
			minLaserPositiony = PosLasery[ipos];
	}
	printf(" Laser \u2778  DEVICE: MAX & MIN: LaserPosition x max %f min %f ... LaserPositiony max %f min %f \n",
			maxLaserPositionx, minLaserPositionx, maxLaserPositiony, minLaserPositiony);
	timer = clock64();

}

__global__ void validateCroppedROI_device(int Nb_ROI) {
	int maxROIx = 0, max2ROIx = 0; // in ROI the max in x is xmax
	int maxROIy = 0, max2ROIy = 0; // in ROI the max in y is ymax
// calculate pPSF Sum and max
	time_start = clock64();
	for (uint row = 0; row < Nb_ROI; row++) {
		d_ROIx[row] = ROIx[row];
		d_ROIy[row] = ROIy[row];
		if ((VERBOSEINITLOOP) && ((row < 10) || !(row % 512))) {
			printf("ROI \u2779 DEVICE:  original ROIx,%d row %d, ROIy %d ....  ", ROIx[row], row, ROIy[row]);
			printf(" copy d_ROIx,%d row %d, d_ROIy %d\n", d_ROIx[row], row, d_ROIy[row]);
		}

		maxROIx = max(maxROIx, d_ROIx[row]);
		max2ROIx = max(max2ROIx, ROIx[row]);
		maxROIy = max(maxROIy, d_ROIy[row]);
		max2ROIy = max(max2ROIy, ROIy[row]);
	}
	if (VERBOSEINITLOOP)
		printf("ROI \u2779 DEVICE: \u21C8 ROI maxROI %d max2ROI %d ...  maxROI %d max2ROI %d .....Nb_ROI %d\n", maxROIx,
			max2ROIx, maxROIy, max2ROIy, Nb_ROI);
	timer = clock64();
}
__managed__ float * SumMI;

__global__ void validate_microimages(int Nb_LaserPositions) {
	float tempv;
	int tempp, tempz;
	int iprint = threadIdx.x + threadIdx.y;
	int iblock = blockIdx.x + blockIdx.y;
	if (!(iprint + iblock))
		time_start = clock64();
	__syncthreads();

	for (int ilaser = 0; ilaser < Nb_LaserPositions; ilaser++)
		for (int row = 0; row < Npixel; row++)
			for (int col = 0; col < Npixel; col++) {
				tempp = (ilaser * Npixel + row) * Npixel + col;
				tempv = *(original_microimages + tempp);
				*(valmicroimages + tempp) = tempv;

				for (int yz = 0; yz < pZOOM; yz++)
					for (int xz = 0; xz < pZOOM; xz++) {
						tempz = ilaser * PixZoomSquare + (row * pZOOM + yz) * PixZoom
								+ pZOOM * col + xz;
				*(zoomed_microimages + tempz) = tempv;
					}
				__syncthreads();
			}
	if (!(iprint + iblock))
		timer = clock64();

}

__global__ void microimages_intiles(int Nb_tiles, int nbLintile) {
}
__global__ void Recvalidate_device(int Nb_Rows_reconstruction, int Nb_Cols_reconstruction) {
	double Sumreconstruction = 0.0f;
	double maxreconstruction = 1.0f;

	float tempv;
	int tempp;
// calculate reconstruction Sum and max
	time_start = clock64();
	for (int row = 0; row < Nb_Rows_reconstruction; row++)
		for (int col = 0; col < Nb_Cols_reconstruction; col++) {
			tempp = row * Nb_Cols_reconstruction + col;
			tempv = *(original_rec + tempp);
			*(val_rec + tempp) = tempv;
			if ((tempv != 0.0f) && (TEST) && VERBOSE) {
				printf(
						"REC \u277D DEVICE ----------------------------------------------------------------------------------------------------\n");
				printf("REC \u277D DEVICE position %d position x: %d y: %d value %f\n", tempp,
						tempp % Nb_Cols_reconstruction, tempp / Nb_Cols_reconstruction, tempv);
				printf(
						"REC \u277D DEVICE ----------------------------------------------------------------------------------------------------\n");
			}
			Sumreconstruction += *(original_rec + row * Nb_Cols_reconstruction + col);
			if (maxreconstruction < *(original_rec + row * Nb_Cols_reconstruction + col))
				maxreconstruction = *(original_rec + row * Nb_Cols_reconstruction + col);
		}
	printf("REC \u277D DEVICE:  Sum reconstruction %f max reconstruction %f ...  ", Sumreconstruction,
			maxreconstruction);
	__syncthreads();
	if ((threadIdx.x == 0) && (threadIdx.y == 0))
		timer = clock64();
	__syncthreads();

}

__global__ void Scratchvalidate_device(int NbTilex, int NbTiley, int dels) {
	float Sumscratchpad = 0.0f, maxscratchpad = 0.0f;
	float tempv;
	int NbTile = NbTilex * NbTiley;
// calculate scratchpad Sum and max
	time_start = clock64();
	for (int tempp = 0; tempp < ASCRATCH * NbTile; tempp++) {
		tempv = *(scratchpad_matrix + tempp);
		*(val_scratchpad + tempp) = tempv;
		Sumscratchpad += *(val_scratchpad + tempp);
		if (maxscratchpad < *(val_scratchpad + tempp))
			maxscratchpad = *(val_scratchpad + tempp);

		if ((*(val_scratchpad + tempp) != 0.0f) && (TEST)) {
			int positionx = (tempp - dels) % (XSCRATCH * NbTilex);
			int positiony = (tempp - dels) / (XSCRATCH * NbTilex);
			if (VERBOSE)
				printf("SCRATCHPAD \u24EC DEVICE TEST:  position %d position x: %d y: %d value %f\n", tempp,
						positionx, positiony, tempv);
		}

	}
	printf("SCRATCHPAD \u24EC DEVICE:  Sum scratchpad %f max scratchpad %f ... \n", Sumscratchpad,
			maxscratchpad);
	timer = clock64();
	__syncthreads();

}

