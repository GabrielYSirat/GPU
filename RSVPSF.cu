#include "hip/hip_runtime.h"
/*
 * readvalidatePSF_control.cu
 *
 *  Created on: Apr 19, 2017
 *      Author: gabriel
 */

#include "NewLoop.h"
double *double_PSF;
float  *original_PSF=NULL;
float MaxPSF=0.0f, SumPSF = 0.0f;

void PSFprepare(void) {
	char * memblock;
	int size;
	XMLDocument doc;
	MaxPSF = 0.; // also used as extern

	string PSFraw = resourcesdirectory + "psf__27x27.raw";
	const char * PSFImagefile = "results/PSFImagefile.pgm";

	filename = resourcesdirectory + "bead_system_PSF.xml";
	printf(" PSF \u24F5 bead_system_PSF:  %s \n", filename.c_str());
	doc.LoadFile(filename.c_str());
	int Nb_Rows_PSF_file = atoi(doc.FirstChildElement("Image_Contents")->FirstChildElement("Nb_Rows")->GetText());
	int Nb_Cols_PSF_file = atoi(doc.FirstChildElement("Image_Contents")->FirstChildElement("Nb_Cols")->GetText());
	if ((Nb_Rows_PSF_file != TA.PSF_Rows)||(Nb_Cols_PSF_file != TA.Nb_Cols_PSF))
		printf(" PSF \u24F5 values stored in xml file differs from parameters File x: %d y: %d parameters x: %d y: %d \n",
				Nb_Rows_PSF_file, Nb_Cols_PSF_file, TA.PSF_Rows, TA.Nb_Cols_PSF);

	unsigned char *i_PSF = (unsigned char *) calloc(TA.PSF_size, sizeof(unsigned char)); // on host
    double* double_PSF = (double*)std::malloc(Nb_Rows_PSF_file*Nb_Cols_PSF_file * sizeof(double));
	hipMallocManaged(&original_PSF, PSFZoom * PSFZoom * sizeof(float));
	hipMallocManaged(&PSFARRAY, PSFZoom * PSFZoom *  sizeof(float));

	//read pPSF bin file
	std::ifstream PSFile(PSFraw.c_str(), ios::in | ios::binary | ios::ate);
	size = (PSFile.tellg()); 	// the data is stored in doubles of 8 bytes in the file
	size -= byte_skipped;  				// removes byte_skipped
	memblock = new char[size];
	PSFile.seekg(byte_skipped, ios::beg); // 4 first bytes are offset
	PSFile.read(memblock, size);
	PSFile.close();

	double_PSF = (double*) memblock; //reinterpret the chars stored in the file as double
	for (int i = 0; i < Nb_Rows_PSF_file*Nb_Cols_PSF_file; i++) {
				*(original_PSF + i) = *(double_PSF + i)+0.000001;			// change to float
				SumPSF += *(original_PSF+i);
				if (MaxPSF < *(original_PSF + i))
					MaxPSF = *(original_PSF + i); // sanity check, check max
	}

	printf(" PSF \u24F5  Nb_Rows: %d Nb_Cols_PSF %d size pPSF = %d max %g Sum %g \n",
			TA.PSF_Rows, TA.Nb_Cols_PSF, size, MaxPSF,SumPSF );
	tile.expectedmax = MaxPSF; // to be updated later on

	//    hipMemcpyToSymbol(HIP_SYMBOL(PSFARRAY), original_PSF, PSFZOOMSQUARE*sizeof(float));


	// write pPSF original image to disk
	/////////////////////////////////
	for (int i = 0; i <= TA.PSF_size; i++)
		i_PSF[i] = 255.0*original_PSF[i]/MaxPSF;			// image value
	printf(" PSF \u24F5 function read: Path to pPSF original %s .....\n", PSFImagefile);

	sdkSavePGM(PSFImagefile, i_PSF, TA.PSF_Rows, TA.Nb_Cols_PSF);
	free(i_PSF);

 }


bool PSFvalidateonhost(void) {
	bool testPSF;
	double MaxPSF;
	double Sum3PSF = 0, max3PSF =0;
		hipMallocManaged(&PSFvalidationdata_managed, TA.PSF_size * sizeof(float)); // representation of pPSF available in global memory
	unsigned char *i_PSF = (unsigned char *) calloc(TA.PSF_size, sizeof(unsigned char)); // on host
	const char * PSFValidationimage = "results/PSFValidationimage.pgm";

    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1,1, 1);
    // Execute the pPSF kernel
    PSFvalidateondevice<<<dimGrid, dimBlock, 0>>>( TA.PSF_Rows, TA.Nb_Cols_PSF);
    hipDeviceSynchronize();

   for(int row = 0; row < TA.PSF_Rows; row++)
    	for( int col = 0; col < TA.Nb_Cols_PSF; col++)
    		{
    		Sum3PSF += *(PSFvalidationdata_managed + row*TA.Nb_Cols_PSF + col);
     		if (max3PSF < *(PSFvalidationdata_managed + row*TA.Nb_Cols_PSF + col)) max3PSF = *(PSFvalidationdata_managed + row*TA.Nb_Cols_PSF + col);
    		}
	printf(" PSF \u24F5 Sum3PSF  %f max3PSF %f ", Sum3PSF, max3PSF);

	// write pPSF image validation to disk
	/////////////////////////////////
	MaxPSF = 0.0f;
	for (int i = 0; i <= TA.PSF_size; i++) {
		MaxPSF = max(MaxPSF, PSFvalidationdata_managed[i]); // sanity check, check max
	}
	cout << "max device = (3 digits) " << MaxPSF << "\n";
	for (int i = 0; i <= TA.PSF_size; i++)
		i_PSF[i] = 255.0*PSFvalidationdata_managed[i]/MaxPSF;			// Validation image value

	printf(" PSF \u24F5 Path to pPSF validation %s .....\n", PSFValidationimage);

	    	sdkSavePGM(PSFValidationimage, i_PSF, TA.PSF_Rows, TA.Nb_Cols_PSF);

	        printf(" PSF \u24F5 Comparing files ... \n");
	    	testPSF = compareData(PSFvalidationdata_managed,
	                                 original_PSF,
	                                 TA.Nb_Cols_PSF*TA.PSF_Rows,
	                                 MAX_EPSILON_ERROR/1000,
	                                 0.15f);

	        for (int jPSF = 0; jPSF < TA.PSF_size; jPSF++)
	        	Sumdel[1] += fabsf(*(PSFvalidationdata_managed+jPSF)- *(original_PSF+jPSF));
hipFree(PSFvalidationdata_managed);
return(testPSF);
}
