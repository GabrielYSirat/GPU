#include "hip/hip_runtime.h"
/*
 * initparameters.cu
 *
 *  Created on: May 7, 2017
 *      Author: gabriel
 */
/*
 * readLaserPositions.cu
 *
 *  Created on: April 18, 2017
 *      Author: gabriel
 */

#include "NewLoop.h"

__managed__ int XTile, YTile, ATile;
__managed__ int THreadsRatio, NThreads;
__managed__ int XDistrib, YDistrib, YDistrib_extended, lostlines, ADistrib;


hipDeviceProp_t deviceProps;

bool initparameters( int argc, char **argv) {
	XMLDocument XMLdoc, ACQXML, doc;
	XMLElement *pRoot, *pParm;
	string sstr, filenamexml;
;
	bool dimfit = TRUE;

	// acquire information on the CUDA device: name and number of multiprocessors
	devID = gpuDeviceInit(devID);

	std::cout << "MAIN PROGRAM  \u24EA NewLoop starting...";
	if (devID < 0) {
		printf("exiting...\n");
		exit(EXIT_FAILURE);
	}
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID)); 	// get device name & properties
	TA.MP = deviceProps.multiProcessorCount;
	TA.sharedmemory = deviceProps.sharedMemPerBlock;
	clockRate = deviceProps.clockRate;
	printf(" INIT PROG \u24EA Number of Multiprocessors (MP) %d, clock rate (KHz) %d SharedMemory %6.3f in KBytes\n\n",
			TA.MP, clockRate, (float) TA.sharedmemory/1024.);

	/***************************** FILES MANAGEMENT ***********************************/
	resourcesdirectory = argv[1]; // Directory with all preprocessing files and data
	pPSF = retrieveargv(argv[2]);
	Npixel = retrieveargv(argv[3]);
	RDISTRIB = retrieveargv(argv[4]);
	pZOOM = retrieveargv(argv[5]);
	Ndistrib = retrieveargv(argv[6]);
	// to read the values in the program and to add tests

	std::cout << "MAIN PROGRAM  \u24EA ARG: EXE arguments number argc: " << argc << endl;
	std::cout << "MAIN PROGRAM  \u24EA Line of command parameters: " << "******************************************"
			<< endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[1]: working directory: " << resourcesdirectory << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[2]: pPSF: " << pPSF << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[3]: Npixel: " << Npixel << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[4]: RDISTRIB: " << RDISTRIB << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[5]: pZOOM: " << pZOOM << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[6]: NDISTRIB: " << Ndistrib << endl;
	std::cout << "MAIN PROGRAM  \u24EA Line of command parameters: " << "******************************************"
			<< endl << endl;
	std::cout << "MAIN PROGRAM  \u24EA NewLoop starting...";

	/** initialize the general parameters and the offset & scale parameters
	 */
	TA.start();
	OFSCAL.start();

	filename = resourcesdirectory + "ACQ.xml";
	printf("filename %s \n", filename.c_str());
	printf(" INIT PROG \u24EA  scan points:  %s \n", filename.c_str());
	int LoadACQOK = XMLError(ACQXML.LoadFile(filename.c_str()));
	pRoot = ACQXML.FirstChildElement("BioAxialAcquisitionRequest");
	pParm = pRoot->FirstChildElement("LambdaParameters")->FirstChildElement("LambdaParameter")->FirstChildElement("dx");
	sstr = pParm->GetText();
	for (unsigned int i = 0; i < strlen(chars); ++i)
		sstr.erase(std::remove(sstr.begin(), sstr.end(), chars[i]), sstr.end());
	stringstream stream_x(sstr);
	stream_x.getline(buff, 10, ',');
	TA.dx = max(atoi(buff), TA.dx);
	if (verbose)
		cout << "verbose x: " << stream_x.str() << " y: ";

	pParm = pRoot->FirstChildElement("Camera_parameters")->FirstChildElement("PixelSize_nm");
	sstr = pParm->GetText();
	for (unsigned int i = 0; i < strlen(chars); ++i)
		sstr.erase(std::remove(sstr.begin(), sstr.end(), chars[i]), sstr.end());
	stringstream stream_p(sstr);
	stream_p.getline(buff, 10, ',');
	TA.Pixel_size_nm = atoi(buff);
	printf(" INIT PROG \u24EA PARAMS :  pixel size %g  nm\n",TA.Pixel_size_nm);

	pParm = pRoot->FirstChildElement("LambdaParameters")->FirstChildElement("LambdaParameter")->FirstChildElement("dy");
	sstr = pParm->GetText();
	for (unsigned int i = 0; i < strlen(chars); ++i)
		sstr.erase(std::remove(sstr.begin(), sstr.end(), chars[i]), sstr.end());
	stringstream streamy(sstr);
	streamy.getline(buff, 10, ',');
	TA.dy = max(atoi(buff), TA.dy);
	if (verbose)
		cout << streamy.str() << endl;
	printf(" INIT PROG \u24EA maximum number of scan points in x and y, x: %d, y:%d \n", TA.dx, TA.dy);

	/** Threads parameters
		 *
		 */
		THreadsRatio = CEILING_POS((1.0*PixZoomSquare)/THREADSVAL);
		NThreads = THreadsRatio * THREADSVAL;

		/** distrib parameters: cover all possible values of the distribution up
		 *  to the corner of the microimage included dummy pixels incremented by the pPSF size -1 to avoid counting twice the pixel
		 */
		XDistrib = (PixZoomo2+PSFZoomo2)*2+1;
		YDistrib = (PixZoomo2+PSFZoomo2)*2+1;
		YDistrib_extended = NThreads/PixZoom+PSFZoom;
		lostlines = ceil((1.0*NThreads)/PixZoom) - PixZoom;

		float tempe = XDistrib * YDistrib_extended;
		ADistrib = CEILING_POS(tempe/THREADSVAL)*THREADSVAL;


	/** SCRATCH and TILE parameters
	 *
	 */
	XTile = XSCRATCH - dxSCR;	// we do not care on x because the distribution will be zero
												// We add lostpixels  at start and end of the scratchpad
												// for "spillover" of the first and last line
	YTile = YSCRATCH - dySCR - lostlines; // in y we need the full size
	if((YTile%2)==0) YTile--;
	TA.XTileSize = (XTile * TA.Pixel_size_nm)/(1000.*pZOOM);
	TA.YTileSize = (YTile * TA.Pixel_size_nm)/(1000.*pZOOM);
	ATile = XTile * YTile;


	printf("************** DATA: PARAMETERS OF MEASUREMENT *******************\n"
			" INIT PROG \u24EA BASIC  : NThreads %d Npixel %d pZOOM %d, pPSF %d RDISTRIB %d\n", NThreads, Npixel, pZOOM,
			pPSF, RDISTRIB);
	printf(" INIT PROG \u24EA PIXEL  : Npixel %d PixZoom %d PixZoomo2 %d\n", Npixel, PixZoom, PixZoomo2);
	printf(" INIT PROG \u24EA PIXEL  : lost lines %d additional lines at the end of microimage\n", lostlines);

	printf(" INIT PROG \u24EA pPSF   : pPSF %d PSFZoom %d PSFZoomo2 %d \n", pPSF, PSFZoom, PSFZoomo2);
	printf(" INIT PROG \u24EA DISTRIB: XDistrib %d YDistrib %d YDistrib_extended %d  Size in KBytes %g ADistrib %d RDISTRIB %d\n",
											XDistrib, YDistrib, YDistrib_extended, ADistrib/1024., ADistrib, RDISTRIB);
	printf(" INIT PROG \u24EA SCRATCH: XSCRATCH %d YSCRATCH %d dxSCR %d dySCR %d\n", XSCRATCH, YSCRATCH, dxSCR, dySCR);
	printf(" INIT PROG \u24EA SCRATCH: DEL SCRATCH %d Additional pixels at start and end of SCRATCH\n", lostpixels);
	printf(" INIT PROG \u24EA TILE   : XTILE %d YTILE %d  size : XTILE:%6.3f µm YTILE %6.3f µm\n",  XTile, YTile, TA.XTileSize, TA.YTileSize);
	printf(" INIT PROG \u24EA PARAMS :  Number of threads %d Threads per batch %d number of batch %d\n\n",
	NThreads, THREADSVAL, THreadsRatio);
	printf(" INIT PROG \u23f3 Data parameters in device memory ...\n");

	printf("******************Retrieving microimages size **************\n");
	filename = resourcesdirectory + "cropped_measurements.xml";
	printf(" INIT PROG \u24EA  microimages:  %s \n", filename.c_str());
	LoadACQOK = XMLError(ACQXML.LoadFile(filename.c_str()));
	pRoot = ACQXML.FirstChildElement("Image_Contents");
	pParm = pRoot->FirstChildElement("Nb_Rows");
	if (verbose)
		printf(" INIT PROG \u24EA  Nb_Rows ");
	sstr = pParm->GetText();
	stringstream streamRows(sstr);
	streamRows.getline(buff, 10, ',');
	TA.Nb_Rows_microimages = atoi(buff);

	pParm = pRoot->FirstChildElement("Nb_Cols");
	sstr = pParm->GetText();
	stringstream streamCols(sstr);
	streamCols.getline(buff, 10, ',');
	TA.Nb_Cols_microimages = atoi(buff);
	printf(" INIT PROG \u24EA microimages Rows %d, columns in file %d and in constants %d\n\n", TA.Nb_Rows_microimages,
			TA.Nb_Cols_microimages, Npixel);

	if ((TA.Nb_Cols_microimages != TA.Nb_Rows_microimages)||(TA.Nb_Cols_microimages != Npixel)) {
		printf(" INIT PROG \u24EA non square image, Nb_Rows_microimages %d\n\n", TA.Nb_Cols_microimages);
		printf(" INIT PROG \u24EA non square image, Nb_Rows_microimages %d\n\n", TA.Nb_Cols_microimages);
		printf(" INIT PROG \u24EA Number of pixels does not fit %d\n\n", TA.Nb_Cols_microimages);
		exit(1);
	}

	filenamexml = resourcesdirectory + "reconstruction.xml";
	printf("INIT PROG \u24EA reconstruction xml:  %s \n", filenamexml.c_str());
	doc.LoadFile(filenamexml.c_str());

	TA.Nb_Rows_reconstruction = atoi(doc.FirstChildElement("Image_Contents")
			->FirstChildElement("Nb_Rows")->GetText());
	TA.Nb_Cols_reconstruction = atoi(doc.FirstChildElement("Image_Contents")
			->FirstChildElement("Nb_Cols")->GetText());
	/** Sanity check */
		printf("INIT PROG \u24EA reconstruction from tiles: Rows: %d Cols %d size %d \n",
				TA.Nb_Rows_reconstruction, TA.Nb_Cols_reconstruction, TA.reconstruction_size);

	return (dimfit);
}

