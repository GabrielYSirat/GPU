#include "hip/hip_runtime.h"
/*
 * initparameters.cu
 *
 *  Created on: May 7, 2017
 *      Author: gabriel
 */
/*
 *  Created on: April 18, 2017
 *      Author: gabriel
 */

#include "0_NewLoop.h"

/*******************PARAMETERS**************/
char buff[BUFFSIZE]; // a buffer to temporarily park the data
double Timestep[16];
char chars[] = "[]()", delimeter('=');
__managed__ int pPSF, Npixel, RDISTRIB, pZOOM, Ndistrib;

__managed__ double  Energy_global =0.0f;
__managed__ clock_t timer, time_init, time_start; // in KHz

std::string resourcesdirectory, filename, name, value, MIFILE, PSFFILE, DISTRIBFILE;

__managed__ int XTile, YTile, ATile;
__managed__ int THreadsRatio, NThreads;
__managed__ int XDistrib, YDistrib, YDistrib_extended, lostlines, ADistrib;

hipDeviceProp_t deviceProps;

bool initparameters( int argc, char **argv) {
	XMLDocument XMLdoc, ACQXML, doc;
	XMLElement *pRoot, *pParm;
	string sstr, filenamexml;
	bool dimfit = TRUE;

	// acquire information on the CUDA device: name and number of multiprocessors
	devID = gpuDeviceInit(devID);
	std::cout << "MAIN PROGRAM  \u24EA NewLoop start ...\n";
	if (devID < 0) {
		printf("exiting...\n");
		exit(EXIT_FAILURE);
	}

	/********************NVIDIA Card assesment and parameters ************************/
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID)); 	// get device name & properties
	TA.MP = deviceProps.multiProcessorCount;
	TA.sharedmemory = deviceProps.sharedMemPerBlock;
	clockRate = deviceProps.clockRate;
	printf(" INIT PROG \u24EA Number of Multiprocessors (MP) %d, clock rate (KHz) %d SharedMemory %6.3f in KBytes\n\n",
			TA.MP, clockRate, (float) TA.sharedmemory/1024.);

	/***************************** command line management*****************************/
	resourcesdirectory = argv[1]; 	// Directory with all preprocessing files and data
	pPSF = retrieveargv(argv[2]);	// PSF Size, without zoom
	Npixel = retrieveargv(argv[3]);	// Pixel number without zoom
	RDISTRIB = retrieveargv(argv[4]);
	pZOOM = retrieveargv(argv[5]);
	Ndistrib = retrieveargv(argv[6]);
	MIFILE = argv[7];
	// to read the values in the program and to add tests

	std::cout << "MAIN PROGRAM  \u24EA ARG: EXE arguments number argc: " << argc << endl;
	std::cout << "MAIN PROGRAM  \u24EA command line parameters: " << "******************************************"
			<< endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[1]: working directory: " << resourcesdirectory << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[2]: pPSF: " << pPSF << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[3]: Npixel: " << Npixel << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[4]: RDISTRIB: " << RDISTRIB << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[5]: pZOOM: " << pZOOM << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[6]: NDISTRIB: " << Ndistrib << endl;
	std::cout << "MAIN PROGRAM  \u24EA ARG: argv[7]: MI directory: " << resourcesdirectory + MIFILE << endl;
	std::cout << "MAIN PROGRAM  \u24EA command line parameters: " << "******************************************"
			<< endl <<endl;

	/** initialize the general parameters and the offset & scale parameters
	 */
	TA.start();
	OFSCAL.start();

	/** Threads parameters
		 *
		 */
		THreadsRatio = CEILING_POS((1.0*PixZoomSquare)/THREADSVAL);
		NThreads = THreadsRatio * THREADSVAL;

		/** distrib parameters: cover all possible values of the distribution up
		 *  to the corner of the microimage included dummy pixels incremented by the pPSF size -1 to avoid counting twice the pixel
		 */
		XDistrib = (PixZoomo2+PSFZoomo2)*2+1;
		YDistrib = (PixZoomo2+PSFZoomo2)*2+1;
		YDistrib_extended = NThreads/PixZoom+PSFZoom;
		lostlines = NThreads/PixZoom - PixZoom +1;

		float tempe = XDistrib * YDistrib_extended;
		ADistrib = CEILING_POS(tempe/THREADSVAL)*THREADSVAL;


	/** SCRATCH and TILE parameters
	 *
	 */
	XTile = XSCRATCH - dxSCR;	// we do not care on x because the distribution will be zero
								// We add lostpixels  at start and end of the scratchpad
								// for "spillover" of the first and last line
	YTile = YSCRATCH - dySCR - lostlines; 	// in y we need the full size
	if((YTile%2)==0) YTile--;				// We insure that YTile is odd
	ATile = XTile * YTile;										// Total size in pixels


	printf("************** DATA: PARAMETERS OF MEASUREMENT *************************************\n"
			" INIT PROG \u24EA BASIC  : THreadsRatio %d NThreads %d Npixel %d pZOOM %d, pPSF %d RDISTRIB %d\n", THreadsRatio, NThreads, Npixel, pZOOM,
			pPSF, RDISTRIB);
	printf(" INIT PROG \u24EA BASIC  : YTile %d YSCRATCH %d dySCR %d \n",YTile ,YSCRATCH, dySCR);

	printf(" INIT PROG \u24EA PIXEL  : Npixel %d PixZoom %d PixZoomo2 %d\n", Npixel, PixZoom, PixZoomo2);
	printf(" INIT PROG \u24EA PIXEL  : lost lines %d additional lines at the end of microimage\n", lostlines);

	printf(" INIT PROG \u24EA pPSF   : pPSF %d PSFZoom %d PSFZoomo2 %d \n", pPSF, PSFZoom, PSFZoomo2);
	printf(" INIT PROG \u24EA DISTRIB: XDistrib %d YDistrib %d YDistrib_extended %d  Size in KBytes %g ADistrib %d RDISTRIB %d\n",
											XDistrib, YDistrib, YDistrib_extended, ADistrib/1024., ADistrib, RDISTRIB);
	printf(" INIT PROG \u24EA SCRATCH: XSCRATCH %d YSCRATCH %d dxSCR %d dySCR %d\n", XSCRATCH, YSCRATCH, dxSCR, dySCR);
	printf(" INIT PROG \u24EA SCRATCH: DEL SCRATCH %d Additional pixels at start and end of SCRATCH\n", lostpixels);
	printf(" INIT PROG \u24EA PARAMS :  Number of threads %d Threads per batch %d number of batch %d\n",
	NThreads, THREADSVAL, THreadsRatio);
	printf("************** DATA: PARAMETERS OF MEASUREMENT *************************************\n\n");

	printf(" INIT PROG \u23f3 Data parameters in device memory ...\n");


	/********************************Reconstruction parameters *************************/
	filenamexml = resourcesdirectory + "reconstruction.xml";
	printf(" INIT PROG \u24EA reconstruction xml:  %s \n", filenamexml.c_str());
	doc.LoadFile(filenamexml.c_str());

	TA.Nb_Rows_reconstruction = atoi(doc.FirstChildElement("Image_Contents")
			->FirstChildElement("Nb_Rows")->GetText());
	TA.Nb_Cols_reconstruction = atoi(doc.FirstChildElement("Image_Contents")
			->FirstChildElement("Nb_Cols")->GetText());
	TA.reconstruction_size = TA.Nb_Cols_reconstruction*TA.Nb_Rows_reconstruction;
	printf(" INIT PROG \u24EA reconstruction from tiles: Cols %d Rows: %d size %d \n",
				 TA.Nb_Cols_reconstruction, TA.Nb_Rows_reconstruction, TA.reconstruction_size);

	/***********************Sizes in nm *************************************************/
	filename = resourcesdirectory + "ACQ.xml";
	int LoadACQOK = XMLError(ACQXML.LoadFile(filename.c_str()));
	pRoot = ACQXML.FirstChildElement("BioAxialAcquisitionRequest");
	pParm = pRoot->FirstChildElement("Camera_parameters")->FirstChildElement("PixelSize_nm");
	sstr = pParm->GetText();
	for (unsigned int i = 0; i < strlen(chars); ++i)
		sstr.erase(std::remove(sstr.begin(), sstr.end(), chars[i]), sstr.end());
	stringstream stream_p(sstr);
	stream_p.getline(buff, 10, ',');
	TA.Pixel_size_nm = atoi(buff);
	printf(" INIT PROG \u24EA PARAMS :  original µimage pixel size %g  nm pixel size reconstruction %g\n",
			TA.Pixel_size_nm, TA.Pixel_size_nm/pZOOM);
	TA.XTileSize = (XTile * TA.Pixel_size_nm)/(1000.*pZOOM); 	// Tile size in nm
	TA.YTileSize = (YTile * TA.Pixel_size_nm)/(1000.*pZOOM);	// Tile size in nm
	printf(" INIT PROG \u24EA TILE   : XTILE %d YTILE %d  size : XTILE:%6.3f µm YTILE %6.3f µm\n",  XTile, YTile, TA.XTileSize, TA.YTileSize);
	printf(" INIT PROG \u24EA RECONSTRUCTION in nm   : X %6.3f µm Y %6.3f µm\n",
			 TA.Nb_Cols_reconstruction*TA.Pixel_size_nm/1000., TA.Nb_Rows_reconstruction*TA.Pixel_size_nm/1000.);

	return (dimfit);
}

