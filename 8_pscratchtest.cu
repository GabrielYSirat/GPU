
#include <hip/hip_runtime.h>
#ifdef PSCRATCHTEST
			if ((ipixel[3] == PSFZoomo2) && (jpixel[3] == PSFZoomo2))
				if (image_to_scratchpad_offset_tile[iblockima]!= (DD.defaultoffsetedge)) {
					printf("\u2465 ipix jpix = iPSF jPSF zero ithreads %d 4:%d offset %d value %g \n", ithreads, pos_3,
							image_to_scratchpad_offset_tile[iblockima], *(Scratchpad + pos_3));
					printf("\u2465 values of pointer position pscratch iblockima %d: %p 2: %p 3: %p 4:%p \n", iblockima,
							*(pscratch_0 + iblockima), *(pscratch_1 + iblockima), *(pscratch_2 + iblockima),
							*(pscratch_3 + iblockima));
				 }

			if (!iblockima)
				if ((*pscratch_0[iblockima] + *pscratch_1[iblockima] + *pscratch_2[iblockima] + *pscratch_3[iblockima])
						> 0.01)// max PSF *5%
					printf(
							"\u2465 NEZ ithreads %d ipixel %d jpixel %d pos %d %d %d %d  *pscratch %6.3f %6.3f %6.3f %6.3f\n",
							ithreads, ipixel[3], jpixel[3], pos_0, pos_1, pos_2, pos_3, *pscratch_0[iblockima],
							*pscratch_1[iblockima], *pscratch_2[iblockima], *pscratch_3[iblockima]);

			__syncthreads();
#endif
