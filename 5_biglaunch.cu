#include "hip/hip_runtime.h"
/*
 * big.cu
 *
 *  Created on: Jul 6, 2017
 *      Author: gabriel
 */
#include "0_Mainparameters.h"
bool biglaunch() {

	verbosefile << "TESTS RETURN:  " << "initialization:  " << Stepdiag[0] << ";  pPSF:  " << Stepdiag[1];
	verbosefile << ";  distrib: " << Stepdiag[2] << endl;
	verbosefile << "TESTS RETURN: laser positions: " << Stepdiag[3] << "; ROI:  " << Stepdiag[4];
	verbosefile << "; microimages:  " << Stepdiag[5] << ";  Reconstruction:  " << Stepdiag[6] << endl;
	verbosefile << "MAIN PROGRAM  **********ready for GPU computation*****************" << endl;
	verbosefile << "******************************************************************" << endl << endl;

	verbosefile << "To be transferred to device: Number of Aggregates in x:" << tile.NbAggregx << " in y:"
			<< tile.NbAggregy;
	verbosefile << "  Number of Tiles per aggregates in x:" << tile.tileperaggregatex << " in y:"
			<< tile.tileperaggregatey << endl;
	verbosefile << "To be transferred to device: Number of Tiles in x:" << tile.NbTilex << " in y:" << tile.NbTiley
			<< endl;
	verbosefile << "To be transferred to device: Max number of laser position in Tile:" << tile.maxLaserintile
			<< " min  " << tile.minLaserintile << endl << endl;
	printf("**************HOST: PARAMETERS OF MEASUREMENT *******************\n"
			"Npixel %d pZOOM %d, pPSF %d\n", Npixel, pZOOM, pPSF);
	printf("pPSF %d XDistrib %d YDistrib %d\n", pPSF, XDistrib, YDistrib);
	printf("XSCRATCH %d YSCRATCH %d XTile %d YTile %d\n\n", XSCRATCH, YSCRATCH, XTile, YTile);
	printf("Number of pixels calculated in parallel %d Number of threads used %d loop on threads %d\n", NThreads,
	THREADSVAL, THreadsRatio);

	/************************* for GPU ********************/
	verbosefile << "MAIN PROGRAM  ********Prepare data for GPU computation**************" << endl;
	verbosefile << "******************************************************************" << endl;

	onhost.NbTilex = tile.NbTilex;
	onhost.NbTiley = tile.NbTiley;
	onhost.NbTileXY = tile.NbTileXY;
	onhost.NbTileXYD = tile.NbTileXYD;
	onhost.NbAggregx = tile.NbAggregx;
	onhost.NbAggregy = tile.NbAggregy;
	onhost.tileperaggregatex = tile.tileperaggregatex;
	onhost.tileperaggregatey = tile.tileperaggregatey;

	onhost.maxLaserintile = tile.maxLaserintile;
	onhost.blocks = tile.blocks;
	onhost.minLaserintile = tile.minLaserintile;
	onhost.Nb_LaserPositions = TA.Nb_LaserPositions;
	onhost.MaxPSF = MaxPSF;
	onhost.MaxRec = MaxRec;
	onhost.Maxmicroimages = Maxmicroimages;
	onhost.Maxdistrib = Maxdistrib;
	onhost.clockRate = clockRate;
	onhost.XTile = XTile;
	onhost.YTile = YTile;

	verbosefile << "HOST: \\u24EA  \n";
	for(int itile = 0; itile < tile.NbTileXYD; itile ++) {
		onhost.NbLaserpertile[itile] = tile.NbLaserpertile[itile];
		verbosefile << "tile n° " << itile << " #laser " << onhost.NbLaserpertile[itile] << " || ";
	}
	verbosefile << endl;
	onhost.imalimitpertile = onhost.Nb_LaserPositions - (onhost.NbTileXY - 1) * onhost.maxLaserintile;
	onhost.Bconstant = tile.Bconstant;
	verbosefile << "Number of laser positions " << onhost.Nb_LaserPositions << " number of tile XY " << onhost.NbTileXY
			<< " imalimitpertile " << onhost.imalimitpertile << " ima limite per tile " << onhost.maxLaserintile << endl;

	bool testbig = FALSE;

	dim3 dimBlock(tile.tileperaggregatex, tile.tileperaggregatey, Ndistrib);
	dim3 dimGrid(THREADSVAL, 1, 1);
	verbosefile << "dimBlock  x: " << dimBlock.x << " y: " << dimBlock.y << " z: " << dimBlock.z << "  ...  ";
	verbosefile << "dimGrid  x: " << dimGrid.x << " y: " << dimGrid.y << " z: " << dimGrid.z << endl << endl;

	verbosefile << "HOST: \\u24EA ************************BigLoop start   *******************************" << endl;
	verbosefile << "HOST: \\u24EA ***********************************************************************" << endl;

	int sharedsize = NIMAGESPARALLEL * sizeof(int) + ASCRATCH * sizeof(float) + ADistrib * sizeof(float);

	if (sharedsize > TA.sharedmemory) {
		verbosefile << "shared memory required is above the memory available" << sharedsize / 1024.0 << "KBytes" << endl;
		exit(1);
	} else
		verbosefile << "HOST: \\u24EA *** SHARED MEMORY SIZE " << sharedsize / 1024.0 << " KBytes" << endl;
	// Execute the Laser positions kernel
	verbosefile << "HOST: \\u24EA ************************BigLoop start   *******************************" << endl;
	verbosefile << "HOST: \\u24EA ***********************************************************************" << endl;
	cout << "HOST: \\u24EA ************************BigLoop start   *******************************" << endl;
	cout << "HOST: \\u24EA ***********************************************************************" << endl;
	BigLoop<<<dimBlock, dimGrid, sharedsize>>>(onhost);
	hipDeviceSynchronize();

	testbig = TRUE;
	return (testbig);
}

