#include "hip/hip_runtime.h"
/*
 * big.cu
 *
 *  Created on: Jul 6, 2017
 *      Author: gabriel
 */
#include "0_Mainparameters.h"
bool biglaunch() {
	onhost.NbTilex = tile.NbTilex;
	onhost.NbTiley = tile.NbTiley;
	onhost.NbTileXY = tile.NbTileXY;
	onhost.NbTileXYD = tile.NbTileXYD;
	onhost.NbAggregx = tile.NbAggregx;
	onhost.NbAggregy = tile.NbAggregy;
	onhost.tileperaggregatex = tile.tileperaggregatex;
	onhost.tileperaggregatey = tile.tileperaggregatey;

	onhost.maxLaserintile = tile.maxLaserintile;
	onhost.blocks = tile.blocks;
	onhost.minLaserintile = tile.minLaserintile;
	onhost.Nb_LaserPositions = TA.Nb_LaserPositions;
	onhost.MaxPSF = MaxPSF;
	onhost.MaxRec = MaxRec;
	onhost.Maxmicroimages = Maxmicroimages;
	onhost.Maxdistrib = Maxdistrib;
	onhost.clockRate = clockRate;
	onhost.XTile = XTile;
	onhost.YTile = YTile;
	onhost.lostpixelsdevice = lostpixels;
	onhost.lostlinesdevice = lostlines;
	onhost.defaultoffsetedge = tile.defaultoffsetedge;
	onhost.Bconstant = tile.Bconstant;

	verbosefile << "HOST: \u24EA  \n";
	for(int itile = 0; itile < tile.NbTileXYD; itile ++) {
		onhost.NbLaserpertile[itile] = tile.NbLaserpertile[itile];
		verbosefile << "tile n° " << itile << " #laser " << onhost.NbLaserpertile[itile] << " || ";
	}
	verbosefile << endl;


	/*********************************DOCUMENT ***************************************************************/
		verbosefile << "TESTS RETURN:  " << "initialization:  " << Stepdiag[0] << ";  pPSF:  " << Stepdiag[1];
		verbosefile << ";  distrib: " << Stepdiag[2] << endl;
		verbosefile << "TESTS RETURN: laser positions: " << Stepdiag[3] << "; ROI:  " << Stepdiag[4];
		verbosefile << "; microimages:  " << Stepdiag[5] << ";  Reconstruction:  " << Stepdiag[6] << endl;
		verbosefile << "MAIN PROGRAM  **********ready for GPU computation*****************" << endl;
		verbosefile << "******************************************************************" << endl << endl;

		verbosefile << "To be transferred to device: Number of Aggregates in x:" << tile.NbAggregx << " in y:"
				<< tile.NbAggregy;
		verbosefile << "  Number of Tiles per aggregates in x:" << tile.tileperaggregatex << " in y:"
				<< tile.tileperaggregatey << endl;
		verbosefile << "To be transferred to device: Number of Tiles in x:" << tile.NbTilex << " in y:" << tile.NbTiley
				<< endl;
		verbosefile << "To be transferred to device: Max number of laser position in Tile:" << tile.maxLaserintile
				<< " min  " << tile.minLaserintile << endl << endl;
		printf("\nHOST: \u24EA Npixel %d pZOOM %d, pPSF %d\n", Npixel, pZOOM, pPSF);
		printf("HOST: \u24EA pPSF %d XDistrib %d YDistrib %d\n", pPSF, XDistrib, YDistrib);
		printf("HOST: \u24EA XSCRATCH %d YSCRATCH %d XTile %d YTile %d\n", XSCRATCH, YSCRATCH, XTile, YTile);
		printf("HOST: \u24EA Number of pixels calculated in parallel %d Number of threads used %d loop on threads %d\n\n", NThreads,
		THREADSVAL, THreadsRatio);

		/************************* for GPU ********************/
		verbosefile << "MAIN PROGRAM  ********Prepare data for GPU computation**************" << endl;
		verbosefile << "******************************************************************" << endl;	bool testbig = FALSE;
		verbosefile << "Number of laser positions " << onhost.Nb_LaserPositions << " number of tile XY " << onhost.NbTileXY
				<<  " max number of lasers per tile " << onhost.maxLaserintile << endl;

		/********************************************Launch Main loop ***************************************/
	dim3 dimBlock(tile.tileperaggregatex, tile.tileperaggregatey, Ndistrib);
	dim3 dimGrid(THREADSVAL, 1, 1);
	int sharedsize = NIMAGESPARALLEL * sizeof(int) + ASCRATCH * sizeof(float) + ADistrib * sizeof(float);

	/*********************************DOCUMENT ***************************************************************/
	verbosefile << "dimBlock  x: " << dimBlock.x << " y: " << dimBlock.y << " z: " << dimBlock.z << "  ...  ";
	verbosefile << "dimGrid  x: " << dimGrid.x << " y: " << dimGrid.y << " z: " << dimGrid.z << endl << endl;
	verbosefile << "HOST: \u24EA ************************BigLoop start   *******************************" << endl;
	verbosefile << "HOST: \u24EA ***********************************************************************" << endl;
	cout << "HOST: \u24EA ************************BigLoop start   *******************************" << endl;
	cout << "HOST: \u24EA ***********************************************************************" << endl;



	if (sharedsize > TA.sharedmemory) {
		verbosefile << "shared memory required is above the memory available" << sharedsize / 1024.0 << "KBytes" << endl;
		exit(1);
	} else
		verbosefile << "HOST: \u24EA *** SHARED MEMORY SIZE " << sharedsize / 1024.0 << " KBytes" << endl;
	// Execute the Laser positions kernel
	BigLoop<<<dimBlock, dimGrid, sharedsize>>>(onhost);
	hipDeviceSynchronize();

	testbig = TRUE;
	return (testbig);
}

