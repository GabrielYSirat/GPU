#include "hip/hip_runtime.h"
/*
 * tileorganization.cu
 *
 *  Created on: Jul 3, 2017
 *      Author: gabriel
 */
#include "0_Mainparameters.h"

__managed__ int *image_to_scratchpad_offset = { 0 }, *valid_image = { 0 };
const char * MIintilefile = "results/C_microimagesintile.pgm", *NIintilefile =
		"results/C_microimagesintile2.pgm";
float * reorganized_data;
int fullnumberoflasers, datafullsize;
int organization_x[16] = { 0, 1, 2, 3, 2, 2, 3, 3, 4, 3, 3 };
int organization_y[16] = { 0, 1, 1, 1, 2, 2, 2, 2, 2, 3, 3 };

bool tileorganization(void) {
	bool Lasertile = TRUE;
	int tilex, tiley, tilenumber, ilasertile, nblasertile;
	filename = resourcesdirectory + "rec_image.xml";

	/************Tiles and aggregates*******************/
	int recdeftile0x = ceil((float) TA.Nb_Cols_reconstruction / XTile);
	int recdeftile0y = ceil((float) TA.Nb_Rows_reconstruction / YTile);
	int laserdeftile0x = CEILING_POS((float )pZOOM * (TA.AmaxLaserx - TA.AminLaserx) / XTile);
	int laserdeftile0y = CEILING_POS((float )pZOOM * (TA.AmaxLasery - TA.AminLasery) / YTile);
	tile.NbTile0x = max(laserdeftile0x, recdeftile0x);
	tile.NbTile0y = max(laserdeftile0y, recdeftile0y);
	if (((TA.AmaxLaserx - TA.AminLaserx) > TA.Nb_Cols_reconstruction)
			|| ((TA.AmaxLasery - TA.AminLasery) > TA.Nb_Rows_reconstruction))
		printf(" INIT PROG \u24FA \u26A0 tiles size bigger then XML parameters!");

	verbosefile << " INIT PROG \u24FA Amax x " << TA.AmaxLaserx << "  Amax y: " << TA.AmaxLasery
			<< "  Amin x: " << TA.AminLaserx << "  Amin y: " << TA.AminLasery << endl;
	verbosefile << " INIT PROG \u24FA recdeftile0x: " << recdeftile0x << " y: " << recdeftile0y;
	verbosefile << " laserdeftile0x: " << laserdeftile0x << " y: " << laserdeftile0y << endl;
	verbosefile << " INIT PROG \u24FA Min (not final!!) Number of tiles x: " << tile.NbTile0x << " y: "
			<< tile.NbTile0y << endl;

	/*************************Aggregates organization depending on MP*******/
	TA.MP_perdistrib = TA.MP / Ndistrib;
	printf("\n INIT PROG \u24FA Total number of MP per distribution %d  ", TA.MP_perdistrib);
	printf("  organized as x:%d,  y:%d \n", organization_x[TA.MP_perdistrib],
			organization_y[TA.MP_perdistrib]);

	/************************Aggregates********************************************/
	tile.NbAggregx = ceil((float) tile.NbTile0x / organization_x[TA.MP_perdistrib]);
	tile.NbAggregy = ceil((float) tile.NbTile0y / organization_y[TA.MP_perdistrib]);
	printf(" INIT PROG \u24FA Number of aggregates x:%d y:%d  \n", tile.NbAggregx, tile.NbAggregy);

	/**********************************Tiles****************************************/
	if (tile.NbAggregx == 1) {
		tile.NbTilex = tile.NbTile0x;
		tile.tileperaggregatex = tile.NbTile0x;
	} else {
		tile.NbTilex = tile.NbAggregx * organization_x[TA.MP_perdistrib];
		tile.tileperaggregatex = organization_x[TA.MP_perdistrib];
	}

	if (tile.NbAggregy == 1) {
		tile.NbTiley = tile.NbTile0y;
		tile.tileperaggregatey = tile.NbTile0y;
	} else {
		tile.NbTiley = tile.NbAggregy * organization_y[TA.MP_perdistrib];
		tile.tileperaggregatey = organization_y[TA.MP_perdistrib];
	}

	tile.NbTileXY = tile.NbTilex * tile.NbTiley;
	tile.NbTileXYD = tile.NbTilex * tile.NbTiley * Ndistrib;
	fullnumberoflasers = tile.maxLaserintile * tile.NbTileXYD;
	datafullsize = fullnumberoflasers * NThreads;

	if (NOVERBOSE)
		printf("TEST full number of lasers %d tile.maxLaserintile %d tile.NbTileXYD %d\n", fullnumberoflasers,
				tile.maxLaserintile, tile.NbTileXYD);

	hipMallocManaged(&image_to_scratchpad_offset, fullnumberoflasers * sizeof(int));
	hipMallocManaged(&valid_image, fullnumberoflasers * sizeof(int));
	/** initialization of the offset to the edge of the scratchpad for all images
	 *
	 */

	int defaultoffsetcenter = dySCR / 2 * XSCRATCH + dxSCR / 2 + lostpixels;
	int defaultoffsetedge = defaultoffsetcenter - (XSCRATCH + 1) * PSFZoomo2;
	verbosefile << " initialization of offset values for " << Ndistrib << " distributions and "
			<< tile.maxLaserintile << " lasers per distribution" << " full numberof lasers "
			<< fullnumberoflasers << endl;
	verbosefile << " default offset value is at center: " << defaultoffsetcenter << "  at edge "
			<< defaultoffsetedge << endl;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
		for (int iLaser = 0; iLaser < fullnumberoflasers; iLaser++)
			*(image_to_scratchpad_offset + iLaser) = defaultoffsetedge;

	/** FUTURE: In the real application the reconstruction
	 * is created by the program and not read from a file
	 * in this case the size data will be consistent by design
	 */
	// can be improved depending on the ratio between TILE0 and TILE: minor
	TA.Nb_Cols_reconstruction = tile.NbTilex * XTile;
	TA.Nb_Rows_reconstruction = tile.NbTiley * YTile;
	tile.startxdomain = TA.AminLaserx; //floor(pZOOM*((AminLaserx + AmaxLaserx)/2 - (tile.NbTilex * XTile)/2));
	tile.startydomain = TA.AminLasery; //floor(pZOOM*((AminLasery + AmaxLasery)/2 - (tile.NbTiley * YTile)/2));
	TA.reconstruction_size = TA.Nb_Rows_reconstruction * TA.Nb_Cols_reconstruction;
	printf(" INIT PROG \u24FA Final number of tiles x: %d y: %d distrib %d  \n\n", tile.NbTilex, tile.NbTiley,
			Ndistrib);
	verbosefile << " INIT PROG \u24FA Reconstruction size x: " << TA.Nb_Cols_reconstruction << " y: "
			<< TA.Nb_Rows_reconstruction << endl;
	verbosefile << "INIT PROG \u24FA NbTileXY " << tile.NbTileXY << " NbTileXYD " << tile.NbTileXYD;
	verbosefile << " start x " << tile.startxdomain << " y " << tile.startydomain << " MinLaser x "
			<< TA.AminLaserx << " MinLaser y " << TA.AminLasery;
	verbosefile << " in REC pixels x: " << tile.startxdomain * pZOOM << " y " << tile.startydomain * pZOOM
			<< endl;

	nblasertile = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		for (int iLaser = nblasertile; iLaser < nblasertile + tile.Nblaserperdistribution[idistrib];
				iLaser++) {
			// position in tiles, tilex and tiley and overall tile number (including distrib)
			tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startxdomain) / XTile;
			tiley = pZOOM * (*(PosLasery + iLaser) - tile.startydomain) / YTile;
			tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
			float deltilex = *(PosLaserx + iLaser) * pZOOM - tilex * XTile;
			float deltiley = *(PosLasery + iLaser) * pZOOM - tiley * YTile;
			float delscratchx = deltilex + (XSCRATCH - XTile) / 2;  // XSCRATCH and XTile are odd
			float delscratchy = deltiley + (YSCRATCH - YTile) / 2;  // ySCRATCH and YTile are odd
			tile.posintile[iLaser] = tile.NbLaserpertile[tilenumber]++;
				ilasertile = tilenumber * tile.maxLaserintile + tile.posintile[iLaser];
			valid_image[ilasertile] = 1;

			if (NOVERBOSE)
			printf("TILE ORG \u24FA POS IN TILE: iLaser %d,tilenumber %d tile.posintile[iLaser]  %d A ilasertile %d valid %d\n",
					iLaser, tilenumber, tile.posintile[iLaser], ilasertile, valid_image[ilasertile]);
			image_to_scratchpad_offset[ilasertile] = *(offsetFULL + iLaser);
			tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[tilenumber]); // acquiring the max value per tile

			// where will be this microimage in the corresponding tile if posintile is 31, and tilenumber is 8?
			//  this microimage is the microimage with index 31 (the indexes begin at 0) of tile of index 8
			// add 1 - to go to 32 - to NbLaserpertile, because we added an image

			if (NOVERBOSE)
				printf("TILE ORG \u24FA POS IN SCRATCH: numeral %d laser pos in x %f in y: %f  tile x: %d y: %d \n"
								"TILE ORG \u24FA POS IN SCRATCH: deltile x: %f and y %f del scratch x:%f y:%f\n"
								"TILE ORG \u24FA POS IN SCRATCH: ilasertile %d SCRATCH POSITION %d\n"
								"********************ilasertile %d offset scratchpad interaction****************** %d\n",
						iLaser, *(PosLaserx + iLaser), *(PosLasery + iLaser), tilex, tiley, deltilex,
						deltiley, delscratchx, delscratchy, ilasertile,
						image_to_scratchpad_offset[ilasertile], ilasertile,
						image_to_scratchpad_offset[ilasertile]);

			if (NOVERBOSE)
				printf("TILE ORG \u24FA POS IN SCRATCH: image number %d tilenumber %d position in tile %d\n",
						iLaser, tilenumber, tile.posintile[iLaser]);
		}

		verbosefile << " TILE ORG \u24FA  idistrib n°" << idistrib
				<< " number of laser positions in tile in distribution "
				<< tile.Nblaserperdistribution[idistrib] << " number of tiles in distribution "
				<< tile.NbTileXY << endl;
		int it0 = tile.NbTilex * tile.NbTiley * idistrib;
		for (int it = it0; it < it0 + tile.NbTilex * tile.NbTiley; it++)
			verbosefile << " \u24FA tile " << it << ": #lasers " << tile.NbLaserpertile[it];
		verbosefile << endl;
		nblasertile += tile.Nblaserperdistribution[idistrib];
	}
	verbosefile << "TILE ORG \u24FA  nblasertile " << nblasertile << endl;
	verbosefile << endl << "images offset" << endl << endl;
	for (int iii = 0; iii < tile.maxLaserintile * tile.NbTileXYD; iii++)
		if (image_to_scratchpad_offset[iii] != defaultoffsetedge)
			verbosefile << " position " << image_to_scratchpad_offset[iii] << " @ " << iii << " | ";
	verbosefile << endl;

	for (int it1 = 0; it1 < tile.NbTileXYD; it1++) {
		if (NOVERBOSE)
			printf(
					"TILE ORG \u24FA Tile number %d tile in x %d tile in y %d distrib %d number of microimages %d\n",
					it1, it1 % (Ndistrib * tile.NbTiley), (it1 / tile.NbTilex) % Ndistrib,
					it1 / (tile.NbTilex * tile.NbTiley), tile.NbLaserpertile[it1]);

		tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[it1]); // acquiring the max value per tile
		tile.minLaserintile = min(tile.minLaserintile, tile.NbLaserpertile[it1]); // acquiring the min value per tile
	}

	verbosefile << " TILE ORG \u24FA Max  " << tile.maxLaserintile << " and Min " << tile.minLaserintile;
	tile.maxLaserintile = CEILING_POS(((float)tile.maxLaserintile)/NIMAGESPARALLEL) * NIMAGESPARALLEL;
	tile.blocks = tile.maxLaserintile / NIMAGESPARALLEL;
	// rounded to next multiple of NIMAGESPARALLEL
	verbosefile << " Max Laser in tile rounded to next multiple of NIMAGESPARALLEL  .." << tile.maxLaserintile
			<< " \n";
	return (Lasertile);
}

bool initializesimusData(void) {
// Initialize new simus and Data
	fullnumberoflasers = tile.maxLaserintile * tile.NbTileXYD;
	datafullsize = fullnumberoflasers * NThreads;
	verbosefile << "TILE ORG \u24FB size simus " << datafullsize << " AminLaserx " << TA.AminLaserx
			<< " AminLasery " << TA.AminLasery << endl;
	hipMallocManaged(&new_simus, datafullsize * sizeof(float));
	hipMallocManaged(&Data, datafullsize * sizeof(float));
	hipMallocManaged(&Rfactor, datafullsize * sizeof(float));

	for (int ii = 0; ii < datafullsize; ii++) {
		new_simus[ii] = 0.0f;
		Data[ii] = 0.0f;
		Rfactor[ii] = 0.0f;
	}
	hipMallocManaged(&distribvalidGPU, TA.MP * PSFZOOMSQUARE * sizeof(float));
	for (int itemp = 0; itemp < Ndistrib * PSFZOOMSQUARE; itemp++)
		*(distribvalidGPU + itemp) = 0.0;
	return (TRUE);
}

bool microimagesintile(void) {
	float ratioMI = 1.0 / (Maxmicroimages - Minmicroimages);
	bool micimintile = FALSE;
	reorganized_data = (float *) calloc(fullnumberoflasers * PixZoomSquare, sizeof(float));

	unsigned char *i_data = (unsigned char *) calloc(PixZoomSquare * tile.NbTileXYD * tile.maxLaserintile,
			sizeof(unsigned char));
	unsigned char *j_data = (unsigned char *) calloc(PixZoomSquare * tile.NbTileXYD * tile.maxLaserintile,
			sizeof(unsigned char));
	verbosefile << "TILE ORG \u24FB Max Laser in tile rounded to multiple NIMAGESPARALLEL  .. "
			<< tile.maxLaserintile;
	verbosefile << endl << "TILE ORG \u24FB Max and min microimages " << Maxmicroimages << " "
			<< Minmicroimages << endl;

	float Maxdata = 0.0f;
	for (int idistrib = 0, disdelta = 0; idistrib < Ndistrib;
			idistrib++, disdelta += tile.Nblaserperdistribution[idistrib])
		for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
			int tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startxdomain) / XTile;
			int tiley = pZOOM * (*(PosLasery + iLaser) - tile.startydomain) / YTile;
			int tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
			int ilasertile = tilenumber * tile.maxLaserintile + tile.posintile[iLaser];
			verbosefile << "TILE ORG \u24FA idistrib " << idistrib << " iLaser " << iLaser << " iLasertile "
					<< ilasertile << " tilenumber " << tilenumber << " tilex " << tilex << " tiley " << tiley
					<< " tileblock " << disdelta << endl;
			for (int ipix = 0; ipix < PixZoomSquare; ipix++) { // copy microimage to its position in the Data
				*(reorganized_data + ilasertile * PixZoomSquare + ipix) = *(zoomed_microimages
						+ iLaser * PixZoomSquare + ipix);
				*(Data + ilasertile * NThreads + ipix) = *(reorganized_data + ilasertile * PixZoomSquare
						+ ipix);
				i_data[ilasertile * PixZoomSquare + ipix] = 255.0
						* (*(reorganized_data + ilasertile * PixZoomSquare + ipix) - Minmicroimages)
						* ratioMI;
				Maxdata = max(Maxdata, i_data[ilasertile * PixZoomSquare + ipix]);
			}
		}
	verbosefile << " TILE ORG \u24FA Maxdata " << Maxdata << " Nbtile XY " << tile.NbTileXY << " NbTile XYD "
			<< tile.NbTileXYD << " Laserintile " << tile.maxLaserintile << endl;
	T4Dto2D(j_data, i_data, tile.NbTileXYD, tile.maxLaserintile, PixZoom, PixZoom);
	verbosefile << "HOST: \u24FB DEVICE TEST in biginspect.cu: Path to calculated new simulations "
			<< MIintilefile << " .....\n";
	sdkSavePGM(MIintilefile, i_data, PixZoom, tile.maxLaserintile * tile.NbTileXYD * PixZoom);
	sdkSavePGM(NIintilefile, j_data, tile.maxLaserintile * PixZoom, tile.NbTileXYD * PixZoom);

	return (micimintile);
}
