
#include <hip/hip_runtime.h>
#ifdef STARTDEVICE
	if (!iprint) { 	// the condition is required to have it printed once
		printf("\nDEVICE: ************************** START *****************************\n");
		printf("DEVICE: \u23f1 Time: Start: sec %f\n", (float) time_start / CLOCKS_PER_SEC);
		printf("DEVICE: \u2460****************PARAMETERS OF MEASUREMENT *******************\n");
		printf("DEVICE: \u2460 PARAMETERS  NThreads %d Npixel %d pZOOM %d, pPSF %d\n", NThreads, Npixel, pZOOM, pPSF);
		printf("DEVICE: \u2460 PARAMETERS dimBlock  x: %d y: %d z: %d   ...   ", blockDim.x, blockDim.y, blockDim.z);
		printf("dimGrid  x: %d y: %d z: %d\n", gridDim.x, gridDim.y, gridDim.z);
		printf("DEVICE: \u2460 PARAMETERS pPSF %d XDistrib %d YDistrib %d ADistrib %d\n", pPSF, XDistrib, YDistrib, ADistrib);
		printf("DEVICE: \u2460 PARAMETERS XSCRATCH %d YSCRATCH %d XTILE %d YTILE %d\n", XSCRATCH, YSCRATCH, XTile, YTile);
		printf("DEVICE: \u2460 PARAMETERS Number of pixels calculated in parallel %d Number of threads used"
				" %d loop on threads %d\n", NThreads, THREADSVAL, THreadsRatio);
		printf("DEVICE: \u2460  TILES: XSCRATCH %d, YSCRATCH %d  ", XSCRATCH, YSCRATCH);
		printf("XTILE %d, YTILE %d\n", XTile, YTile);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Aggregates in x: %d in y:%d\n", DD.NbAggregx,
				DD.NbAggregy);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Tiles per aggregates in x: %d in y:%d\n",
				DD.tileperaggregatex, DD.tileperaggregatey);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Number of Tiles in x: %d in y:%d\n", DD.NbTilex, DD.NbTiley);
		printf("DEVICE: \u2460  TILES & AGGREGATES: Max number of laser position in Tile: %d min value:%d Number of blocks %d\n",
				DD.maxLaserintile, DD.minLaserintile, DD.blocks);
		if (!iprint)
			printf("\u2460*******************************PARAMETERS OF MEASUREMENT ****************\n");
	}
	__syncthreads();
	
	DD.step++;
	if (!itb) timer = clock64();
		if (!iprint)  	// the condition is required to have it printed once
			printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING ** processing  %g from start  %g  total %g \n\n",
					DD.step, (float) (timer - time_start) / CLOCKS_PER_SEC,
					(float) (  time_start - time_init) / CLOCKS_PER_SEC,
					(float) (timer - time_init) / CLOCKS_PER_SEC);
		__syncthreads();
			if (!itb) time_start = clock64();
		__syncthreads();
	
	
#endif


