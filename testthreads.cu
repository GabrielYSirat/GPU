
#include <hip/hip_runtime.h>
#ifdef TESTTHREADS
	if (!iprint)
		printf("\n\u2461*******************************DEVICE:  THREADS *********************\n");
	for (int apix = 0; apix < THreadsRatio; apix++) {
		int tmpi = (ithreads + apix * THREADSVAL);
		if(!itb)
			if ((ithreads == 0)||(ithreads == (THREADSVAL-1)) ||(tmpi == PixZoomSquare-1) ||(tmpi == PixZoomSquare))
				printf("DEVICE: \u2461 THREAD : apix %d  ipixel %d, jpixel %d  valid %d distribpos0 %d\n",
						 apix,  ipixel[apix], jpixel[apix], valid_pixel[apix], distribpos0[apix]);
	if(tmpi == (PixZoomSquare-1) || tmpi == PixZoomSquare)
				printf("DEVICE: \u2461 THREAD : apix %d ipixel %d, jpixel %d  valid %d distribpos0 %d\n",
						 apix, ipixel[apix], jpixel[apix], valid_pixel[apix], distribpos0[apix]);
				}
	for (int apix = 0; apix < THreadsRatio; apix++)
	if(!ipixel[apix] && !jpixel[apix]){
				printf("DEVICE CENTER: \u2461 THREAD : apix %d tmpi %d ipixel %d, jpixel %d  valid %d distribpos0 %d center %d\n",
						apix, tmpi[apix], ipixel[apix], jpixel[apix], valid_pixel[apix], distribpos0[apix], center_distrib);
}
	if (!iprint)
		printf("\u2461 **********************************DEVICE:  THREADS  ********************\n\n");
	__syncthreads();
#endif
