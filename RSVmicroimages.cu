#include "hip/hip_runtime.h"
/*
 * microimagesRSV.cu
 *
 *  Created on: May 3, 2017
 *      Author: gabriel
 */

#include "NewLoop.h"

double * double_microimages;
std::string endMI = ".bin";
float Maxmicroimages = 0.0f, Summicroimages = 0.0f, Minmicroimages = 1.e20;

void readstoremicroimages(void) {
	char * memblock;
	long size;
	const char * MIRawfile = "results/MIRawfile.pgm";

	// buffer allocation, buffer in double for original data, buffer in float for working (to go to FP16) character for display
	double_microimages = (double *) calloc(TA.Nb_LaserPositions * PixSquare, sizeof(double));
	hipMallocManaged(&original_microimages, TA.Nb_LaserPositions * PixSquare * sizeof(float));
	hipMallocManaged(&zoomed_microimages, TA.Nb_LaserPositions * PixZoomSquare * sizeof(float));
	hipMallocManaged(&MIintile, tile.NbTile * tile.maxLaserintile * PixZoomSquare * sizeof(float));
	unsigned char *i_MIraw = (unsigned char*) calloc(Ndistrib*PixSquare * tile.maxlaserperdistribution, sizeof(char));

	printf("MICROIMAGES \u2464 Total number of images for all distributions %d\n", TA.Nb_LaserPositions);
	int numberofpixels = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		;
		std::string MIraw = resourcesdirectory + MIFILE + std::to_string(idistrib + 1) + endMI;
		printf("MICROIMAGES \u2464 function read: distribution n°%d Path to distrib original .....\n %s \n", idistrib,
				MIraw.c_str());

		//read distrib bin file
		std::ifstream MIrawfile(MIraw.c_str(), ios::in | ios::binary | ios::ate);
		size = (MIrawfile.tellg()); 	// the data is stored in doubles of 8 bytes in the file
		size -= byte_skipped;  				// removes the 4 first bytes, Why??
		std::cout << "MICROIMAGES \u2464 function read: distrib n°" << idistrib << " number laser positions "
				<< tile.Nblaserperdistribution[idistrib] << " size microimages = " << size << endl;
		memblock = new char[size];
		MIrawfile.seekg(byte_skipped, ios::beg); // byte_skipped first bytes are offset
		MIrawfile.read(memblock, size);
		MIrawfile.close();

		double_microimages = (double*) memblock; //reinterpret the chars stored in the file as double
		printf("number of images %d Number of pixels %d \n", tile.Nblaserperdistribution[idistrib],
				tile.Nblaserperdistribution[idistrib] * PixSquare);
		for (int i = 0; i < tile.Nblaserperdistribution[idistrib] * PixSquare; i++) {
			*(original_microimages + i + numberofpixels) = *(double_microimages + i);			// change to float
			Summicroimages += original_microimages[i];
			Maxmicroimages = max(Maxmicroimages, *(original_microimages + i));
			Minmicroimages = min(Minmicroimages, *(original_microimages + i));
		}
		numberofpixels += tile.Nblaserperdistribution[idistrib] * PixSquare;
		printf("MICROIMAGES \u2464 original on host: Average  %f max microimages %f \n",
				Summicroimages / numberofpixels, Maxmicroimages);

	}
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
		for (int ilaser = 0; ilaser < tile.Nblaserperdistribution[idistrib]; ilaser++)
			for (int xpix = 0; xpix < Npixel; xpix++)
				for (int ypix = 0; ypix < Npixel; ypix++) {
					int itemp = xpix + Npixel * ypix + PixSquare * ilaser + idistrib*PixSquare * tile.maxlaserperdistribution;
					i_MIraw[itemp] = 255.0 * (original_microimages[itemp] - Minmicroimages)
							/ (Maxmicroimages - Minmicroimages);
				}
	printf("MICROIMAGES \u2464 host: Path to microimages original %s .....\n", MIRawfile);

	sdkSavePGM(MIRawfile, i_MIraw,  Npixel * Ndistrib, tile.maxlaserperdistribution * Npixel);

	free(i_MIraw);

}

bool validatemicroimages_control(void) {
	bool testmicroimages;
	double Sum3microimages = 0, max3microimages = 0;
	double Sum4microimages = 0, max4microimages = 0;
	const char * MIValfile = "results/MIVALfile.pgm";
	const char * MIzoomfile = "results/MIZOOMfile.pgm";

	// write microimages in memory and validate
	hipMallocManaged(&valmicroimages, (TA.Nb_LaserPositions * PixSquare) * sizeof(float));
	unsigned char *i_MIVal = (unsigned char*) calloc(TA.Nb_LaserPositions * PixSquare, sizeof(char));
	unsigned char *i_MIzoom = (unsigned char*) calloc(TA.Nb_LaserPositions * PixZoom * PixZoom, sizeof(char));

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the microimages kernel
	validate_microimages<<<dimBlock, dimGrid,  0>>>(TA.Nb_LaserPositions);
	hipDeviceSynchronize();
	for (int imicroimages = 0; imicroimages < (TA.Nb_LaserPositions * PixSquare); imicroimages++) {
		Sum3microimages += *(valmicroimages + imicroimages);
		max3microimages = max(max3microimages, *(valmicroimages + imicroimages));
	}
	printf("MICROIMAGES \u2464 Copy from device: Average  %f max3microimages %f \n",
			Sum3microimages / (TA.Nb_LaserPositions * PixSquare), max3microimages);

	for (int imicroimages = 0; imicroimages < (TA.Nb_LaserPositions * PixZoom * PixZoom); imicroimages++) {
		Sum4microimages += *(zoomed_microimages + imicroimages);
		max4microimages = max(max4microimages, *(zoomed_microimages + imicroimages));
	}
	printf("MICROIMAGES \u2464 Copy from device: zoomed image Average  %f max3microimages %f \n",
			Sum4microimages / (TA.Nb_LaserPositions * PixZoom * PixZoom), max4microimages);

	// write microimages image validation to disk
	/////////////////////////////////

	printf("MICROIMAGES \u2464 Comparing files ... ");
	testmicroimages = compareData(valmicroimages, original_microimages,
			TA.Nb_Cols_microimages * TA.Nb_Rows_microimages * Ndistrib,
			MAX_EPSILON_ERROR, 0.15f);

	for (int jmicroimages = 0; jmicroimages < (TA.Nb_LaserPositions * PixSquare); jmicroimages++) {
		Sumdel[4] += fabsf(*(valmicroimages + jmicroimages) - *(original_microimages + jmicroimages));
	}
	printf("Sumdel[4] %f  ", Sumdel[4]);
	cout << "testmicroimages = " << testmicroimages << "\n";

	// write microimages copy to disk
	/////////////////////////////////
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
		for (int ilaser = 0; ilaser < tile.Nblaserperdistribution[idistrib]; ilaser++)
			for (int xpix = 0; xpix < Npixel; xpix++)
				for (int ypix = 0; ypix < Npixel; ypix++) {
					int itemp = xpix + Npixel * ypix + PixSquare * ilaser
							+ idistrib * PixSquare * tile.maxlaserperdistribution;
					i_MIVal[itemp] = 255.0 * (valmicroimages[itemp] - Minmicroimages)
							/ (Maxmicroimages - Minmicroimages);
					for (int xpixzoom = 0; xpixzoom < PixZoom; xpixzoom++)
						for (int ypixzoom = 0; ypixzoom < PixZoom; ypixzoom++) {
							int itemp = xpixzoom + PixZoom * ypixzoom + PixZoomSquare * ilaser
									+ idistrib * PixZoomSquare * tile.maxlaserperdistribution;
							i_MIzoom[itemp] = 255.0 * (zoomed_microimages[itemp] - Minmicroimages)
									/ (Maxmicroimages - Minmicroimages);
						}
				}
	printf("MICROIMAGES \u2464 host: Path to microimages copy %s .....\n", MIValfile);

	sdkSavePGM(MIValfile, i_MIVal,  Npixel * Ndistrib, tile.maxlaserperdistribution * Npixel);
	sdkSavePGM(MIzoomfile, i_MIzoom, PixZoom * Ndistrib, tile.maxlaserperdistribution * PixZoom);

	free(i_MIVal);	free(i_MIzoom);
	hipFree(valmicroimages);
	return (testmicroimages);
}

