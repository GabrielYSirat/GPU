#include "hip/hip_runtime.h"
/*
 * microimagesRSV.cu
 *
 *  Created on: May 3, 2017
 *      Author: gabriel
 */

#include "NewLoop.h"
void readstoremicroimages(void) {
	char * memblock;
	long size;
	XMLDocument doc;
	float Maxmicroimages = 0.0f, Summicroimages = 0.0f;

	string MIraw = resourcesdirectory + "MI_created_11_11.raw";
	const char * MIRawfile   = "results/MIRawfile.pgm";

	printf("MICROIMAGES \u2464 ******************Read raw microimages **************\n");

	// buffer allocation, buffer in double for original data, buffer in float for working (to go to FP16) character for display
	hipMallocManaged(&double_microimages, TA.Nb_LaserPositions * Npixel * Npixel * sizeof(double));
	hipMallocManaged(&original_microimages, TA.Nb_LaserPositions * Npixel * Npixel * sizeof(float));
	hipMallocManaged(&zoomed_microimages, TA.Nb_LaserPositions * PixZoomSquare * sizeof(float));
	hipMallocManaged(&MIintile, tile.NbTile * tile.maxLaserintile * PixZoomSquare * sizeof(float));
	unsigned char *i_MIraw = (unsigned char*) calloc( TA.Nb_LaserPositions * Npixel * Npixel, sizeof(char));

	//read distrib bin file
	std::ifstream MIrawfile(MIraw.c_str(), ios::in | ios::binary | ios::ate);
	size = (MIrawfile.tellg()); 	// the data is stored in doubles of 8 bytes in the file
	size -= byte_skipped;  				// removes the 4 first bytes, Why??
	std::cout << "MICROIMAGES \u2464 function read: size microimages = " << size << endl;
	memblock = new char[size];
	MIrawfile.seekg(byte_skipped, ios::beg); // 4 first bytes are offset
	MIrawfile.read(memblock, size);
	MIrawfile.close();

	double_microimages = (double*) memblock; //reinterpret the chars stored in the file as double

	for (int i = 0; i < TA.Nb_LaserPositions * Npixel * Npixel; i++) {
		*(original_microimages + i) = *(double_microimages + i);			// change to float
		Summicroimages += original_microimages[i];
		Maxmicroimages = max(Maxmicroimages, *(original_microimages + i));
	}
	printf("MICROIMAGES \u2464 function read: Path to distrib original %s .....\n", MIRawfile);
	printf("MICROIMAGES \u2464 original on host: Average  %f max microimages %f \n", Summicroimages / (TA.Nb_LaserPositions * Npixel * Npixel),
			Maxmicroimages);


	// write microimages copy to disk
	/////////////////////////////////
	for (int i = 0; i < TA.Nb_LaserPositions * Npixel * Npixel; i++)
		i_MIraw[i] = 255.0 * original_microimages[i] / Maxmicroimages;			// image value
	printf("MICROIMAGES \u2464 host: Path to microimages original %s .....\n", MIRawfile);

	sdkSavePGM(MIRawfile, i_MIraw, TA.Nb_LaserPositions * Npixel,Npixel);

	free(i_MIraw);

}

bool validatemicroimages_control(void) {
	bool testmicroimages;
	double Sum3microimages = 0, max3microimages = 0;
	double Sum4microimages = 0, max4microimages = 0;
	const char * MIValfile   = "results/MIVALfile.pgm";
	const char * MIzoomfile   = "results/MIZOOMfile.pgm";

	// write microimages in memory and validate
	hipMallocManaged(&valmicroimages, (TA.Nb_LaserPositions * Npixel * Npixel) * sizeof(float));
	unsigned char *i_MIVal = (unsigned char*) calloc( TA.Nb_LaserPositions * Npixel * Npixel, sizeof(char));
	unsigned char *i_MIzoom = (unsigned char*) calloc( TA.Nb_LaserPositions * PixZoom * PixZoom, sizeof(char));

	dim3 dimBlock(Npixel, Npixel, 1);
	dim3 dimGrid(pZOOM, pZOOM, 1);
	// Execute the microimages kernel
	validate_microimages<<<dimGrid, dimBlock, 0>>>( TA.Nb_LaserPositions);
	hipDeviceSynchronize();
	for (int imicroimages = 0; imicroimages < (TA.Nb_LaserPositions * Npixel * Npixel); imicroimages++){
				Sum3microimages += *(valmicroimages + imicroimages);
				max3microimages = max(max3microimages, *(valmicroimages + imicroimages));
			}
	printf("MICROIMAGES \u2464 Copy from device: Average  %f max3microimages %f \n", Sum3microimages / (TA.Nb_LaserPositions * Npixel * Npixel),
			max3microimages);

	for (int imicroimages = 0; imicroimages < (TA.Nb_LaserPositions * PixZoom * PixZoom); imicroimages++){
				Sum4microimages += *(zoomed_microimages + imicroimages);
				max4microimages = max(max4microimages, *(zoomed_microimages + imicroimages));
			}
	printf("MICROIMAGES \u2464 Copy from device: zoomed image Average  %f max3microimages %f \n", Sum4microimages / (TA.Nb_LaserPositions * PixZoom * PixZoom),
			max4microimages);

	// write microimages image validation to disk
	/////////////////////////////////

	printf("MICROIMAGES \u2464 Comparing files ... ");
	testmicroimages = compareData(valmicroimages, original_microimages,
			TA.Nb_Cols_microimages * TA.Nb_Rows_microimages * Ndistrib,
			MAX_EPSILON_ERROR, 0.15f);

	for (int jmicroimages = 0; jmicroimages < (TA.Nb_LaserPositions * Npixel * Npixel); jmicroimages++) {
		Sumdel[4] += fabsf(*(valmicroimages + jmicroimages) - *(original_microimages + jmicroimages));
	}
	printf("Sumdel[4] %f  ", Sumdel[4]);
	cout << "testmicroimages = " << testmicroimages << "\n";

	// write microimages copy to disk
	/////////////////////////////////
	for (int i = 0; i < TA.Nb_LaserPositions * Npixel * Npixel; i++)
		i_MIVal[i] = 255.0 * valmicroimages[i] / max3microimages;			// image value
	printf("MICROIMAGES \u2464 host: Path to microimages copy %s .....\n", MIValfile);

	sdkSavePGM(MIValfile, i_MIVal,  TA.Nb_LaserPositions * Npixel,Npixel);

	free(i_MIVal);
	// write zoomed microimages  to disk
	/////////////////////////////////
	for (int i = 0; i < TA.Nb_LaserPositions * PixZoomSquare; i++)
		i_MIzoom[i] = 255.0 * zoomed_microimages[i] / max3microimages;			// correct sum image value
	printf("MICROIMAGES \u2464 host: Path to microimages zoomed %s .....\n", MIValfile);

	sdkSavePGM(MIzoomfile, i_MIzoom, TA.Nb_LaserPositions * PixZoom,PixZoom);
	free(i_MIzoom);


	hipFree(valmicroimages);

	return (testmicroimages);
}

