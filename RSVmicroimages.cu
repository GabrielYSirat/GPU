#include "hip/hip_runtime.h"
/*
 * microimagesRSV.cu
 *
 *  Created on: May 3, 2017
 *      Author: gabriel
 */

#include "NewLoop.h"

std::string MIFILE = "lambda_488/Measure/T_0/Z_0/DATA";
std::string endMI = ".bin";
float Maxmicroimages = 0.0f, Summicroimages = 0.0f;

void readstoremicroimages(void) {
	char * memblock;
	long size;
	const char * MIRawfile = "results/MIRawfile.pgm";

	printf("MICROIMAGES \u2464 ******************Read raw microimages **************\n");

	// buffer allocation, buffer in double for original data, buffer in float for working (to go to FP16) character for display
	hipMallocManaged(&double_microimages, TA.Nb_LaserPositions * PixSquare * sizeof(double));
	hipMallocManaged(&original_microimages, TA.Nb_LaserPositions * PixSquare * sizeof(float));
	hipMallocManaged(&zoomed_microimages, TA.Nb_LaserPositions * PixZoomSquare * sizeof(float));
	hipMallocManaged(&MIintile, tile.NbTile * tile.maxLaserintile * PixZoomSquare * sizeof(float));
	unsigned char *i_MIraw = (unsigned char*) calloc(TA.Nb_LaserPositions * PixSquare, sizeof(char));

	int numberofpixels = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		;
		std::string MIraw = resourcesdirectory + MIFILE + std::to_string(idistrib + 1) + endMI;
		printf("MICROIMAGES \u2464 function read: distribution n°%d Path to distrib original %s .....\n", idistrib,
				MIraw.c_str());
		//read distrib bin file
		std::ifstream MIrawfile(MIraw.c_str(), ios::in | ios::binary | ios::ate);
		size = (MIrawfile.tellg()); 	// the data is stored in doubles of 8 bytes in the file
		size -= byte_skipped;  				// removes the 4 first bytes, Why??
		std::cout << "MICROIMAGES \u2464 function read: distrib n°" << idistrib << " number laser positions "
				<< tile.Nblaserperdistribution[idistrib] << " size microimages = " << size << endl;
		memblock = new char[size];
		MIrawfile.seekg(byte_skipped, ios::beg); // byte_skipped first bytes are offset
		MIrawfile.read(memblock, size);
		MIrawfile.close();

		double_microimages = (double*) memblock; //reinterpret the chars stored in the file as double
		printf("number of images %d Number of pixels %d \n\n", tile.Nblaserperdistribution[idistrib],
				tile.Nblaserperdistribution[idistrib] * PixSquare);
		for (int i = 0; i < tile.Nblaserperdistribution[idistrib] * PixSquare; i++) {
			*(original_microimages + i + numberofpixels) = *(double_microimages + i);			// change to float
			Summicroimages += original_microimages[i];
			Maxmicroimages = max(Maxmicroimages, *(original_microimages + i));
		}
		numberofpixels += tile.Nblaserperdistribution[idistrib] * PixSquare;
		printf("MICROIMAGES \u2464 original on host: Average  %f max microimages %f \n",
				Summicroimages / numberofpixels, Maxmicroimages);

	}
	// write microimages original to disk
	/////////////////////////////////
/*	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
		for (int ilaser = 0; ilaser < tile.Nblaserperdistribution[idistrib]; ilaser++)
			for (int xpix = 0; xpix < Npixel; xpix++)
				for (int ypix = 0; ypix < Npixel; ypix++) {
					int itemp = xpix + idistrib * Npixel + (ypix + ilaser * Npixel) * Npixel * Ndistrib;
					int itemp2 = xpix + idistrib * PixSquare * tile.maxlaserperdistribution
							+ (ypix * Npixel + ilaser * PixSquare);*/
	for (int ilaser = 0; ilaser < TA.Nb_LaserPositions; ilaser++)
		for (int xpix = 0; xpix < Npixel; xpix++)
			for (int ypix = 0; ypix < Npixel; ypix++) {
				int itemp = xpix + Npixel * ypix + PixSquare * ilaser;
				int itemp2 = itemp;
				if (original_microimages[itemp2] > 0.0f)
					i_MIraw[itemp] = 255.0 * original_microimages[itemp2] / Maxmicroimages;			// image value
				}
	printf("MICROIMAGES \u2464 host: Path to microimages original %s .....\n", MIRawfile);

	sdkSavePGM(MIRawfile, i_MIraw,  Npixel, TA.Nb_LaserPositions * Npixel);

	free(i_MIraw);

}

bool validatemicroimages_control(void) {
	bool testmicroimages;
	double Sum3microimages = 0, max3microimages = 0;
	double Sum4microimages = 0, max4microimages = 0;
	const char * MIValfile = "results/MIVALfile.pgm";
	const char * MIzoomfile = "results/MIZOOMfile.pgm";

	// write microimages in memory and validate
	hipMallocManaged(&valmicroimages, (TA.Nb_LaserPositions * PixSquare) * sizeof(float));
	unsigned char *i_MIVal = (unsigned char*) calloc(TA.Nb_LaserPositions * PixSquare, sizeof(char));
	unsigned char *i_MIzoom = (unsigned char*) calloc(TA.Nb_LaserPositions * PixZoom * PixZoom, sizeof(char));

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the microimages kernel
	validate_microimages<<<dimBlock, dimGrid,  0>>>(TA.Nb_LaserPositions);
	hipDeviceSynchronize();
	for (int imicroimages = 0; imicroimages < (TA.Nb_LaserPositions * PixSquare); imicroimages++) {
		Sum3microimages += *(valmicroimages + imicroimages);
		max3microimages = max(max3microimages, *(valmicroimages + imicroimages));
	}
	printf("MICROIMAGES \u2464 Copy from device: Average  %f max3microimages %f \n",
			Sum3microimages / (TA.Nb_LaserPositions * PixSquare), max3microimages);

	for (int imicroimages = 0; imicroimages < (TA.Nb_LaserPositions * PixZoom * PixZoom); imicroimages++) {
		Sum4microimages += *(zoomed_microimages + imicroimages);
		max4microimages = max(max4microimages, *(zoomed_microimages + imicroimages));
	}
	printf("MICROIMAGES \u2464 Copy from device: zoomed image Average  %f max3microimages %f \n",
			Sum4microimages / (TA.Nb_LaserPositions * PixZoom * PixZoom), max4microimages);

	// write microimages image validation to disk
	/////////////////////////////////

	printf("MICROIMAGES \u2464 Comparing files ... ");
	testmicroimages = compareData(valmicroimages, original_microimages,
			TA.Nb_Cols_microimages * TA.Nb_Rows_microimages * Ndistrib,
			MAX_EPSILON_ERROR, 0.15f);

	for (int jmicroimages = 0; jmicroimages < (TA.Nb_LaserPositions * PixSquare); jmicroimages++) {
		Sumdel[4] += fabsf(*(valmicroimages + jmicroimages) - *(original_microimages + jmicroimages));
	}
	printf("Sumdel[4] %f  ", Sumdel[4]);
	cout << "testmicroimages = " << testmicroimages << "\n";

	// write microimages copy to disk
	/////////////////////////////////
	for (int ilaser = 0; ilaser < TA.Nb_LaserPositions; ilaser++)
		for (int xpix = 0; xpix < Npixel; xpix++)
			for (int ypix = 0; ypix < Npixel; ypix++) {
				int itemp = xpix + Npixel * ypix + PixSquare * ilaser;
				int itemp2 = itemp;
				if (valmicroimages[itemp2] > 0.0f)
					i_MIVal[itemp] = 255.0 * valmicroimages[itemp2] / Maxmicroimages;			// image value
			}
				for (int ilaser = 0; ilaser < TA.Nb_LaserPositions; ilaser++)
					for (int xpixzoom = 0; xpixzoom < PixZoom; xpixzoom++)
						for (int ypixzoom = 0; ypixzoom < PixZoom; ypixzoom++) {
							int itemp = xpixzoom + PixZoom * ypixzoom + PixZoomSquare * ilaser;
							int itemp2 = itemp;
							if (zoomed_microimages[itemp2] > 0.0f)
								i_MIzoom[itemp] = 255.0 * zoomed_microimages[itemp2] / max4microimages;			// image value
				}
	printf("MICROIMAGES \u2464 host: Path to microimages copy %s .....\n", MIValfile);

	sdkSavePGM(MIValfile, i_MIVal,  Npixel, TA.Nb_LaserPositions * Npixel);
	sdkSavePGM(MIzoomfile, i_MIzoom, PixZoom, TA.Nb_LaserPositions * PixZoom);

	free(i_MIVal);	free(i_MIzoom);
	hipFree(valmicroimages);

	return (testmicroimages);
}

