#include "hip/hip_runtime.h"
/*
 * big.cu
 *
 *  Created on: Jul 6, 2017
 *      Author: gabriel
 */
#include "NewLoop.h"
bool biglaunch() {

	std::cout << "TESTS RETURN:  " << "initialization:  " << Stepdiag[0] << ";  pPSF:  " << Stepdiag[1];
	std::cout << ";  distrib: " << Stepdiag[2] << endl;
	std::cout << "TESTS RETURN: laser positions: " << Stepdiag[3] << "; ROI:  " << Stepdiag[4];
	std::cout << "; microimages:  " << Stepdiag[5] << ";  Reconstruction:  " << Stepdiag[6] << endl;
	std::cout << "MAIN PROGRAM  **********ready for GPU computation*****************" << endl;
	std::cout << "******************************************************************" << endl << endl;

	std::cout << "To be transferred to device: Number of Aggregates in x:" << tile.NbAggregx << " in y:"
			<< tile.NbAggregy;
	std::cout << "  Number of Tiles per aggregates in x:" << tile.tileperaggregatex << " in y:"
			<< tile.tileperaggregatey << endl;
	std::cout << "To be transferred to device: Number of Tiles in x:" << tile.NbTilex << " in y:" << tile.NbTiley
			<< endl;
	std::cout << "To be transferred to device: Max number of laser position in Tile:" << tile.maxLaserintile
			<< " min value" << tile.minLaserintile << endl << endl;
	printf("**************HOST: PARAMETERS OF MEASUREMENT *******************\n"
			"Npixel %d pZOOM %d, pPSF %d\n", Npixel, pZOOM, pPSF);
	printf("pPSF %d XDistrib %d YDistrib %d\n", pPSF, XDistrib, YDistrib);
	printf("XSCRATCH %d YSCRATCH %d XTile %d YTile %d\n\n", XSCRATCH, YSCRATCH, XTile, YTile);
	printf("Number of pixels calculated in parallel %d Number of threads used %d loop on threads %d\n", NThreads,
	THREADSVAL, THreadsRatio);

	/************************* for GPU ********************/
	std::cout << "MAIN PROGRAM  ********Prepare data for GPU computation**************" << endl;
	std::cout << "******************************************************************" << endl;
	onhost.NbTilex = tile.NbTilex;
	onhost.NbTiley = tile.NbTiley;
	onhost.NbTile = tile.NbTile;
	onhost.NbAggregx = tile.NbAggregx;
	onhost.NbAggregy = tile.NbAggregy;
	onhost.tileperaggregatex = tile.tileperaggregatex;
	onhost.tileperaggregatey = tile.tileperaggregatey;

	onhost.maxLaserintile = tile.maxLaserintile;
	onhost.blocks = tile.blocks;
	onhost.minLaserintile = tile.minLaserintile;
	onhost.Nb_LaserPositions = TA.Nb_LaserPositions;
	onhost.expectedmax = tile.expectedmax;
	onhost.imalimitpertile = onhost.Nb_LaserPositions - (onhost.NbTile - 1) * onhost.maxLaserintile;
	onhost.Bconstant = tile.Bconstant;
	printf("Number of laser positions %d imalimitpertile %d\n", onhost.Nb_LaserPositions, onhost.imalimitpertile);
	bool testbig = FALSE;

	dim3 dimBlock(tile.tileperaggregatex, tile.tileperaggregatey, Ndistrib);
	dim3 dimGrid(THREADSVAL, 1, 1);
	std::cout << "dimBlock  x: " << dimBlock.x << " y: " << dimBlock.y << " z: " << dimBlock.z << "  ...  ";
	std::cout << "dimGrid  x: " << dimGrid.x << " y: " << dimGrid.y << " z: " << dimGrid.z << endl << endl;

	std::cout << "HOST: \u24F3 ************************BigLoop start   *******************************" << endl;
	std::cout << "HOST: \u24F3 ***********************************************************************" << endl;

	int sharedsize = NIMAGESPARALLEL * sizeof(int) + ASCRATCH * sizeof(float) + ADistrib * sizeof(float);
/* 	int *image_to_scratchpad_offset_tile = (int *) shared;				// Offset of each image in NIMAGESPARALLEL block
	float *Scratchpad = (float *) &image_to_scratchpad_offset_tile[NIMAGESPARALLEL];   // ASCRATCH floats for Scratchpad
	float *shared_distrib = (float*) &Scratchpad[ASCRATCH]; 		    		// XDISTRIB*YDISTRIB floats for distrib
 */

	if (sharedsize > TA.sharedmemory) {
		std::cout << "shared memory required is above the memory available" << sharedsize / 1024.0 << "KBytes" << endl;
		exit(1);
	} else
		std::cout << "HOST: \u24F3 *** SHARED MEMORY SIZE " << sharedsize / 1024.0 << " KBytes" << endl;
	// Execute the Laser positions kernel
	BigLoop<<<dimBlock, dimGrid, sharedsize>>>(onhost);
	hipDeviceSynchronize();

	testbig = TRUE;
	return (testbig);
}

