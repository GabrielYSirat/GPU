#include "hip/hip_runtime.h"
/*
 * BigLoop.cu
 *
 *  Created on: Jun 12, 2017
 *      Author: gabriel
 */
/** version without tiles and aggregates: Contains  the next simulation of microimages, in the full measured surface
 *  with optionally number of laser positions below 16 the value of NIMAGESPARALLEL!!
 *  */
#include "NewLoop.h"
#define VERBOSELOOP 1
#include "include.tst"
__managed__ float *new_simus , *Data , *Rfactor, *distribvalidGPU;

__managed__ float EnergyGlobal;
__global__ void BigLoop(devicedata DD) {

	extern __shared__ int shared[]; /***************semi-global variables stored in shared memory ***************/
	int *image_to_scratchpad_offset_tile = (int *) shared;// Offset of each image in NIMAGESPARALLEL block
	float *Scratchpad = (float *) &image_to_scratchpad_offset_tile[NIMAGESPARALLEL]; // ASCRATCH floats for Scratchpad
	float *shared_distrib = (float*) &Scratchpad[ASCRATCH]; // XDISTRIB*YDISTRIB floats for distrib

	int MemoryOffsetscratch = 0; // to be redefine with aggregates
	float MaxNewSimus = 0.0f;
	float * scrglobal;

	/*****************constant values & auxiliary variables stored in registers *****************/
	register float PSFDISVAL[MAXTHRRATIO] = { 0.0f };// multiplication of pPSF and distribution
	register int tmpi[MAXTHRRATIO], ipixel[MAXTHRRATIO], jpixel[MAXTHRRATIO],
			valid_pixel[MAXTHRRATIO], distribpos0[MAXTHRRATIO], distribpos[MAXTHRRATIO];

	/****************Larger segmented areas to be stored in registers **************************/
	// new simus values kept in registers for speed issues
	register float new_simu_inregister_float_0[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_1[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_2[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_3[NIMAGESPARALLEL] = { 0.0f };
	// Running position on the scratchpad, different for:

	/***** INITIALIZATION *****************/

	DD.step = 0;
	int ithreads = threadIdx.x;
	int itb = ithreads + blockIdx.x + blockIdx.y + blockIdx.z;
	int itc = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

	int distrib_number = blockIdx.z;
	if(!ithreads) printf("block x %d y %d z %d distrib number %d itc %d\n", blockIdx.x, blockIdx.y, blockIdx.z, distrib_number, itc );
	__syncthreads();
	int iprint = !VERBOSELOOP + itb;
	int center_distrib = ((YDistrib / 2) * XDistrib) + XDistrib / 2;
	int center_microimage = (PixZoomo2) * PixZoom + PixZoomo2;
	time_init = clock64();
	time_start = time_init;
#include "startdevice.cu"

	/***************************Basic parameters **************************************************/
	/*************************Threads and pixels related parameters *******************************/

	for (int apix = 0; apix < THreadsRatio; apix++) { // ipixel, jpixel have 0 values too often
		tmpi[apix] = (ithreads + apix * THREADSVAL);
		ipixel[apix] = tmpi[apix] % PixZoom - PixZoomo2; // centered on the center of the zoomed microimage
		jpixel[apix] = tmpi[apix] / PixZoom - PixZoomo2; // centered on the center of the zoomed microimage
		valid_pixel[apix] = tmpi[apix] < PixZoomSquare;
		distribpos0[apix] = center_distrib + ipixel[apix] - PSFZoomo2
				+ (jpixel[apix] - PSFZoomo2) * XDistrib;
	}
#include "testthreads.cu"

	__syncthreads();

	/*************************************************************************************************/
	/**O. Initialize zoomed distrib as calculated  by the preprocessing                               /
	 /** the mosaic has to be prepared before hand on the host and copied in global memory            /
	 /************************************************************************************************/
#pragma unroll
	for (int idistrub = ithreads; idistrub < ADistrib; idistrub += THREADSVAL)
			*(shared_distrib + idistrub) = *(original_distrib + idistrub + distrib_number * ADistrib);

#include "testdistrib.tst"

	/*********************  ***********/
	/**A  Outer Loop on aggregates   **/
	/*********************  ***********/
	int tilex = blockIdx.x;
	int tiley = blockIdx.y;
	int tile = tilex + DD.NbTilex * tiley;
	MemoryOffsetscratch = ASCRATCH * (tilex + tiley * DD.NbTilex);
	/************************************************************************************************/
	/******************************               end of A            *******************************/

	/**B. Initialize Scratchpad to previous reconstruction in float : OPTIMIZED, also with aggregates/
	 /** the mosaic has to be prepared before hand on the host and copied in global memory            /
	 /************************************************************************************************/
	scrglobal = scratchpad_matrix + MemoryOffsetscratch;
#pragma unroll
	for (int iscratch = ithreads; iscratch < ASCRATCH; iscratch += THREADSVAL)
		*(Scratchpad + iscratch) = *(scrglobal + iscratch);
#include "testaggregates.tst"
#include "testdevice.cu"
	/*********************************************************************************/
	/**       END of B                                             *******************/

	/**C  Intermediate Loop on images blocks of NIMAGESPARALLEL   ********************/
	/*********************************************************************************/
	/** preparation of intermediate data for each block of NIMAGESPARALLEL************/
	/******************does not worth additional parallelization!*********************/

	for (int iglobal = 0; iglobal < DD.maxLaserintile; iglobal +=
			NIMAGESPARALLEL) { // image number in global tile list
		int zero_posimages = ithreads
				+ (iglobal + tile * DD.maxLaserintile) * NThreads;
		for (int apix = 0; apix < THreadsRatio; apix++)
			distribpos[apix] = distribpos0[apix];

//		if(!iprint)for (int apix = 0; apix < THreadsRatio; apix++)printf("APIX DISTRIB: apix %d distribpos[apix] %d \n", apix, distribpos[apix]);
// validé
		// zero of microimages and simus
		//each thread, for each SM, for each image, on several pixels separated by THREADSVAL of the small block
		register float *pscratch_0[NIMAGESPARALLEL];
		register float *pscratch_1[NIMAGESPARALLEL];
		register float *pscratch_2[NIMAGESPARALLEL];
		register float *pscratch_3[NIMAGESPARALLEL];

		for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {

			// C_1. Transfer from global to shared memory the relative position of the beginning of the scratchpad
			// region impacted by the pPSF,relative to the scratchpad start for each image  of the small group
			if ((iblockima + iglobal) < DD.NbLaserpertile[tile])
				*(image_to_scratchpad_offset_tile + iblockima) =
						*(image_to_scratchpad_offset + iglobal + iblockima)
								- (XSCRATCH + 1) * PSFZoomo2;
			else
				*(image_to_scratchpad_offset_tile + iblockima) =
				dySCR * XSCRATCH + dxSCR - (XSCRATCH + 1) * PSFZoomo2;
			if(!iprint)printf("OFFSET:iblockima %d offset %d iglobal %d DD.NbLaserpertile[tile] %d\n",
					iblockima,*(image_to_scratchpad_offset_tile + iblockima), iglobal, DD.NbLaserpertile[tile]);
			// validé
		}

		// C.2	Initialize new_simu for all pixels of this thread of simus ,THreadsRatio of them, to zero
		// this occurs for each of image iglobal used in this particular thread,
		for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
			new_simu_inregister_float_0[iblockima] = 0.0f;
			new_simu_inregister_float_1[iblockima] = 0.0f;
			new_simu_inregister_float_2[iblockima] = 0.0f;
			new_simu_inregister_float_3[iblockima] = 0.0f;
		}

		// C.3 initialize the scratch position for each image for each pixel of the group dealt in this thread
		for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
			int pos_0 = image_to_scratchpad_offset_tile[iblockima] + ipixel[0]
					+ jpixel[0] * XSCRATCH;
			int pos_1 = image_to_scratchpad_offset_tile[iblockima] + ipixel[1]
					+ jpixel[1] * XSCRATCH;
			int pos_2 = image_to_scratchpad_offset_tile[iblockima] + ipixel[2]
					+ jpixel[2] * XSCRATCH;
			int pos_3 = image_to_scratchpad_offset_tile[iblockima] + ipixel[3]
					+ jpixel[3] * XSCRATCH;
			pscratch_0[iblockima] = (Scratchpad + pos_0); // Change (simplify) in CUDA 9.0
			pscratch_1[iblockima] = (Scratchpad + pos_1);
			pscratch_2[iblockima] = (Scratchpad + pos_2);
			pscratch_3[iblockima] = (Scratchpad + pos_3);
#include "pscratch.cu"
		}
#include "testdevice.cu"

		/**************************************/
		/******D. SIMUS CALCULATION************/
		/**************************************/
		/** D_1 Loop on pPSF on y axis - Medium level loop
		 position on: pPSF from 0 to PSFZoom,
		 * distribution from jpixelPSF pixel position
		 */
		for (int jPSF = 0; jPSF < PSFZoom; jPSF++) {

# pragma unroll
			for (int iPSF = 0; iPSF < PSFZoom; iPSF++) {
				int PSFpos = iPSF + jPSF * PSFZoom;

				for (int apix = 0; apix < THreadsRatio; apix++)
					//				PSFDISVAL[apix] = valid_pixel[apix]  * *(original_distrib + distribpos[apix]);
					PSFDISVAL[apix] = valid_pixel[apix]
							* *(original_PSF + PSFpos)
							* *(original_distrib + distribpos[apix]);
				/** D_3 Inner loops on THreadsRatio pixels block and on block of NIMAGESPARALLEL images require best optimization
				 * */
# pragma unroll
				for (int iblockima = 0; iblockima < NIMAGESPARALLEL;
						iblockima++) {
					float tmp_0 = *(pscratch_0[iblockima]);
					pscratch_0[iblockima]++;
					new_simu_inregister_float_0[iblockima] += PSFDISVAL[0]
							* tmp_0;
					float tmp_1 = *(pscratch_1[iblockima]);
					pscratch_1[iblockima]++;
					new_simu_inregister_float_1[iblockima] += PSFDISVAL[1]
							* tmp_1;
					float tmp_2 = *(pscratch_2[iblockima]);
					pscratch_2[iblockima]++;
					new_simu_inregister_float_2[iblockima] += PSFDISVAL[2]
							* tmp_2;
					float tmp_3 = *(pscratch_3[iblockima]);
					pscratch_3[iblockima]++;
					new_simu_inregister_float_3[iblockima] += PSFDISVAL[3]
							* tmp_3;
#include "TESTPSFDISVAL.cu"
				}
				for (int apix = 0; apix < THreadsRatio; apix++) {
					if ((ithreads + THREADSVAL * apix) == center_microimage) {
						*(distribvalidGPU + iPSF + jPSF * PSFZoom + itc*PSFZOOMSQUARE) =
								*(shared_distrib + distribpos[apix]);
						distribpos[apix]++;  // update intermediate value of distrib
					}
				}
			}  // iPSF loop

			for (int apix = 0; apix < THreadsRatio; apix++) {
				if ((ithreads + THREADSVAL * apix) == center_microimage) {
					*(distribvalidGPU + jPSF * PSFZoom + itc *PSFZOOMSQUARE) = *(shared_distrib + distribpos[apix]);
					distribpos[apix] += XDistrib - PSFZoom; // update intermediate value of distrib for a full line
				}
			}

			for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
				pscratch_0[iblockima] += XSCRATCH - PSFZoom;
				pscratch_1[iblockima] += XSCRATCH - PSFZoom;
				pscratch_2[iblockima] += XSCRATCH - PSFZoom;
				pscratch_3[iblockima] += XSCRATCH - PSFZoom;
			}
		} // loop on PSFpos which spans all PSF values
#include "testdevice.cu"

		for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {	// Removing images which are not valid (putting value to zero)
			new_simu_inregister_float_0[iblockima] = valid_image[iblockima]
					* new_simu_inregister_float_0[iblockima];
			new_simu_inregister_float_1[iblockima] = valid_image[iblockima]
					* new_simu_inregister_float_1[iblockima];
			new_simu_inregister_float_2[iblockima] = valid_image[iblockima]
					* new_simu_inregister_float_2[iblockima];
			new_simu_inregister_float_3[iblockima] = valid_image[iblockima]
					* new_simu_inregister_float_3[iblockima];
		}
#include "testdevice.cu"

		int it = zero_posimages;
# pragma unroll
		for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
			new_simus[it] = new_simu_inregister_float_0[iblockima];
			new_simus[it + 1 * THREADSVAL] =
					new_simu_inregister_float_1[iblockima];
			new_simus[it + 2 * THREADSVAL] =
					new_simu_inregister_float_2[iblockima];
			new_simus[it + 3 * THREADSVAL] =
					new_simu_inregister_float_3[iblockima];
			it += NThreads;
		}

#ifdef TESTSIMU

		for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {

			MaxNewSimus = max(MaxNewSimus,
					new_simu_inregister_float_0[iblockima]);
			MaxNewSimus = max(MaxNewSimus,
					new_simu_inregister_float_1[iblockima]);
			MaxNewSimus = max(MaxNewSimus,
					new_simu_inregister_float_2[iblockima]);
			MaxNewSimus = max(MaxNewSimus,
					new_simu_inregister_float_3[iblockima]);
		}
		if (!iprint)
			printf("DEVICE:\u2467 new simus max %f\n\n", MaxNewSimus);
		__syncthreads();
#endif

//#include "testsimu.tst"
	}
//	*DD.stepval = DD.step;
}

