/*
 * MainGPU.cu
 *
 *  Created on: June 5, 2017
 *      Author: gabriel Sirat
 */

#include "NewLoop.h"

/*******************PARAMETERS**************/
char buff[BUFFSIZE]; // a buffer to temporarily park the data
double Timestep[16];
char chars[] = "[]()", delimeter('=');
int clockRate, devID, stepval = 0; // in KHz
__managed__ clock_t timer, time_init, time_start, time_loop_stop; // in KHz
__managed__ int pPSF, Npixel, RDISTRIB, pZOOM, Ndistrib;
__managed__ double  Energy_Global =0.0f;

hipEvent_t start, stop, init_t;
float time_event;
std::string resourcesdirectory, filename, name, value;

/************* Classes and structures*******/
GPU_init TA;
COS OFSCAL;
Ctile tile;
devicedata onhost;

__managed__ double Energy_global;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {


	/***********initialization of  parameters step 0 *******/
	bool InitParameters = initparameters(argc, argv);
	stepinit(InitParameters, stepval);

	/*step 1**********************pPSF initialization ********/
	PSFprepare();
	bool TestPSF = PSFvalidateonhost();
	stepinit(TestPSF, stepval);

	/*step 2 ************distrib initialization **************/
	readstoredistrib();
	bool Testdistrib = Distribvalidate_host();
	stepinit(Testdistrib, stepval);

	/*step 3 ***********************Laser positions***********/
	readstoreLaserPositions();
	bool TestLaserPositions = validateLaserPositions_control();
	stepinit(TestLaserPositions, stepval);

	/*step 4 **************************Cropped ROI************/
	readstoreCroppedROI();
	bool TestROI = validateCroppedROI_control();
	stepinit(TestROI, stepval);

	/*step 5 *********************microimages  ****************/
	readstoremicroimages();
	bool testmicroimages = validatemicroimages_control();
	stepinit(testmicroimages, stepval);

	/*step 6 ************************Laser in tile ************/
	bool Lasertile = tileorganization();
	stepinit(Lasertile, stepval);

	/*step 7 ************************µimages in tile **********/
	bool boolMI = microimagesintile();
	onhost.MaxData = displaydata( Data,  stepval);
	stepinit(boolMI, stepval);

	/*step 8 *********************Reconstruction  *************/
	Recprepare();
	bool testreconstruction = Recvalidate_host();
	stepinit(testreconstruction, stepval);

	/*step 9 *********************Scratchpad  *****************/
	Scratchprepare();
	bool testscratchpad = Scratchvalidate_host();
	stepinit(testscratchpad, stepval);

	report_gpu_mem();

	/*step 10 *********************launch BigLoop  ************/

	bool testLoop = biglaunch();
	stepinit(testLoop, stepval);

	/*step 11 *********************Energy  ********************/
Energy_global = EnergyCal();
	bool testEnergy = (Energy_Global !=0.0f);
	stepinit(testEnergy, stepval);

	/*step 12 *********************Inspect results  ***********/

	bool testinspect = biginspect(stepval);
	stepinit(testinspect, stepval);

}

