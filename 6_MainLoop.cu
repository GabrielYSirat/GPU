#include "hip/hip_runtime.h"
/*
 * 6_MainLoop.cu
 *
 *  Created on: November 23, 2017
 *      Author: Gabriel Y Sirat */
/** Contains  the next simulation of microimages, in the full measured surface
 *  with optionally number of laser positions below 16 the value of NIMAGESPARALLEL!!
 *  **/
#include "0_Mainparameters.h"
#define VERBOSELOOP 1
#define SPARSEDATA 1
#include "0_include.tst"
__managed__ float *new_simus, *Data, *Rfactor, *distribvalidGPU;
__managed__ float EnergyGlobal = 0.0f;
__global__ void BigLoop(devicedata DD) {

	extern __shared__ int shared[]; /***************semi-global variables stored in shared memory ***************/
	int *image_to_scratchpad_offset_tile = (int *) shared; // Offset of each image in NIMAGESPARALLEL block
	float *Scratchpad = (float *) &image_to_scratchpad_offset_tile[NIMAGESPARALLEL]; // ASCRATCH floats for Scratchpad
	float *shared_distrib = (float*) &Scratchpad[ASCRATCH]; // ASCRATCH floats for distrib

	int MemoryOffsetscratch = 0, tilex, tiley, tileXY;
	float * scrglobal;

	/*****************constant values & auxiliary variables stored in registers *****************/
	register float PSFDISVAL[MAXTHRRATIO] = { 0.0f }; // multiplication of pPSF and distribution
	register int tmpi[MAXTHRRATIO], ipixel[MAXTHRRATIO], jpixel[MAXTHRRATIO], valid_pixel[MAXTHRRATIO],
			distribpos0[MAXTHRRATIO], distribpos[MAXTHRRATIO];

	/****************Larger segmented areas to be stored in registers, for speed issues **************************/
	register float new_simu_inregister_float_0[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_1[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_2[NIMAGESPARALLEL] = { 0.0f };
	register float new_simu_inregister_float_3[NIMAGESPARALLEL] = { 0.0f };

	/***** INITIALIZATION *****************/
	int ithreads = threadIdx.x;
	int distrib_number = blockIdx.z;
	int itb = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y; // Block position
	int itc = ithreads + itb;
	int iprint = !VERBOSELOOP + itc;
	int center_distrib = ((YDistrib / 2) * XDistrib) + XDistrib / 2;
	int center_microimage = (PixZoomo2) * PixZoom + PixZoomo2;
	DD.step = 1;
	time_init = clock64(); time_start = clock64(); timer = clock64();
#include "8_startdevice.cu"

	/*************************Threads and pixels related parameters *******************************/

	for (int apix = 0; apix < THreadsRatio; apix++) {
		tmpi[apix] = (ithreads + apix * THREADSVAL);
		ipixel[apix] = tmpi[apix] % PixZoom - PixZoomo2; // centered on the center of the zoomed microimage
		jpixel[apix] = tmpi[apix] / PixZoom - PixZoomo2; // centered on the center of the zoomed microimage
		valid_pixel[apix] = tmpi[apix] < PixZoomSquare;
		distribpos0[apix] = center_distrib + ipixel[apix] - PSFZoomo2 + (jpixel[apix] - PSFZoomo2) * XDistrib;
	}
#include "8_testthreads.cu"

	/*************************************************************************************************/
	/**O. Initialize zoomed distrib as calculated  by the preprocessing                               /
	 /************************************************************************************************/
#pragma unroll
	for (int idistrub = ithreads; idistrub < ADistrib; idistrub += THREADSVAL)
		*(shared_distrib + idistrub) = *(original_distrib + idistrub + distrib_number * ADistrib);
#include "8_testdistrib.cu"

	/*********************  ***********/
	/**A  Outer Loop on aggregates   **/
	/*********************  ***********/
	for (int aggregx = 0; aggregx < DD.NbAggregx; aggregx++)
		for (int aggregy = 0; aggregy < DD.NbAggregy; aggregy++) {
			tilex = blockIdx.x + aggregx * DD.tileperaggregatex;
			tiley = blockIdx.y + aggregy * DD.tileperaggregatey;
			tileXY = tilex + DD.NbTilex * tiley;
			MemoryOffsetscratch = ASCRATCH * tileXY;
			scrglobal = scratchpad_matrix + MemoryOffsetscratch;
#include "8_testaggreg.cu"

			/**B. Initialize Scratchpad to previous reconstruction in float : OPTIMIZED, also with aggregates/
			 /************************************************************************************************/
#pragma unroll
			for (int iscratch = ithreads; iscratch < ASCRATCH; iscratch += THREADSVAL)
				*(Scratchpad + iscratch) = *(scrglobal + iscratch);
#include "8_testscratch.cu"
			/**       END of B                                             *******************/

			/**C  Intermediate Loop on images blocks of NIMAGESPARALLEL   ********************/
			/*********************************************************************************/
			/** preparation of intermediate data for each block of NIMAGESPARALLEL************/
			register float *pscratch_0[NIMAGESPARALLEL], *pscratch_1[NIMAGESPARALLEL], *pscratch_2[NIMAGESPARALLEL], *pscratch_3[NIMAGESPARALLEL];

			for (int iglobal = 0; iglobal < DD.maxLaserintile; iglobal += NIMAGESPARALLEL) { // image number in global tile list
				int zero_posimages = ithreads + (iglobal + tileXY * DD.maxLaserintile) * NThreads;
				for (int apix = 0; apix < THreadsRatio; apix++) distribpos[apix] = distribpos0[apix];
#include "8_distribpos.cu"
				//Thread, for each SM, for each image, on several pixels separated by THREADSVAL of the small block

				// C_1. Transfer from global to shared memory the relative position of the beginning of the scratchpad
				for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
					if ((iblockima + iglobal) < DD.NbLaserpertile[tileXY])
						*(image_to_scratchpad_offset_tile + iblockima) = *(image_to_scratchpad_offset+ iglobal + iblockima) - (XSCRATCH + 1) * PSFZoomo2;
					else
						*(image_to_scratchpad_offset_tile + iblockima) = dySCR * XSCRATCH + dxSCR - (XSCRATCH + 1) * PSFZoomo2;
				}

#include "8_offset.cu"
				// C.2	Initialize new_simu for all pixels of this thread of simus ,THreadsRatio of them, to zero
				for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
					new_simu_inregister_float_0[iblockima] = 0.0f;
					new_simu_inregister_float_1[iblockima] = 0.0f;
					new_simu_inregister_float_2[iblockima] = 0.0f;
					new_simu_inregister_float_3[iblockima] = 0.0f;
				}

				// C.3 initialize the scratch position for each image for each pixel of the group dealt in this thread
				for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
					int pos_0 = image_to_scratchpad_offset_tile[iblockima] + ipixel[0] + jpixel[0] * XSCRATCH;
					int pos_1 = image_to_scratchpad_offset_tile[iblockima] + ipixel[1] + jpixel[1] * XSCRATCH;
					int pos_2 = image_to_scratchpad_offset_tile[iblockima] + ipixel[2] + jpixel[2] * XSCRATCH;
					int pos_3 = image_to_scratchpad_offset_tile[iblockima] + ipixel[3] + jpixel[3] * XSCRATCH;
					pscratch_0[iblockima] = (Scratchpad + pos_0); // Change (simplify) in CUDA 9.0
					pscratch_1[iblockima] = (Scratchpad + pos_1);
					pscratch_2[iblockima] = (Scratchpad + pos_2);
					pscratch_3[iblockima] = (Scratchpad + pos_3);
#include "8_pscratchtest.cu"
				} // end of blockima loop

				/**************************************/
				/******D. SIMUS CALCULATION************/
				/**************************************/
				/** D_1 Loop on pPSF on y axis -  on: pPSF from 0 to PSFZoom,
				 * distribution from jpixelPSF pixel position
				 */
				for (int jPSF = 0; jPSF < PSFZoom; jPSF++) { // loop on jPSF

# pragma unroll
					for (int iPSF = 0; iPSF < PSFZoom; iPSF++) { // loop on iPSF
						int PSFpos = iPSF + jPSF * PSFZoom;

						for (int apix = 0; apix < THreadsRatio; apix++)
							PSFDISVAL[apix] = valid_pixel[apix] * *(original_PSF + PSFpos) * *(original_distrib + distribpos[apix]);
						/** D_3 Inner loops on THreadsRatio pixels block and on block of NIMAGESPARALLEL images
						 * require best optimization in assembler **/
# pragma unroll
						for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
							float tmp_0 = *(pscratch_0[iblockima]);
							new_simu_inregister_float_0[iblockima] += PSFDISVAL[0] * tmp_0;
							float tmp_1 = *(pscratch_1[iblockima]);
							new_simu_inregister_float_1[iblockima] += PSFDISVAL[1] * tmp_1;
							float tmp_2 = *(pscratch_2[iblockima]);
							new_simu_inregister_float_2[iblockima] += PSFDISVAL[2] * tmp_2;
							float tmp_3 = *(pscratch_3[iblockima]);
							new_simu_inregister_float_3[iblockima] += PSFDISVAL[3] * tmp_3;
#include "8_testdisval.cu"
							pscratch_0[iblockima]++;pscratch_1[iblockima]++;
							pscratch_2[iblockima]++;pscratch_3[iblockima]++;
						}
#include "8_testdistribvalA.cu"
						for (int apix = 0; apix < THreadsRatio; apix++) distribpos[apix]++;  // update intermediate value of distrib
					}  // iPSF loop
					for (int apix = 0; apix < THreadsRatio; apix++) distribpos[apix] += XDistrib - PSFZoom; // update intermediate value of distrib for a full line
#include "8_testdistribvalB.cu"

					for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
						pscratch_0[iblockima] += XSCRATCH - PSFZoom;
						pscratch_1[iblockima] += XSCRATCH - PSFZoom;
						pscratch_2[iblockima] += XSCRATCH - PSFZoom;
						pscratch_3[iblockima] += XSCRATCH - PSFZoom;
					} // loop on iblockima
				} // loop on jPSF which spans all PSF values

				for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {	// Removing images which are not valid (putting value to zero)
					new_simu_inregister_float_0[iblockima] = valid_image[iblockima] * new_simu_inregister_float_0[iblockima];
					new_simu_inregister_float_1[iblockima] = valid_image[iblockima] * new_simu_inregister_float_1[iblockima];
					new_simu_inregister_float_2[iblockima] = valid_image[iblockima] * new_simu_inregister_float_2[iblockima];
					new_simu_inregister_float_3[iblockima] = valid_image[iblockima] * new_simu_inregister_float_3[iblockima];
				} // loop on iblockima

				int it = zero_posimages;
		# pragma unroll
				for (int iblockima = 0; iblockima < NIMAGESPARALLEL; iblockima++) {
					new_simus[it] = new_simu_inregister_float_0[iblockima];
					new_simus[it + 1 * THREADSVAL] = new_simu_inregister_float_1[iblockima];
					new_simus[it + 2 * THREADSVAL] = new_simu_inregister_float_2[iblockima];
					new_simus[it + 3 * THREADSVAL] = new_simu_inregister_float_3[iblockima];
					it += NThreads;
				} // loop on iblockima
			}
// end of iglobal loop
		} // end of Aggregates loop
	if(!iprint && VERBOSE) printf("Energy %8.6f absolute difference %8.6f\n\n", EnergyGlobal, absdiff);
} // end of 6_MainLoop

