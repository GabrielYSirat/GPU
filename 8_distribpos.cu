
#include <hip/hip_runtime.h>
#ifdef DISTRIBPOS
	DD.step++;

	if(!iprint && ((aggregx+1) == DD.NbAggregx) && ((aggregy+1) == DD.NbAggregy)) {
		for (int apix = 0; apix < THreadsRatio; apix++)
		printf("DEVICE: \u23f2 APIX DISTRIB: apix %d distribpos[apix] %d \n", apix, distribpos[apix]);
// validé

	if (!iprint) timer = clock64();
	if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	}
	__syncthreads();
#endif
