#include "hip/hip_runtime.h"
/*
 * biginspect.cu
 *
 *  Created on: Sep 16, 2017
 *      Author: gabriel
 */

#include "0_Mainparameters.h"
bool biginspect(int stepval) {
	bool boolinspect;
	float MaxScratchpad = 0.0f, SumScratchpad = 0.0f;
//	float MaxDispos = 0.0f, SumDispos = 0.0f;

	onhost.MaxSimus = displaydata(new_simus, stepval);

	stepval ++;

	onhost.MaxRfactor = displaydata(Rfactor, stepval);



	verbosefile << "HOST: \u24F3 *************************BigLoop terminated ***************************" << endl;
	verbosefile << "HOST: \u24F3 ***********************************************************************" << endl << endl;
	unsigned char *i_Scratchpad = (unsigned char *) calloc(XSCRATCH * YSCRATCH * tile.NbTileXY * Ndistrib, sizeof(unsigned char)); // on host
	const char * ScratchpadVal2Imagefile = "results/ScratchpadVal2Imagefile.pgm";

	for (int arg = 0; arg < ASCRATCH * tile.NbTileXY; arg++) SumScratchpad += *(val_scratchpad + arg);
	for (int arg = 0; arg < ASCRATCH * tile.NbTileXY; arg++) MaxScratchpad = max(MaxScratchpad, *(val_scratchpad + arg));

	if (VERBOSE)
		for (int ity = 0; ity < tile.NbTiley; ity++)
			for (int itx = 0; itx < tile.NbTilex; itx++) {
				int it = itx + ity * tile.NbTilex;
				for (int arg = lostpixels; arg < XSCRATCH * YSCRATCH + lostpixels; arg++) {
					int arg1D = arg + it * ASCRATCH;
					int argy = (arg - lostpixels) / XSCRATCH;
					int argx = (arg - lostpixels) % XSCRATCH;
					int arg2D = argx + itx * XSCRATCH + argy * XSCRATCH * tile.NbTilex
							+ ity * YSCRATCH * XSCRATCH * tile.NbTilex;
					i_Scratchpad[arg2D] = 255.0 * val2_scratchpad[arg1D] / MaxScratchpad;// Validation image value
					if (val2_scratchpad[arg1D] != 0.0f)
					verbosefile << "DEVICE TEST in big.cu: SCRATCHPAD :  arg1D " << arg1D;
					verbosefile << "arg2D " << arg2D << " arg x & y " << argx << "  " << argy;
					verbosefile << " value " << val2_scratchpad[arg1D] << "  " << MaxScratchpad << endl;
				}
			}

	verbosefile << "SCRATCHPAD \u24EC Path to Scratchpad validation  ....." << ScratchpadVal2Imagefile << endl;
	sdkSavePGM(ScratchpadVal2Imagefile, i_Scratchpad, XSCRATCH * tile.NbTilex, YSCRATCH * tile.NbTiley);
	for (int i = 0; i < XSCRATCH * YSCRATCH + lostpixels; i++) {
		if (MaxScratchpad < val2_scratchpad[i])
			MaxScratchpad = val2_scratchpad[i]; // sanity check, check max
	}
	verbosefile << "max device after BigLoop =" << MaxScratchpad << "\n";
/*	for (int i = 0; i < XSCRATCH * YSCRATCH + lostpixels; i++)
		i_Scratchpad[i - lostpixels] = 255.0 * val2_scratchpad[i] / MaxScratchpad;			// Validation image value


	sdkSavePGM(ScratchpadVal2Imagefile, i_Scratchpad, XSCRATCH, YSCRATCH);*/

	boolinspect = ((onhost.MaxSimus == 0.0f)&&(onhost.MaxRfactor==0.0f));

	unsigned char *i_distribpos = (unsigned char *) calloc(TA.MP* PSFZOOMSQUARE, sizeof(unsigned char)); // on host
	unsigned char *j_distribpos = (unsigned char *) calloc(XDistrib * TA.MP * YDistrib_extended, sizeof(unsigned char)); // on host
	const char * DistribPosImage = "results/DistribPos.pgm";
	const char * DistribTestImage = "results/DistribTest.pgm";
	verbosefile << "SCRATCHPAD \u24EC Path to DistribPos validation ....." << DistribPosImage << endl;


	for (int i = 0; i < TA.MP * PSFZOOMSQUARE; i++)	i_distribpos[i] = 255.0 * distribvalidGPU[i] / Maxdistrib;			// Validation image value
	sdkSavePGM(DistribPosImage, i_distribpos, PSFZoom, TA.MP * PSFZoom);

	for (int i = 0; i < TA.MP * ADistrib; i++)	{
		int tempa = i%ADistrib;
		int tempb = i/ADistrib;
		if (tempa < XDistrib * YDistrib_extended)
		j_distribpos[tempa + tempb * XDistrib * YDistrib_extended ] = 255.0 * test2_distrib[i] / Maxdistrib;			// Validation image value
	}
	sdkSavePGM(DistribTestImage, j_distribpos, XDistrib, TA.MP * YDistrib_extended);

	return (boolinspect);
}

