#include "hip/hip_runtime.h"
/*
 * biginspect.cu
 *
 *  Created on: Sep 16, 2017
 *      Author: gabriel
 */

#include "0_NewLoop.h"
bool biginspect(int stepval) {
	bool boolinspect;
	float MaxScratchpad = 0.0f, SumScratchpad = 0.0f;
//	float MaxDispos = 0.0f, SumDispos = 0.0f;

	onhost.MaxSimus = displaydata(new_simus, stepval);

	stepval ++;

	onhost.MaxRfactor = displaydata(Rfactor, stepval);



	std::cout << "HOST: \u24F3 *************************BigLoop terminated ***************************" << endl;
	std::cout << "HOST: \u24F3 ***********************************************************************" << endl << endl;
	unsigned char *i_Scratchpad = (unsigned char *) calloc(XSCRATCH * YSCRATCH * tile.NbTile, sizeof(unsigned char)); // on host
	const char * ScratchpadVal2Imagefile = "results/ScratchpadVal2Imagefile.pgm";

	for (int arg = 0; arg < ASCRATCH * tile.NbTile; arg++)		SumScratchpad += *(val_scratchpad + arg);
	for (int arg = 0; arg < ASCRATCH * tile.NbTile; arg++)   	MaxScratchpad = max(MaxScratchpad, *(val_scratchpad + arg));
	for (int ity = 0; ity < tile.NbTiley; ity++)
		for (int itx = 0; itx < tile.NbTilex; itx++) {
			int it = itx + ity * tile.NbTilex;
			for (int arg = lostpixels; arg < XSCRATCH * YSCRATCH + lostpixels; arg++) {
				int arg1D = arg + it * ASCRATCH;
				int argy = (arg - lostpixels) / XSCRATCH;
				int argx = (arg - lostpixels) % XSCRATCH;
				int arg2D = argx + itx * XSCRATCH + argy * XSCRATCH * tile.NbTilex
						+ ity * YSCRATCH * XSCRATCH * tile.NbTilex;
				i_Scratchpad[arg2D] = 255.0 * val2_scratchpad[arg1D] / MaxScratchpad;		// Validation image value
				if (val2_scratchpad[arg1D] != 0.0f)
					printf(
							"DEVICE TEST in big.cu: SCRATCHPAD :  arg1D %d, arg2D %d, argx %d, argy %d  value %f  max %f\n",
							arg1D, arg2D, argx, argy, val2_scratchpad[arg1D], MaxScratchpad);
			}
		}

	printf("SCRATCHPAD \u24EC Path to Scratchpad validation %s .....\n", ScratchpadVal2Imagefile);
	sdkSavePGM(ScratchpadVal2Imagefile, i_Scratchpad, XSCRATCH * tile.NbTilex, YSCRATCH * tile.NbTiley);
	for (int i = 0; i < XSCRATCH * YSCRATCH + lostpixels; i++) {
		if (MaxScratchpad < val2_scratchpad[i])
			MaxScratchpad = val2_scratchpad[i]; // sanity check, check max
	}
	std::cout << "max device after BigLoop =" << MaxScratchpad << "\n";
	for (int i = 0; i < XSCRATCH * YSCRATCH + lostpixels; i++)
		i_Scratchpad[i - lostpixels] = 255.0 * val2_scratchpad[i] / MaxScratchpad;			// Validation image value


	sdkSavePGM(ScratchpadVal2Imagefile, i_Scratchpad, XSCRATCH, YSCRATCH);

	boolinspect = ((onhost.MaxSimus == 0.0f)&&(onhost.MaxRfactor==0.0f));

	unsigned char *i_distribpos = (unsigned char *) calloc(TA.MP* PSFZOOMSQUARE, sizeof(unsigned char)); // on host
	unsigned char *j_distribpos = (unsigned char *) calloc(XDistrib * TA.MP * YDistrib_extended, sizeof(unsigned char)); // on host
	const char * DistribPosImage = "results/DistribPos.pgm";
	const char * DistribTestImage = "results/DistribTest.pgm";
	printf("SCRATCHPAD \u24EC Path to DistribPos validation %s .....\n", DistribPosImage);


	for (int i = 0; i < TA.MP * PSFZOOMSQUARE; i++)	i_distribpos[i] = 255.0 * distribvalidGPU[i] / Maxdistrib;			// Validation image value
	sdkSavePGM(DistribPosImage, i_distribpos, PSFZoom, TA.MP * PSFZoom);

	for (int i = 0; i < TA.MP * ADistrib; i++)	{
		int tempa = i%ADistrib;
		int tempb = i/ADistrib;
		if (tempa < XDistrib * YDistrib_extended)
		j_distribpos[tempa + tempb * XDistrib * YDistrib_extended ] = 255.0 * test2_distrib[i] / Maxdistrib;			// Validation image value
	}
	sdkSavePGM(DistribTestImage, j_distribpos, XDistrib, TA.MP * YDistrib_extended);

	return (boolinspect);
}

