
#include <hip/hip_runtime.h>
#ifdef TESTTHREADS
	DD.step++;
	if (!iprint) printf("\n\u2461*******************************DEVICE:  THREADS *********************\n");
		for (int apix = 0; apix < THreadsRatio; apix++)
		if(!itb)
			if ((ithreads == 0)|| (!ipixel[apix] && !jpixel[apix]))
		{
						printf("DEVICE: \u2461 : apix %d ithreads %d tmpi[apix] %d ipixel %d, jpixel %d  valid %d distribpos0 %d\n",
								apix, ithreads, tmpi[apix], ipixel[apix], jpixel[apix], valid_pixel[apix], distribpos0[apix]);
				}
		if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %g this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	time_start = clock64();
	if (!iprint) printf("\u2461 **********************************DEVICE:  THREADS  ********************\n\n");
	__syncthreads();
#endif
