
#include <hip/hip_runtime.h>
#ifdef TESTTHREADS
	DD.step++;
	if (!iprint) printf("\n\u2461*******************************DEVICE:  THREADS *********************\n");
	__syncthreads();

		for (int apix = 0; apix < THreadsRatio; apix++)
				if (   (ithreads == 32))
						printf("DEVICE: \u2461 : apix %d ithreads %d\n", apix, ithreads);
	if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %g this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	time_start = clock64();
	if (!iprint) printf("\u2461 **********************************DEVICE:  THREADS  ********************\n\n");
	__syncthreads();
#endif
