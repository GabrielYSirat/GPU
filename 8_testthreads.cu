
#include <hip/hip_runtime.h>
#ifdef TESTTHREADS
	DD.step++;
	if (!iprint) printf("\n\u2461*******************************DEVICE:  THREADS *********************\n");
	__syncthreads();
//		int timerlocal = clock64();
	if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %g this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	time_start = clock64();
	if (!iprint) printf("\u2461 **********************************DEVICE:  THREADS  ********************\n\n");
	__syncthreads();
#endif
