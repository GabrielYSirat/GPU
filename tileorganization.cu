#include "hip/hip_runtime.h"
/*
 * tileorganization.cu
 *
 *  Created on: Jul 3, 2017
 *      Author: gabriel
 */
#include "NewLoop.h"

__managed__ int *image_to_scratchpad_offset = { 0 }, *valid_image = { 0 };
__managed__ float *image_to_scratchpad_offset_global = { 0 };
int AmaxLaserx , AmaxLasery , AminLaserx, AminLasery;


bool tileorganization(void) {
	bool Lasertile = TRUE;
	int organization_x[16] = { 0, 1, 2, 3, 2, 2, 3, 3, 4, 3, 3 };
	int organization_y[16] = { 0, 1, 1, 1, 2, 2, 2, 2, 2, 3, 3 };
	int tilex, tiley, tilenumber, posintile, ilasertile;

	filename = resourcesdirectory + "rec_image.xml";
	hipMallocManaged(&image_to_scratchpad_offset, MAXNUMBERLASERTILE * MAXTILE * sizeof(int));
	hipMallocManaged(&valid_image, MAXNUMBERLASERTILE * MAXTILE * sizeof(int));

	// min and max Laser positions rounded to integer of camera pixels
	AmaxLaserx = ceil(TA.maxLaserx); AmaxLasery = ceil(TA.maxLasery);
	AminLaserx = floor(TA.minLaserx); AminLasery = floor(TA.minLasery);

	/************Tiles and aggregates*******************/
	int recdeftile0x = ceil((float) TA.Nb_Cols_reconstruction / XTile);
	int recdeftile0y = ceil((float) TA.Nb_Rows_reconstruction / YTile);
	int laserdeftile0x = CEILING_POS((float )pZOOM*(AmaxLaserx - AminLaserx) / XTile);
	int laserdeftile0y = CEILING_POS((float )pZOOM*(AmaxLasery - AminLasery) / YTile);
	tile.NbTile0x = max(laserdeftile0x, recdeftile0x);
	tile.NbTile0y = max(laserdeftile0y, recdeftile0y);
	if(((AmaxLaserx - AminLaserx) > TA.Nb_Cols_reconstruction) || ((AmaxLasery - AminLasery) > TA.Nb_Rows_reconstruction))
		printf(" INIT PROG \u24FA \u26A0 tiles reconstruction too small!");

	printf(" INIT PROG \u24FA AmaxLaserx %d AmaxLasery %d AminLaserx %d AminLasery %d\n",
			AmaxLaserx, AmaxLasery, AminLaserx, AminLasery);
			printf(" INIT PROG \u24FA recdeftile0x %d recdeftile0y %d laserdeftile0x %d laserdeftile0y %d\n",
			recdeftile0x, recdeftile0y, laserdeftile0x, laserdeftile0y);
	printf(" INIT PROG \u24FA Min Number of tiles x:  %d y: %d\n", tile.NbTile0x, tile.NbTile0y);

	/*************************Aggregates organization depending on MP*******/
	TA.MP_perdistrib = TA.MP / Ndistrib;
	printf(" INIT PROG \u24FA Total number of MP per distribution %d  ", TA.MP_perdistrib);
	printf("  organized as x:%d,  y:%d \n", organization_x[TA.MP_perdistrib],
			organization_y[TA.MP_perdistrib]);

	/************************Aggregates********************************************/
	tile.NbAggregx = ceil((float) tile.NbTile0x / organization_x[TA.MP_perdistrib]);
	tile.NbAggregy = ceil((float) tile.NbTile0y / organization_y[TA.MP_perdistrib]);
	printf(" INIT PROG \u24FA Number of aggregates x:%d y:%d  \n", tile.NbAggregx, tile.NbAggregy);

	/**********************************Tiles****************************************/
	if (tile.NbAggregx == 1) {
		tile.NbTilex = tile.NbTile0x;
		tile.tileperaggregatex = tile.NbTile0x;
	} else {
		tile.NbTilex = tile.NbAggregx * organization_x[TA.MP_perdistrib];
		tile.tileperaggregatex = organization_x[TA.MP_perdistrib];
	}

	if (tile.NbAggregy == 1) {
		tile.NbTiley = tile.NbTile0y;
		tile.tileperaggregatey = tile.NbTile0y;
	} else {
		tile.NbTiley = tile.NbAggregy * organization_y[TA.MP_perdistrib];
		tile.tileperaggregatey = organization_y[TA.MP_perdistrib]  ;
	}

	tile.NbTile = tile.NbTilex * tile.NbTiley * Ndistrib;
	/** FUTURE: In the real application the reconstruction
	 * is created by the program and not read from a file
	 * in this case the size data will be consistent by design
	 */
	// can be improved depending on the ratio between TILE0 and TILE: minor
	tile.reconstructionsizex = tile.NbTilex * XTile; // + (XSCRATCH - XTILE);
	TA.Nb_Cols_reconstruction = tile.reconstructionsizex;
	tile.reconstructionsizey = tile.NbTiley * YTile; //+ (YSCRATCH - YTILE);
	TA.Nb_Rows_reconstruction = tile.reconstructionsizey;
	tile.startx = AminLaserx; //floor(pZOOM*((AminLaserx + AmaxLaserx)/2 - (tile.NbTilex * XTile)/2));
	tile.starty = AminLasery ; //floor(pZOOM*((AminLasery + AmaxLasery)/2 - (tile.NbTiley * YTile)/2));
	TA.reconstruction_size = TA.Nb_Rows_reconstruction * TA.Nb_Cols_reconstruction;
	printf(" INIT PROG \u24FA Final number of tiles x: %d y: %d distrib %d  \n", tile.NbTilex, tile.NbTiley, Ndistrib);
	printf(" INIT PROG \u24FA Reconstruction size x: %d, y:%d \n", tile.reconstructionsizex, tile.reconstructionsizey);
	printf(" INIT PROG \u24FA NbTile %d start x %d y %d MinLaser %d %d in REC pixels %d %d  \n",
			tile.NbTile, tile.startx, tile.starty, AminLaserx, AminLasery, tile.startx*pZOOM, tile.starty*pZOOM);

	tile.NbLaserTotal = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		for (int iLaser = tile.NbLaserTotal; iLaser < tile.NbLaserTotal + tile.Nblaserperdistribution[idistrib]; iLaser++) {
			// position in tiles, tilex and tiley and overall tile number (including distrib)
			tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
			tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
			tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
			float deltilex = *(PosLaserx + iLaser) * pZOOM - tilex * XTile ;
			float deltiley = *(PosLasery + iLaser) * pZOOM - tiley * YTile;
			float delscratchx = deltilex + (XSCRATCH - XTile) / 2;  // XSCRATCH and XTile are odd
			float delscratchy = deltiley + (YSCRATCH - YTile) / 2;  // ySCRATCH and YTile are odd
			posintile = tile.NbLaserpertile[tilenumber]++;
			ilasertile = tilenumber * MAXNUMBERLASERTILE + posintile;
			valid_image[ilasertile] = 1;
			image_to_scratchpad_offset[ilasertile] = *(offsetFULL + iLaser);
			tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[tilenumber]); // acquiring the max value per tile

			// where will be this microimage in the corresponding tile if posintile is 31, and tilenumber is 8?
			//  this microimage is the microimage with index 31 (the indexes begin at 0) of tile of index 8
			// add 1 - to go to 32 - to NbLaserpertile, because we added an image

			if (VERBOSE)
				printf(
						"TILE ORG \u24FA POS IN SCRATCH: numeral %d laser pos in x %f in y: %f  tile x: %d y: %d \n"
								"TILE ORG \u24FA POS IN SCRATCH: deltile x: %f and y %f del scratch x:%f y:%f\n"
								"TILE ORG \u24FA POS IN SCRATCH: ilasertile %d SCRATCH POSITION %d\n"
								"********************ilasertile %d offset scratchpad interaction****************** %d\n",
						iLaser, *(PosLaserx + iLaser), *(PosLasery + iLaser), tilex, tiley, deltilex,
						deltiley, delscratchx, delscratchy, ilasertile,
						image_to_scratchpad_offset[ilasertile], ilasertile,
						image_to_scratchpad_offset[ilasertile]);

			if (VERBOSE)
				printf("TILE ORG \u24FA POS IN SCRATCH: image number %d tilenumber %d position in tile %d\n",
						iLaser, tilenumber, posintile);
		}
		printf("TILE ORG \u24FA  idistrib n°%d number of laser positions in tile in distribution %d\n",
				idistrib, tile.Nblaserperdistribution[idistrib]);
		if (VERBOSE){
		int it0 = tile.NbTilex*tile.NbTiley*idistrib;
		for(int it=it0; it < it0 +tile.NbTilex*tile.NbTiley; it++)
			printf("tile %d: #lasers %d ...\n", it, tile.NbLaserpertile[it]);
		printf("\n");
		}
		tile.NbLaserTotal += tile.Nblaserperdistribution[idistrib];
	}
	printf("TILE ORG \u24FA  tile.NbLaserTotal %d \n",tile.NbLaserTotal);

	for (int it1 = 0; it1 < tile.NbTile; it1++) {
		if (VERBOSE)
			printf("TILE ORG \u24FA Tile number %d tile in x %d tile in y %d distrib %d number of microimages %d\n",
					it1, it1 % (Ndistrib * tile.NbTiley), (it1 / tile.NbTilex) % Ndistrib,
					it1 / (tile.NbTilex * tile.NbTiley), tile.NbLaserpertile[it1]);

		tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[it1]);  // acquiring the max value per tile
		tile.minLaserintile = min(tile.minLaserintile, tile.NbLaserpertile[it1]);  // acquiring the min value per tile
	}

	printf("TILE ORG \u24FA Max  %d and Min %d  Laser in tile ...   \n", tile.maxLaserintile, tile.minLaserintile);
	tile.maxLaserintile = CEILING_POS(((float)tile.maxLaserintile)/NIMAGESPARALLEL) * NIMAGESPARALLEL;
	tile.blocks = tile.maxLaserintile / NIMAGESPARALLEL;
	// rounded to next multiple of NIMAGESPARALLEL
	printf("TILE ORG \u24FA Max Laser in tile rounded to next multiple of NIMAGESPARALLEL  .. %d\n",
			tile.maxLaserintile);

return (Lasertile);
}

bool initializesimusData(void) {
// Initialize new simus and Data
	int tempa = tile.maxLaserintile * tile.NbTile * NThreads;
	printf("TILE ORG \u2466 size simus %d AminLaserx %d AminLasery %d\n", tempa, AminLaserx, AminLasery);
	hipMallocManaged(&new_simus, tempa * sizeof(float));
	hipMallocManaged(&Data, tempa * sizeof(float));
	hipMallocManaged(&Rfactor, tempa * sizeof(float));
	hipMallocManaged(&distribvalidGPU, TA.MP * PSFZOOMSQUARE * sizeof(float));
	for (int itemp = 0; itemp < Ndistrib * PSFZOOMSQUARE; itemp++) *(distribvalidGPU + itemp) = 0.0;

	for (int ii = 0; ii < tempa; ii++) {
		new_simus[ii] = 0.0f;
		Data[ii] = 0.0f;
		Rfactor[ii] = 0.0f;
	}
	return(TRUE);
}

bool microimagesintile(void) {
	float Maxdata = 0.0f;
	int n_colintern = PixZoom * tile.blocks * tile.NbTilex;
	int n_rowintern = PixZoom * NIMAGESPARALLEL * tile.NbTiley;
	int tilex, tiley, tilenumber, ilasertile;

printf("HOST: \u2466 DATA:  n_rowintern %d n_colintern %d, total %d Max data %g\n",
		n_rowintern, n_colintern, tile.maxLaserintile * NThreads, Maxdata);

int disdelta = 0;
for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
	for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
		tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
		tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
		tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
		ilasertile = tilenumber * tile.maxLaserintile + tile.NbLaserpertile[tilenumber];
		printf("temp idistrib %d, iLaser %d tilex %d tiley %d, tilenumber %d ilasertiel %d\n",
				idistrib, iLaser, tilex, tiley, tilenumber, ilasertile);
		for(int ipix = 0; ipix < PixZoomSquare; ipix++) // copy microimage to its position in the Data
			*(Data + ilasertile*PixZoomSquare + ipix) =  *(zoomed_microimages + iLaser* PixZoomSquare + ipix);
	}
	disdelta += tile.Nblaserperdistribution[idistrib];
	printf("TILE ORG \u24FA Max Laser in tile rounded to multiple NIMAGESPARALLEL  .. %d\n", tile.maxLaserintile);
}

const char * DataFile = "results/DataFile.pgm";
unsigned char *i_data = (unsigned char *) calloc(n_colintern * n_rowintern, sizeof(unsigned char)); // on host
for (int i = 0; i < tile.maxLaserintile * NThreads; i++) Maxdata = max(Maxdata, *(Data + i));
printf("HOST: \u277D DATA:  n_rowintern %d n_colintern %d, total %d Max data %g\n",
		n_rowintern, n_colintern, tile.maxLaserintile * NThreads, Maxdata);

bool micimintile = FALSE;
for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {

	for (int j_rowintern = 0; j_rowintern < n_rowintern; j_rowintern++)
		for (int i_colintern = 0; i_colintern < n_colintern; i_colintern++) {
			int i_microimage = i_colintern % PixZoom;
			int i_blocknumber = (i_colintern % tile.NbTilex) / PixZoom;
			int i_tile = i_colintern / (PixZoom * tile.blocks);
			int j_microimage = j_rowintern % PixZoom;
			int j_positioninblock = (j_rowintern % tile.NbTiley) / PixZoom;
			int j_tile = j_rowintern / (PixZoom * NIMAGESPARALLEL);

			int i = i_microimage + j_microimage * PixZoom
					+ (j_positioninblock + i_blocknumber * NIMAGESPARALLEL) * PixZoomSquare // microimage
					+ (i_tile + j_tile * tile.NbTilex) * PixZoomSquare * tile.maxLaserintile; // list of microimages
			int tempi = 255.0 * Data[i] / Maxdata;
			i_data[(i_microimage + i_blocknumber * PixZoom + i_tile * PixZoom * tile.blocks) + 			// x value
					(j_microimage + j_positioninblock * PixZoom + j_tile * PixZoom * NIMAGESPARALLEL) 	// y value
					* PixZoom * tile.blocks] = tempi;
		}
	printf("HOST: \u277D DEVICE TEST in biginspect.cu: Path to calculated new simulations %s .....\n", DataFile);
	sdkSavePGM(DataFile, i_data, n_colintern, n_rowintern);
}
return (micimintile);
}

