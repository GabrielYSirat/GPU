#include "hip/hip_runtime.h"
/*
 * tileorganization.cu
 *
 *  Created on: Jul 3, 2017
 *      Author: gabriel
 */
#include "NewLoop.h"

__managed__ int *image_to_scratchpad_offset = { 0 }, *valid_image = { 0 };
__managed__ float *image_to_scratchpad_offset_global = { 0 };
int AmaxLaserx , AmaxLasery , AminLaserx, AminLasery;


bool tileorganization(void) {
	bool Lasertile = TRUE;
	int organization_x[16] = { 0, 1, 2, 3, 2, 2, 3, 3, 4, 3, 3 };
	int organization_y[16] = { 0, 1, 1, 1, 2, 2, 2, 2, 2, 3, 3 };
	int tilex, tiley, tilenumber, ilasertile;

	filename = resourcesdirectory + "rec_image.xml";
	hipMallocManaged(&image_to_scratchpad_offset, MAXNUMBERLASERTILE * MAXTILE * sizeof(int));
	hipMallocManaged(&valid_image, MAXNUMBERLASERTILE * MAXTILE * sizeof(int));

	// min and max Laser positions rounded to integer of camera pixels
	AmaxLaserx = ceil(TA.maxLaserx); AmaxLasery = ceil(TA.maxLasery);
	AminLaserx = floor(TA.minLaserx); AminLasery = floor(TA.minLasery);

	/************Tiles and aggregates*******************/
	int recdeftile0x = ceil((float) TA.Nb_Cols_reconstruction / XTile);
	int recdeftile0y = ceil((float) TA.Nb_Rows_reconstruction / YTile);
	int laserdeftile0x = CEILING_POS((float )pZOOM*(AmaxLaserx - AminLaserx) / XTile);
	int laserdeftile0y = CEILING_POS((float )pZOOM*(AmaxLasery - AminLasery) / YTile);
	tile.NbTile0x = max(laserdeftile0x, recdeftile0x);
	tile.NbTile0y = max(laserdeftile0y, recdeftile0y);
	if(((AmaxLaserx - AminLaserx) > TA.Nb_Cols_reconstruction) || ((AmaxLasery - AminLasery) > TA.Nb_Rows_reconstruction))
		printf(" INIT PROG \u24FA \u26A0 tiles reconstruction too small!");

	printf(" INIT PROG \u24FA AmaxLaserx %d AmaxLasery %d AminLaserx %d AminLasery %d\n",
			AmaxLaserx, AmaxLasery, AminLaserx, AminLasery);
			printf(" INIT PROG \u24FA recdeftile0x %d recdeftile0y %d laserdeftile0x %d laserdeftile0y %d\n",
			recdeftile0x, recdeftile0y, laserdeftile0x, laserdeftile0y);
	printf(" INIT PROG \u24FA Min Number of tiles x:  %d y: %d\n", tile.NbTile0x, tile.NbTile0y);

	/*************************Aggregates organization depending on MP*******/
	TA.MP_perdistrib = TA.MP / Ndistrib;
	printf(" INIT PROG \u24FA Total number of MP per distribution %d  ", TA.MP_perdistrib);
	printf("  organized as x:%d,  y:%d \n", organization_x[TA.MP_perdistrib],
			organization_y[TA.MP_perdistrib]);

	/************************Aggregates********************************************/
	tile.NbAggregx = ceil((float) tile.NbTile0x / organization_x[TA.MP_perdistrib]);
	tile.NbAggregy = ceil((float) tile.NbTile0y / organization_y[TA.MP_perdistrib]);
	printf(" INIT PROG \u24FA Number of aggregates x:%d y:%d  \n", tile.NbAggregx, tile.NbAggregy);

	/**********************************Tiles****************************************/
	if (tile.NbAggregx == 1) {
		tile.NbTilex = tile.NbTile0x;
		tile.tileperaggregatex = tile.NbTile0x;
	} else {
		tile.NbTilex = tile.NbAggregx * organization_x[TA.MP_perdistrib];
		tile.tileperaggregatex = organization_x[TA.MP_perdistrib];
	}

	if (tile.NbAggregy == 1) {
		tile.NbTiley = tile.NbTile0y;
		tile.tileperaggregatey = tile.NbTile0y;
	} else {
		tile.NbTiley = tile.NbAggregy * organization_y[TA.MP_perdistrib];
		tile.tileperaggregatey = organization_y[TA.MP_perdistrib]  ;
	}

	tile.NbTile = tile.NbTilex * tile.NbTiley * Ndistrib;
	/** FUTURE: In the real application the reconstruction
	 * is created by the program and not read from a file
	 * in this case the size data will be consistent by design
	 */
	// can be improved depending on the ratio between TILE0 and TILE: minor
	tile.reconstructionsizex = tile.NbTilex * XTile; // + (XSCRATCH - XTILE);
	TA.Nb_Cols_reconstruction = tile.reconstructionsizex;
	tile.reconstructionsizey = tile.NbTiley * YTile; //+ (YSCRATCH - YTILE);
	TA.Nb_Rows_reconstruction = tile.reconstructionsizey;
	tile.startx = AminLaserx; //floor(pZOOM*((AminLaserx + AmaxLaserx)/2 - (tile.NbTilex * XTile)/2));
	tile.starty = AminLasery ; //floor(pZOOM*((AminLasery + AmaxLasery)/2 - (tile.NbTiley * YTile)/2));
	TA.reconstruction_size = TA.Nb_Rows_reconstruction * TA.Nb_Cols_reconstruction;
	printf(" INIT PROG \u24FA Final number of tiles x: %d y: %d distrib %d  \n", tile.NbTilex, tile.NbTiley, Ndistrib);
	printf(" INIT PROG \u24FA Reconstruction size x: %d, y:%d \n", tile.reconstructionsizex, tile.reconstructionsizey);
	printf(" INIT PROG \u24FA NbTile %d start x %d y %d MinLaser %d %d in REC pixels %d %d  \n",
			tile.NbTile, tile.startx, tile.starty, AminLaserx, AminLasery, tile.startx*pZOOM, tile.starty*pZOOM);

	tile.NbLaserTotal = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		for (int iLaser = tile.NbLaserTotal; iLaser < tile.NbLaserTotal + tile.Nblaserperdistribution[idistrib]; iLaser++) {
			// position in tiles, tilex and tiley and overall tile number (including distrib)
			tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
			tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
			tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
			float deltilex = *(PosLaserx + iLaser) * pZOOM - tilex * XTile ;
			float deltiley = *(PosLasery + iLaser) * pZOOM - tiley * YTile;
			float delscratchx = deltilex + (XSCRATCH - XTile) / 2;  // XSCRATCH and XTile are odd
			float delscratchy = deltiley + (YSCRATCH - YTile) / 2;  // ySCRATCH and YTile are odd
			tile.posintile[iLaser] = tile.NbLaserpertile[tilenumber]++;
			if(VERBOSE)
				printf("TILE ORG \u2479 POS IN TILE: iLaser %d,tilenumber %d tile.posintile[iLaser]  %d\n",
						iLaser, tilenumber, tile.posintile[iLaser]);
			ilasertile = tilenumber * MAXNUMBERLASERTILE + tile.posintile[iLaser];
			valid_image[ilasertile] = 1;
			image_to_scratchpad_offset[ilasertile] = *(offsetFULL + iLaser);
			tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[tilenumber]); // acquiring the max value per tile

			// where will be this microimage in the corresponding tile if posintile is 31, and tilenumber is 8?
			//  this microimage is the microimage with index 31 (the indexes begin at 0) of tile of index 8
			// add 1 - to go to 32 - to NbLaserpertile, because we added an image

			if (VERBOSE)
				printf("TILE ORG \u2479 POS IN SCRATCH: numeral %d laser pos in x %f in y: %f  tile x: %d y: %d \n"
								"TILE ORG \u24FA POS IN SCRATCH: deltile x: %f and y %f del scratch x:%f y:%f\n"
								"TILE ORG \u24FA POS IN SCRATCH: ilasertile %d SCRATCH POSITION %d\n"
								"********************ilasertile %d offset scratchpad interaction****************** %d\n",
						iLaser, *(PosLaserx + iLaser), *(PosLasery + iLaser), tilex, tiley, deltilex,
						deltiley, delscratchx, delscratchy, ilasertile,
						image_to_scratchpad_offset[ilasertile], ilasertile,
						image_to_scratchpad_offset[ilasertile]);

			if (VERBOSE)
				printf("TILE ORG \u2479 POS IN SCRATCH: image number %d tilenumber %d position in tile %d\n",
						iLaser, tilenumber, tile.posintile[iLaser]);
		}
		printf("TILE ORG \u24FA  idistrib n°%d number of laser positions in tile in distribution %d\n",
				idistrib, tile.Nblaserperdistribution[idistrib]);
		if (VERBOSE){
		int it0 = tile.NbTilex*tile.NbTiley*idistrib;
		for(int it=it0; it < it0 +tile.NbTilex*tile.NbTiley; it++)
			printf(" \u2479 tile %d: #lasers %d ...\n", it, tile.NbLaserpertile[it]);
		printf("\n");
		}
		tile.NbLaserTotal += tile.Nblaserperdistribution[idistrib];
	}
	printf("TILE ORG \u24FA  tile.NbLaserTotal %d \n",tile.NbLaserTotal);

	for (int it1 = 0; it1 < tile.NbTile; it1++) {
		if (VERBOSE)
			printf("TILE ORG \u2479 Tile number %d tile in x %d tile in y %d distrib %d number of microimages %d\n",
					it1, it1 % (Ndistrib * tile.NbTiley), (it1 / tile.NbTilex) % Ndistrib,
					it1 / (tile.NbTilex * tile.NbTiley), tile.NbLaserpertile[it1]);

		tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[it1]);  // acquiring the max value per tile
		tile.minLaserintile = min(tile.minLaserintile, tile.NbLaserpertile[it1]);  // acquiring the min value per tile
	}

	printf("TILE ORG \u24FA Max  %d and Min %d  Laser in tile ...   \n", tile.maxLaserintile, tile.minLaserintile);
	tile.maxLaserintile = CEILING_POS(((float)tile.maxLaserintile)/NIMAGESPARALLEL) * NIMAGESPARALLEL;
	tile.blocks = tile.maxLaserintile / NIMAGESPARALLEL;
	// rounded to next multiple of NIMAGESPARALLEL
	printf("TILE ORG \u24FA Max Laser in tile rounded to next multiple of NIMAGESPARALLEL  .. %d\n",
			tile.maxLaserintile);

return (Lasertile);
}

bool initializesimusData(void) {
// Initialize new simus and Data
	int tempa = tile.maxLaserintile * tile.NbTile * NThreads;
	printf("TILE ORG \u2466 size simus %d AminLaserx %d AminLasery %d\n", tempa, AminLaserx, AminLasery);
	hipMallocManaged(&new_simus, tempa * sizeof(float));
	hipMallocManaged(&Data, tempa * sizeof(float));
	hipMallocManaged(&Rfactor, tempa * sizeof(float));
	hipMallocManaged(&distribvalidGPU, TA.MP * PSFZOOMSQUARE * sizeof(float));
	for (int itemp = 0; itemp < Ndistrib * PSFZOOMSQUARE; itemp++) *(distribvalidGPU + itemp) = 0.0;

	for (int ii = 0; ii < tempa; ii++) {
		new_simus[ii] = 0.0f;
		Data[ii] = 0.0f;
		Rfactor[ii] = 0.0f;
	}
	return(TRUE);
}

bool microimagesintile(void) {
	bool micimintile = FALSE;

printf("TILE ORG \u24FA Max Laser in tile rounded to multiple NIMAGESPARALLEL  .. %d MaxMicroimages %f MinMicroimages %f\n", tile.maxLaserintile, Maxmicroimages, Minmicroimages);
unsigned char *i_data = (unsigned char *) calloc(PixZoomSquare * tile.NbTile * tile.maxLaserintile, sizeof(unsigned char)); // on host
const char * DataFile = "results/DataFile.pgm";

	for (int idistrib = 0, disdelta = 0; idistrib < Ndistrib;
			idistrib++, disdelta += tile.Nblaserperdistribution[idistrib])
		for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
			int tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
			int tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
			int tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
			int ilasertile = tilenumber * tile.maxLaserintile + tile.posintile[iLaser];
			printf("TILE ORG \u2479 idistrib %d, iLaser %d tilenumber %d ilasertile %d\n", idistrib, iLaser,
					tilenumber, ilasertile);
			for (int ipix = 0; ipix < PixZoomSquare; ipix++) { // copy microimage to its position in the Data
				*(Data + ilasertile * PixZoomSquare + ipix) = *(zoomed_microimages + iLaser * PixZoomSquare + ipix);
				int xpix = ipix%PixZoom; int ypix = ipix/PixZoom;
				i_data[xpix + ypix * PixZoom * tile.maxLaserintile + ilasertile%tile.maxLaserintile * PixZoomSquare + (ilasertile/tile.maxLaserintile) * tile.maxLaserintile * PixZoomSquare]
				       = 255.0 * (*(Data + ilasertile * PixZoomSquare + ipix) - Minmicroimages)
						/(Maxmicroimages - Minmicroimages);
			}
		}

	printf("HOST: \u277D DEVICE TEST in biginspect.cu: Path to calculated new simulations %s .....\n", DataFile);
	sdkSavePGM(DataFile, i_data, PixZoom * tile.maxLaserintile, tile.NbTile * PixZoom);

return (micimintile);
}

