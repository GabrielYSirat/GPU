#include "hip/hip_runtime.h"
/*
 * tileorganization.cu
 *
 *  Created on: Jul 3, 2017
 *      Author: gabriel
 */
#include "NewLoop.h"
#define TEST  1

__managed__ int *image_to_scratchpad_offset = { 0 }, *valid_image = { 0 };
__managed__ float *image_to_scratchpad_offset_global = { 0 };
__managed__ float *new_simus = { 0 }, *Data = { 0 }, *Rfactor = { 0 }, *distribvalidGPU = { 0 };

bool tileorganization(void) {
	bool Lasertile = TRUE;
	int organization_x[16] = { 0, 1, 2, 3, 2, 3, 3, 3, 4, 3, 3 };
	int organization_y[16] = { 0, 1, 1, 1, 2, 1, 2, 2, 2, 3, 3 };
	int tilex, tiley, tilenumber, posintile, ilasertile;

	hipMallocManaged(&image_to_scratchpad_offset, MAXNUMBERLASERTILE * MAXTILE * sizeof(int));
	hipMallocManaged(&valid_image, MAXNUMBERLASERTILE * MAXTILE * sizeof(int));

	// min and max Laser positions rounded to integer of camera pixels
	int AmaxLaserx = ceil(TA.maxLaserx);
	int AmaxLasery = ceil(TA.maxLasery);
	int AminLaserx = floor(TA.minLaserx);
	int AminLasery = floor(TA.minLasery);
	/************Tiles and aggregates*******************/
	printf(" ***********tiles and aggregates************** \n");
	filename = resourcesdirectory + "rec_image.xml";
	printf(" INIT PROG \u24FA tiles & aggregates:   \n");

	int temptile0x = TA.Nb_Cols_reconstruction / XTile;
	int temptile0y = TA.Nb_Rows_reconstruction / YTile;
	tile.NbTile0x = CEILING_POS((float )pZOOM*(AmaxLaserx - AminLaserx) / XTile);
	tile.NbTile0x = max(tile.NbTile0x, temptile0x);
	tile.NbTile0y = CEILING_POS((float )pZOOM*(AmaxLasery - AminLasery) / YTile);
	tile.NbTile0y = max(tile.NbTile0y, temptile0y);
	if(((AmaxLaserx - AminLaserx) > TA.Nb_Cols_reconstruction) ||
			((AmaxLasery - AminLasery) > TA.Nb_Rows_reconstruction))
		printf(" INIT PROG \u24FA \u26A0 tiles reconstruction too small!");

	printf(" INIT PROG \u24FA Min Number of tiles x:  %d y: %d\n ", tile.NbTile0x, tile.NbTile0y);

	/*************************Aggregates organization depending on MP*******/
	TA.MP_perdistrib = TA.MP / Ndistrib;
	printf("INIT PROG \u24FA Total number of MP per distribution %d  ", TA.MP_perdistrib);
	printf("  organized as x:%d,  y:%d \n", organization_x[TA.MP_perdistrib],
			organization_y[TA.MP_perdistrib]);

	/************************Aggregates********************************************/
	tile.NbAggregx = ceil((float) tile.NbTile0x / organization_x[TA.MP_perdistrib]);
	tile.NbAggregy = ceil((float) tile.NbTile0y / organization_y[TA.MP_perdistrib]);
	printf(" INIT PROG \u24FA Number of aggregates x:%d y:%d  \n", tile.NbAggregx, tile.NbAggregy);

	/**********************************Tiles****************************************/
	if (tile.NbAggregx == 1) {
		tile.NbTilex = tile.NbTile0x;
		tile.tileperaggregatex = tile.NbTile0x;
	} else {
		tile.NbTilex = tile.NbAggregx * organization_x[TA.MP_perdistrib];
		tile.tileperaggregatex = organization_x[TA.MP_perdistrib];
	}

	if (tile.NbAggregy == 1) {
		tile.NbTiley = tile.NbTile0y;
		tile.tileperaggregatey = tile.NbTile0y;
	} else {
		tile.NbTiley = tile.NbAggregy * organization_y[TA.MP_perdistrib];
		tile.tileperaggregatey = organization_y[TA.MP_perdistrib];
	}

	tile.NbTile = tile.NbTilex * tile.NbTiley * Ndistrib;
	/** FUTURE: In the real application the reconstruction
	 * is created by the program and not read from a file
	 * in this case the size data will be consistent by design
	 */
	// can be improved depending on the ratio between TILE0 and TILE: minor
	tile.reconstructionsizex = tile.NbTilex * XTile; // + (XSCRATCH - XTILE);
	TA.Nb_Cols_reconstruction = tile.reconstructionsizex;
	tile.reconstructionsizey = tile.NbTiley * YTile; //+ (YSCRATCH - YTILE);
	TA.Nb_Rows_reconstruction = tile.reconstructionsizey;
	tile.startx = AminLaserx; //floor(pZOOM*((AminLaserx + AmaxLaserx)/2 - (tile.NbTilex * XTile)/2));
	tile.starty = AminLasery ; //floor(pZOOM*((AminLasery + AmaxLasery)/2 - (tile.NbTiley * YTile)/2));
	printf("INIT PROG \u24FA start x %d y %d MinLaser %d %d in REC pixels %d %d  \n",
			tile.startx, tile.starty, AminLaserx, AminLasery, tile.startx*pZOOM, tile.starty*pZOOM);


	TA.reconstruction_size = TA.Nb_Rows_reconstruction * TA.Nb_Cols_reconstruction;
	printf(" INIT PROG \u24FA Reconstruction size x: %d, y:%d \n", tile.reconstructionsizex, tile.reconstructionsizey);
	printf(" INIT PROG \u24FA Final number of tiles x: %d y: %d  \n\n", tile.NbTilex, tile.NbTiley);

	int disdelta = 0;
	for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
		for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
			// ATTENTION: REDEFINE BY THE TILE ORIGIN!!
			// position in tiles, tilex and tiley and overall tile number (including distrib)
			tilex = pZOOM * (*(PosLaserx + iLaser) - tile.startx) / XTile;
			tiley = pZOOM * (*(PosLasery + iLaser) - tile.starty) / YTile;
			tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
			float deltilex = *(PosLaserx + iLaser) * pZOOM - tilex * XTile;
			float deltiley = *(PosLasery + iLaser) * pZOOM - tiley * YTile;
			float delscratchx = deltilex + (XSCRATCH - XTile) / 2;  // XSCRATCH and XTile are odd
			float delscratchy = deltiley + (YSCRATCH - YTile) / 2;  // ySCRATCH and YTile are odd
			posintile = tile.NbLaserpertile[tilenumber]++;
			ilasertile = tilenumber * MAXNUMBERLASERTILE + posintile;
			valid_image[ilasertile] = 1;
			image_to_scratchpad_offset[ilasertile] = *(offsetFULL + iLaser);
			tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[tilenumber]); // acquiring the max value per tile

			// where will be this microimage in the corresponding tile if posintile is 31, and tilenumber is 8?
			//  this microimage is the microimage with index 31 (the indexes begin at 0) of tile of index 8
			// add 1 - to go to 32 - to NbLaserpertile, because we added an image

			printf("TILE ORG \u24FA POS IN SCRATCH: numeral %d laser pos in x %f in y: %f  tile x: %d y: %d \n"
					"TILE ORG \u24FA POS IN SCRATCH: deltile x: %f and y %f del scratch x:%f y:%f\n"
					"TILE ORG \u24FA POS IN SCRATCH: ilasertile %d SCRATCH POSITION %d\n"
					"********************ilasertile %d offset scratchpad interaction****************** %d\n", iLaser,
					*(PosLaserx + iLaser), *(PosLasery + iLaser), tilex, tiley, deltilex, deltiley, delscratchx,
					delscratchy, ilasertile, image_to_scratchpad_offset[ilasertile], ilasertile,
					image_to_scratchpad_offset[ilasertile]);

			printf("TILE ORG \u24FA POS IN SCRATCH: image number %d tilenumber %d position in tile %d\n", iLaser,
					tilenumber, posintile);
		}
		disdelta += tile.Nblaserperdistribution[idistrib];

		printf("TILE ORG \u24FA idistrib %d  Nb laser per distribution %d\n", idistrib,
				tile.Nblaserperdistribution[idistrib]);
	}

	for (int it1 = 0; it1 < tile.NbTile; it1++) {
		printf("TILE ORG \u24FA Tile number %d tile in x %d tile in y %d distrib %d number of microimages %d\n", it1,
				it1 % (Ndistrib * tile.NbTiley), (it1 / tile.NbTilex) % Ndistrib, it1 / (tile.NbTilex * tile.NbTiley),
				tile.NbLaserpertile[it1]);

		tile.maxLaserintile = max(tile.maxLaserintile, tile.NbLaserpertile[it1]);  // acquiring the max value per tile
		tile.minLaserintile = min(tile.minLaserintile, tile.NbLaserpertile[it1]);  // acquiring the min value per tile
	}

	printf("TILE ORG \u24FA Max  %d and Min %d  Laser in tile ...   \n", tile.maxLaserintile, tile.minLaserintile);
	tile.maxLaserintile = CEILING_POS(((float)tile.maxLaserintile)/NIMAGESPARALLEL) * NIMAGESPARALLEL;
	tile.blocks = tile.maxLaserintile / NIMAGESPARALLEL;
	// rounded to next multiple of NIMAGESPARALLEL
	printf("TILE ORG \u24FA Max Laser in tile rounded to next multiple of NIMAGESPARALLEL  .. %d\n",
			tile.maxLaserintile);

return (Lasertile);
}

bool microimagesintile(void) {
float Maxdata = 0.0f;
int n_colintern = PixZoom * tile.blocks * tile.NbTilex;
int n_rowintern = PixZoom * NIMAGESPARALLEL * tile.NbTiley;
int tilex, tiley, tilenumber, ilasertile;
int AminLaserx = floor(TA.minLaserx);
int AminLasery = floor(TA.minLasery);

// Initialize new simus and Data
int tempa = tile.maxLaserintile * tile.NbTile * NThreads;
printf("TILE ORG \u2466 size simus %d \n", tempa);
hipMallocManaged(&new_simus, tempa*sizeof(float));
hipMallocManaged(&Data, tempa*sizeof(float));
hipMallocManaged(&Rfactor, tempa*sizeof(float));
hipMallocManaged(&distribvalidGPU, PSFZOOMSQUARE*sizeof(float));

for (int ii = 0; ii < tempa; ii++) {
	new_simus[ii] = 0.0f;
	Data[ii] = 0.0f;
	Rfactor[ii] = 0.0f;
}

printf("HOST: \u2466 DATA:  n_rowintern %d n_colintern %d, total %d Max data %g\n",
		n_rowintern, n_colintern, tile.maxLaserintile * NThreads, Maxdata);

int disdelta = 0;
for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {
	for (int iLaser = disdelta; iLaser < disdelta + tile.Nblaserperdistribution[idistrib]; iLaser++) {
		tilex = pZOOM * (*(PosLaserx + iLaser) - AminLaserx) / XTile;
		tiley = pZOOM * (*(PosLasery + iLaser) - AminLasery) / YTile;
		tilenumber = tilex + tile.NbTilex * tiley + tile.NbTilex * tile.NbTiley * idistrib;
		ilasertile = tilenumber * MAXNUMBERLASERTILE + tile.NbLaserpertile[tilenumber];
		for(int ipix = 0; ipix < PixZoomSquare; ipix++) // copy microimage to its position in the Data
			*(Data + ilasertile*PixZoomSquare + ipix) =  *(zoomed_microimages + iLaser* PixZoomSquare + ipix);
	}
	disdelta += tile.Nblaserperdistribution[idistrib];
	printf("TILE ORG \u24FA Max Laser in tile rounded to multiple NIMAGESPARALLEL  .. %d\n", tile.maxLaserintile);
}

const char * DataFile = "results/DataFile.pgm";
unsigned char *i_data = (unsigned char *) calloc(n_colintern * n_rowintern, sizeof(unsigned char)); // on host
for (int i = 0; i < tile.maxLaserintile * NThreads; i++) Maxdata = max(Maxdata, *(Data + i));
printf("HOST: \u277D DATA:  n_rowintern %d n_colintern %d, total %d Max data %g\n",
		n_rowintern, n_colintern, tile.maxLaserintile * NThreads, Maxdata);

bool micimintile = FALSE;
for (int idistrib = 0; idistrib < Ndistrib; idistrib++) {

	for (int j_rowintern = 0; j_rowintern < n_rowintern; j_rowintern++)
		for (int i_colintern = 0; i_colintern < n_colintern; i_colintern++) {
			int i_microimage = i_colintern % PixZoom;
			int i_blocknumber = (i_colintern % tile.NbTilex) / PixZoom;
			int i_tile = i_colintern / (PixZoom * tile.blocks);
			int j_microimage = j_rowintern % PixZoom;
			int j_positioninblock = (j_rowintern % tile.NbTiley) / PixZoom;
			int j_tile = j_rowintern / (PixZoom * NIMAGESPARALLEL);

			int i = i_microimage + j_microimage * PixZoom
					+ (j_positioninblock + i_blocknumber * NIMAGESPARALLEL) * PixZoomSquare // microimage
					+ (i_tile + j_tile * tile.NbTilex) * PixZoomSquare * tile.maxLaserintile; // list of microimages
			int tempi = 255.0 * Data[i] / Maxdata;
			i_data[(i_microimage + i_blocknumber * PixZoom + i_tile * PixZoom * tile.blocks) + 			// x value
					(j_microimage + j_positioninblock * PixZoom + j_tile * PixZoom * NIMAGESPARALLEL) 	// y value
					* PixZoom * tile.blocks] = tempi;
		}
	printf("HOST: \u277D DEVICE TEST in biginspect.cu: Path to calculated new simulations %s .....\n", DataFile);
	sdkSavePGM(DataFile, i_data, n_colintern, n_rowintern);
}
return (micimintile);
}

