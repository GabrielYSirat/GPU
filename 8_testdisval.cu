
#include <hip/hip_runtime.h>

#ifdef 	TESTDISVAL
float epsilon = 0.5e-4, valmax = 100;
if(valid_image[iblockima]){
	if (*(pscratch_0[iblockima]) > valmax && PSFDISVAL[0] > epsilon)
			printf("\u2465 NFZ0 iblockima %d iglobal %d ithreads %d PSFpos %d  ipixel %d jpixel %d val %g PSFDISVAL[0] %g scratch %g new_simus %12.5g \n",
				iblockima, iglobal, ithreads, PSFpos, ipixel[0], jpixel[0], tmp_0, PSFDISVAL[0], *(pscratch_0[iblockima]), NSIF_0[iblockima]);
	if (*(pscratch_1[iblockima]) > valmax && PSFDISVAL[1] > epsilon)
			printf("\u2465 NFZ1 iblockima %d iglobal %d ithreads %d PSFpos %d  ipixel %d jpixel %d val %g PSFDISVAL[3] %g scratch %g  new_simus %8.5f \n",
				iblockima, iglobal, ithreads, PSFpos, ipixel[1], jpixel[1], tmp_1, PSFDISVAL[1], *(pscratch_1[iblockima]), NSIF_1[iblockima]);
	if (*(pscratch_2[iblockima]) > valmax && PSFDISVAL[2] > epsilon)
			printf("\u2465 NFZ2 iblockima %d iglobal %d ithreads %d PSFpos %d  ipixel %d jpixel %d val %g PSFDISVAL[2] %g scratch %g  new_simus %8.5f \n",
					iblockima, iglobal, ithreads, PSFpos, ipixel[2], jpixel[2], tmp_2, PSFDISVAL[2], *(pscratch_2[iblockima]), NSIF_2[iblockima]);
	if (*(pscratch_3[iblockima]) > valmax && PSFDISVAL[3] > epsilon)
			printf("\u2465 NFZ3 iblockima %d iglobal %d ithreads %d PSFpos %d  ipixel %d jpixel %d val %g PSFDISVAL[3] %g scratch %g  new_simus %8.5f \n",
					iblockima, iglobal, ithreads, PSFpos, ipixel[3], jpixel[3], tmp_3, PSFDISVAL[3], *(pscratch_3[iblockima]), NSIF_3[iblockima]);
	}
			__syncthreads();
#endif
