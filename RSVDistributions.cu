#include "hip/hip_runtime.h"
/*
 * readStorevalidatedistrib_control.cu
 *
 *  Created on: Apr 23, 2017
 *      Author: gabriel
 */
#include "NewLoop.h"
int byte_skipped = 16;
float Maxdistrib = 0.0f, Sumdistrib = 0.0f;

void readstoredistrib(void) {
	char * memblock;
	int size;

	string beadraw = resourcesdirectory + "distributions.raw";
	printf(" DISTRIBUTIONS \u2461 distrib:  %s \n", filename.c_str());
	const char * distribImagefile = "results/distribImagefile.pgm";

	unsigned char *i_distrib = (unsigned char *) calloc( YDistrib_extended * XDistrib * Ndistrib, sizeof(unsigned char)); // on host
	hipMallocManaged(&original_distrib, ADistrib * Ndistrib * sizeof(double));
	hipMallocManaged(&double_distrib, YDistrib_extended * XDistrib * Ndistrib * sizeof(float));
	hipMallocManaged(&test_distrib, XDistrib * YDistrib * Ndistrib * sizeof(double));

	//read distrib bin file
	std::ifstream distribile(beadraw.c_str(), ios::in | ios::binary | ios::ate);
	size = (distribile.tellg()); // the data is stored in doubles of 8 bytes in the file
	size -= byte_skipped;  				// removes the 4 first bytes, Why??
	cout << "DISTRIBUTIONS \u2461 function read: size distrib = " << size << endl;
	memblock = new char[size];
	distribile.seekg(byte_skipped, ios::beg); // 4 first bytes are offset
	distribile.read(memblock, size);
	distribile.close();

	double_distrib = (double*) memblock; //reinterpret the chars stored in the file as double

	for (int i = 0; i < ADistrib * Ndistrib; i++) {
		*(original_distrib + i) = *(double_distrib + i);	// change to float
		Sumdistrib += original_distrib[i];
		Maxdistrib = max(Maxdistrib, *(original_distrib + i));
	}
	printf("DISTRIBUTIONS \u2461: Original max %g Sum %g\n", Maxdistrib, Sumdistrib);

	// write distrib image to disk
	/////////////////////////////////
	for (int i = 0; i < YDistrib_extended * XDistrib * Ndistrib; i++)
		i_distrib[i] = 255.0 * original_distrib[i] / Maxdistrib;// image value
	printf(
			" DISTRIBUTIONS \u2461 function read: Path to distrib original %s .....\n",
			distribImagefile);

	sdkSavePGM(distribImagefile, i_distrib, XDistrib, YDistrib_extended * Ndistrib);

	free(i_distrib);

}

bool Distribvalidate_host(void) {
	bool testdistrib;
	double Sum3distrib = 0, max3distrib = 0;

	unsigned char *i_distrib = (unsigned char *) calloc( YDistrib_extended * XDistrib * Ndistrib, sizeof(unsigned char)); // on host
	// write distrib in memory and validate
	hipMallocManaged(&val_distrib, YDistrib_extended * XDistrib * Ndistrib * sizeof(float));
	const char * distribValImagefile = "results/distribValImagefile.pgm";

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the distrib kernel
	validate_distrib<<<dimGrid, dimBlock, 0>>>(YDistrib_extended, XDistrib, Ndistrib);
	hipDeviceSynchronize();

	for (int arg = 0; arg < ADistrib * Ndistrib; arg++)	Sum3distrib += *(val_distrib + arg);
	for (int arg = 0; arg < ADistrib * Ndistrib; arg++) max3distrib = max(max3distrib, *(val_distrib + arg));

	printf("DISTRIBUTIONS \u2461: validation max %g Sum %g\n", max3distrib, Sum3distrib);

	for (int i = 0; i < YDistrib_extended * XDistrib * Ndistrib; i++)
		i_distrib[i] = (255.0 * val_distrib[i]) / max3distrib;// Validation image value

	printf(" DISTRIBUTIONS \u2461 Path to distrib validation %s .....\n", distribValImagefile);

	sdkSavePGM(distribValImagefile, i_distrib, XDistrib, YDistrib_extended * Ndistrib);

	printf(" DISTRIBUTIONS \u2461 Comparing files ... ");
	testdistrib = compareData(val_distrib, original_distrib,
			XDistrib * YDistrib_extended * Ndistrib,
			MAX_EPSILON_ERROR, 0.15f);

	for (int jdistrib = 0; jdistrib < YDistrib_extended * XDistrib * Ndistrib;
			jdistrib++) {
		Sumdel[1] += fabsf(
				*(val_distrib + jdistrib) - *(double_distrib + jdistrib));
	}
	printf("Sumdel[1] %f  ", Sumdel[1]);
	cout << "testdistrib = " << testdistrib << "\n";
	hipFree(val_distrib);
	return (testdistrib);
}

