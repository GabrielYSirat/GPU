#include "hip/hip_runtime.h"
/*
 * readStorevalidatedistrib_control.cu
 *
 *  Created on: Apr 23, 2017
 *      Author: gabriel
 */
#include "NewLoop.h"
int byte_skipped = 16;
float Maxdistrib = 0.0f, Sumdistrib = 0.0f;
std::string DISDATA = "/lambda_488/Calib/distribution";
const char * distribImagefile = "results/distribImagefile.pgm";


void readstoredistrib(void) {
	char * memblock;
	int size;

	unsigned char *ii_distrib = (unsigned char *) calloc( YDistrib_extended * XDistrib * Ndistrib, sizeof(unsigned char)); // on host
	hipMallocManaged(&original_distrib, ADistrib * Ndistrib * sizeof(double));
	hipMallocManaged(&double_distrib, YDistrib_extended * XDistrib * Ndistrib * sizeof(float));
	hipMallocManaged(&test_distrib, XDistrib * YDistrib * Ndistrib * sizeof(double));

	for(int idistrib = 0; idistrib < Ndistrib; idistrib++)
	{
	std::string beadraw = resourcesdirectory + DISDATA + std::to_string(idistrib+1) + ".bin";
	printf("DISTRIBUTIONS \u2461: data file %s\n",beadraw.c_str());

	//read distrib bin file
	std::ifstream distribile(beadraw.c_str(), ios::in | ios::binary | ios::ate);
	size = (distribile.tellg()); // the data is stored in doubles of 8 bytes in the file
	size -= byte_skipped;  				// removes the 4 first bytes, Why??
	cout << "DISTRIBUTIONS \u2461 function read: distribution # " << idistrib << " size distrib = "<< size << endl;
	memblock = new char[size];
	distribile.seekg(byte_skipped, ios::beg); // 4 first bytes are offset
	distribile.read(memblock, size);
	distribile.close();

	double_distrib = (double*) memblock; //reinterpret the chars stored in the file as double

	for (int i = 0; i < ADistrib; i++) {
		*(original_distrib + i + idistrib*XDistrib*YDistrib_extended) = *(double_distrib + i);	// change to float
		Sumdistrib += double_distrib[i];
		Maxdistrib = max(Maxdistrib, *(double_distrib + i));
	}
	printf("DISTRIBUTIONS \u2461: idistrib %d Original max %g Sum %g\n", idistrib, Maxdistrib, Sumdistrib);
	}
	// write distrib image to disk
	/////////////////////////////////
	for (int i = 0; i < YDistrib_extended * XDistrib * Ndistrib; i++)
		ii_distrib[i] = 255.0 * original_distrib[i] / Maxdistrib;// image value
	printf(
			"DISTRIBUTIONS \u2461 function read: Path to distrib original %s .....\n",
			distribImagefile);

	sdkSavePGM(distribImagefile, ii_distrib, XDistrib, YDistrib_extended * Ndistrib);

	free(ii_distrib);

}

bool Distribvalidate_host(void) {
	bool testdistrib;
	double Sum3distrib = 0, max3distrib = 0;

	unsigned char *ii_distrib = (unsigned char *) calloc( YDistrib_extended * XDistrib * Ndistrib, sizeof(unsigned char)); // on host
	// write distrib in memory and validate
	hipMallocManaged(&val_distrib, YDistrib_extended * XDistrib * Ndistrib * sizeof(float));
	const char * distribValImagefile = "results/distribValImagefile.pgm";

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the distrib kernel
	validate_distrib<<<dimGrid, dimBlock, 0>>>(YDistrib_extended, XDistrib, Ndistrib);
	hipDeviceSynchronize();

	for (int arg = 0; arg < ADistrib * Ndistrib; arg++)	Sum3distrib += *(val_distrib + arg);
	for (int arg = 0; arg < ADistrib * Ndistrib; arg++) max3distrib = max(max3distrib, *(val_distrib + arg));

	printf("DISTRIBUTIONS \u2461: validation max %g Sum %g\n", max3distrib, Sum3distrib);

	for (int i = 0; i < YDistrib_extended * XDistrib * Ndistrib; i++)
		ii_distrib[i] = (255.0 * val_distrib[i]) / max3distrib;// Validation image value

	printf(" DISTRIBUTIONS \u2461 Path to distrib validation %s .....\n", distribValImagefile);

	sdkSavePGM(distribValImagefile, ii_distrib, XDistrib, YDistrib_extended * Ndistrib);

	printf(" DISTRIBUTIONS \u2461 Comparing files ... ");
	testdistrib = compareData(val_distrib, original_distrib,
			XDistrib * YDistrib_extended * Ndistrib,
			MAX_EPSILON_ERROR, 0.15f);

	for (int jdistrib = 0; jdistrib < YDistrib_extended * XDistrib * Ndistrib;
			jdistrib++) {
		Sumdel[1] += fabsf(
				*(val_distrib + jdistrib) - *(original_distrib + jdistrib));
	}
	printf("Sumdel[1] %f  ", Sumdel[1]);
	cout << "testdistrib = " << testdistrib << "\n";
	hipFree(val_distrib);
	return (testdistrib);
}

