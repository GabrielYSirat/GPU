#include "hip/hip_runtime.h"
/*
 * readStorevalidatedistrib_control.cu
 *
 *  Created on: Apr 23, 2017
 *      Author: gabriel
 */
#include "0_Mainparameters.h"
int byte_skipped = 16;
float Maxdistrib = 0.0f, Sumdistrib = 0.0f;
std::string DISDATA = "/lambda_488/Calib/distribution";
std::string enddistrib = ".bin";
const char * distribImagefile = "results/B_distributions.pgm";

void readstoredistrib(void) {
	char * memblock;
	int size;

	unsigned char *ii_distrib = (unsigned char *) calloc( YDistrib_extended * XDistrib * Ndistrib, sizeof(unsigned char)); // on host
	hipMallocManaged(&original_distrib, ADistrib * Ndistrib * sizeof(float));
	hipMallocManaged(&test2_distrib, ADistrib * TA.MP * sizeof(float));
	hipMallocManaged(&double_distrib, YDistrib_extended * XDistrib * Ndistrib * sizeof(double));
	verbosefile << endl ;
	for(int idistrib = 0; idistrib < Ndistrib; idistrib++)
	{
	std::string beadraw = resourcesdirectory + DISDATA + std::to_string(idistrib+1) + enddistrib;
	verbosefile << "DISTRIBUTIONS \u2461: data file " << beadraw.c_str() << endl; //read distrib bin file
		std::ifstream distribile(beadraw.c_str(), ios::in | ios::binary | ios::ate);
	size = (distribile.tellg()); // the data is stored in doubles of 8 bytes in the file
	size -= byte_skipped;  				// removes the "bytes skipped"
	verbosefile << "DISTRIBUTIONS \u2461 function read: distribution # " << idistrib
			<< " size distrib = "<< size << " equal to XDistrib*YDistrib*number of bytes " << endl;
	verbosefile << "DISTRIBUTIONS \u2461 size in x " << XDistrib << " in y "<< YDistrib << " extended "
			<< YDistrib_extended << " size from geometry " << XDistrib*YDistrib_extended << " ADistrib "<< ADistrib << endl;
	memblock = new char[size];
	distribile.seekg(byte_skipped, ios::beg); // bytes skipped are offset
	distribile.read(memblock, size);
	distribile.close();

	double_distrib = (double*) memblock; //reinterpret the chars stored in the file as double

	for (int i = 0; i < ADistrib; i++) {
//		if(*(double_distrib + i) > 10.) printf(" i %d *(double_distrib + i) %f \n", i, *(double_distrib + i));
		*(original_distrib + i + idistrib*XDistrib*YDistrib_extended) = *(double_distrib + i);	// change to float
		Sumdistrib += double_distrib[i];
		Maxdistrib = max(Maxdistrib, *(double_distrib + i));
	}
	verbosefile << "DISTRIBUTIONS \u2461: idistrib " << idistrib << " Original max " << Maxdistrib << " Sumdistrib " << Sumdistrib << endl;
	}
	// write distrib image to disk
	/////////////////////////////////
	for (int i = 0; i < YDistrib_extended * XDistrib * Ndistrib; i++)
		ii_distrib[i] = 255.0 * original_distrib[i] / Maxdistrib;// image value
	verbosefile << "DISTRIBUTIONS \u2461 function read: Path to distrib original " << distribImagefile << endl;

	sdkSavePGM(distribImagefile, ii_distrib, XDistrib, YDistrib_extended * Ndistrib);

	free(ii_distrib);

}

bool Distribvalidate_host(void) {
	bool testdistrib;
	double Sum3distrib = 0, max3distrib = 0;

	unsigned char *ii_distrib = (unsigned char *) calloc( YDistrib_extended * XDistrib * Ndistrib, sizeof(unsigned char)); // on host
	// write distrib in memory and validate
	hipMallocManaged(&val_distrib, YDistrib_extended * XDistrib * Ndistrib * sizeof(float));
	const char * distribValImagefile = "results/B_distributionsdevice.pgm";

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the distrib kernel
	validate_distrib<<<dimGrid, dimBlock, 0>>>(YDistrib_extended, XDistrib, Ndistrib);
	hipDeviceSynchronize();

	for (int arg = 0; arg < ADistrib * Ndistrib; arg++)	Sum3distrib += *(val_distrib + arg);
	for (int arg = 0; arg < ADistrib * Ndistrib; arg++) max3distrib = max(max3distrib, *(val_distrib + arg));

	printf("DISTRIBUTIONS \u2461: validation max %g Sum %g\n", max3distrib, Sum3distrib);

	for (int i = 0; i < YDistrib_extended * XDistrib * Ndistrib; i++)
		ii_distrib[i] = (255.0 * val_distrib[i]) / max3distrib;// Validation image value

	verbosefile << "DISTRIBUTIONS \u2461 Path to distrib validation " << distribValImagefile << endl;

	sdkSavePGM(distribValImagefile, ii_distrib, XDistrib, YDistrib_extended * Ndistrib);

	verbosefile << "DISTRIBUTIONS \u2461 Comparing files ... ";
	testdistrib = compareData(val_distrib, original_distrib,
			XDistrib * YDistrib_extended * Ndistrib,
			MAX_EPSILON_ERROR, 0.15f);

	for (int jdistrib = 0; jdistrib < YDistrib_extended * XDistrib * Ndistrib;
			jdistrib++) {
		Sumdel[1] += fabsf(
				*(val_distrib + jdistrib) - *(original_distrib + jdistrib));
	}
	verbosefile << "Sumdel[1]  " << Sumdel[1];
	verbosefile << "testdistrib = " << testdistrib << "\n";
	hipFree(val_distrib);
	return (testdistrib);
}

