#include "hip/hip_runtime.h"
/*
 * 7_displayresults.cu
 *
 *  Created on: December 16, 2017
 *      Author: gabriel
 */

#include"0_Mainparameters.h"

float displaySimus(float * simusvalues, string filebase) {
	int sizesimus = tile.maxLaserintile * tile.NbTileXYD * NThreads;
	int sizesimus2D = tile.maxLaserintile * tile.NbTileXYD * PixZoomSquare;
	float MaxSimus2D = 0.0f, MinSimus2D = 1.E6, SumSimus2D = 0.0;
	int Nancounter = 0;
	string file;
	int n_colintern = PixZoom * tile.NbTileXYD;
	int n_rowintern = PixZoom * tile.maxLaserintile;

	unsigned char *i_simus = (unsigned char *) calloc(sizesimus2D, sizeof(unsigned char)); // on host

	for (int i = 0; i < sizesimus; i++) MaxSimus2D = max(MaxSimus2D, *(simusvalues + i)); // all distributions!!
	for (int i = 0; i < sizesimus; i++) SumSimus2D += *(simusvalues + i); // all distributions!!
	for (int i = 0; i < sizesimus; i++) MinSimus2D = min(MinSimus2D, *(simusvalues + i));
	for (int i = 0; i < sizesimus; i++) if(std::isnan(*(simusvalues + i))) {
		printf(" i %d value %f \n", i, *(simusvalues + i));
		Nancounter++;
	}
	float ratio = 255. / (MaxSimus2D - MinSimus2D);


	verbosefile << "HOST: \u24EF parameters: row simus " << n_rowintern << " col simus " << n_colintern << endl;
	cout << " Nan counter " << Nancounter << endl;
	verbosefile << " Maximum Simulations " << MaxSimus2D << " Minimum simulations " << MinSimus2D << " Sum " << SumSimus2D
			<<  " ratio " << ratio << " size " << sizesimus << endl;


		file = filebase + ".pgm"; verbosefile << "file " << file << endl;

		int ipix = 0;
		for (int isimus = 0; isimus < sizesimus; isimus++) {
			int imicro = isimus / NThreads; // number of microimage
			int ipixel = isimus % NThreads; // pixel number in microimage including void pixels
			if (ipixel < PixZoomSquare)
			{
				int ix = (ipixel % PixZoom) + PixZoom * (imicro % tile.maxLaserintile);
				int iy = (ipixel / PixZoom) + PixZoom * (imicro/tile.maxLaserintile );
				i_simus[ix + iy * PixZoom*tile.NbTileXYD] = ratio * (simusvalues[isimus] - MinSimus2D);
			}
		}
		sdkSavePGM(file.c_str(), i_simus, n_colintern, n_rowintern);


	return (TRUE);
}
