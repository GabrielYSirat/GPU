
#include <hip/hip_runtime.h>
#ifdef TESTDISTRIBDEVICE
	DD.step++;
	for (int idistrub = ithreads; idistrub < ADistrib; idistrub += THREADSVAL)
			*(test2_distrib + idistrub + itc * ADistrib) = *(shared_distrib + idistrub);
	timer = clock64();
	if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f from start  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	if (!iprint) printf("\u2461 **********************************DEVICE:  DISTRIBUTIONS  ********************\n\n");
	__syncthreads();
#endif