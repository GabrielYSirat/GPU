
#include <hip/hip_runtime.h>
#ifdef TESTDISTRIBDEVICE
	DD.step++;
	for (int idistrub = ithreads; idistrub < ADistrib; idistrub += THREADSVAL)
			*(test2_distrib + idistrub + itb * ADistrib) = *(shared_distrib + idistrub);
	for (int ipsf = 0; ipsf < PSFZOOMSQUARE; ipsf ++)
			*(test2_psf + ipsf) = *(original_PSF + ipsf);
	__syncthreads();
//    timer = clock64();
	if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	if (!iprint) printf("\u2461 **********************************DEVICE:  DISTRIBUTIONS  ********************\n\n");
	__syncthreads();
#endif
