
#include <hip/hip_runtime.h>
#ifdef TESTDISTRIBDEVICE
float Maxdistrib2 = 0.0f;
	DD.step++;
	for (int idistrub = ithreads; idistrub < ADistrib; idistrub += THREADSVAL)
			*(test2_distrib + idistrub + itb * ADistrib) = *(shared_distrib + idistrub);
	for (int ipsf = 0; ipsf < PSFZOOMSQUARE; ipsf ++)
			*(test2_psf + ipsf) = *(original_PSF + ipsf);
	for (int idistrub = 0; idistrub < ADistrib; idistrub ++){
		Maxdistrib2 = max(Maxdistrib2, *(test2_distrib + idistrub + itb * ADistrib));
	}

		__syncthreads();
//    timer = clock64();
		if (!iprint)
			printf("DEVICE: \u23f1**DEVICE: MaxDistribution2 %f       \n", Maxdistrib2);
	if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f this step  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	if (!iprint) printf("\u2461 **********************************DEVICE:  DISTRIBUTIONS  ********************\n\n");
	__syncthreads();
#endif
