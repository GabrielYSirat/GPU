#include "hip/hip_runtime.h"
/*
 * RSVReconstruction.cu
 *
 *  Created on: Jul 8, 2017
 *      Author: gabriel
 */
#include "0_Mainparameters.h"
std::string filenameimage;
std::string RECFILE = "image_iteration_0__63x114_4em";
std::string endREC = ".raw";
const char * reconstructionImagefile = "results/D_reconstruction.pgm";
const char * recValImagefile = "results/D_reconstructiondevice.pgm";
const char * rectilereconstructionfile = "results/D_reconstructionreorganized.pgm";
const char * scratchpadImagefile = "results/E_scratchpad.pgm";
const char * ScratchpadValImagefile = "results/E_Scratchpaddevice.pgm";
float MaxRec = 0.0f, SumRec = 0.0f;
__managed__ float Maxscratch = 0.0f, Sumscratch = 0.0f, maxTile = 0.0f, SumTile = 0.0f;
double *double_rec;
// double* double_rec;

void Recprepare(void) {
	hipMallocManaged(&original_rec, TA.reconstruction_size * sizeof(float)); // on device with a shadow on host
	hipMallocManaged(&double_rec, TA.reconstruction_size * sizeof(double)); // on device with a shadow on host
//	double* double_rec = (double*) std::malloc(TA.reconstruction_size * sizeof(double)); // on host

	char * memblock;
	int size;

	filenameimage = resourcesdirectory + RECFILE + endREC;
	verbosefile << "REC \u24FC reconstruction image:  " << filenameimage.c_str() << " \n";
	/** *****************************data arrays allocation*********************************/
	/** original_reconstruction data in float stored on device with a shadow copy on host
	 *  double_rec data in double read from the file
	 *  i_reconstruction normalized data in char on host for image display
	 */

	//read reconstruction raw file
	std::ifstream RecFile(filenameimage.c_str(), ios::in | ios::binary | ios::ate);
	size = (RecFile.tellg()); 	// the data is stored in float of 4 bytes in the file
	size -= byte_skipped; 	// WE REMOVE byte_skipped BYTES
	verbosefile << "REC \u24FC ************file read: size reconstruction in bytes = " << size << endl;
	memblock = new char[size];
	RecFile.seekg(byte_skipped, ios::beg); // byte_skipped first bytes are skipped
	RecFile.read(memblock, size);
	RecFile.close();
	verbosefile << "REC \u24FC *******complete size  " << TA.reconstruction_size << "  Size in Bytes "
			<< TA.reconstruction_size * sizeof(double) << endl;

	/** read reconstruction data from file in double, transfer to float on the global memory of the device and create a normalized image
	 *
	 */
	double_rec = (double*) memblock; //reinterpret the chars stored in the file as float
	for (int i = 0; i < TA.reconstruction_size; i++) {
		*(original_rec + i) = *(double_rec + i);
		SumRec += *(original_rec + i);
		MaxRec = max(*(original_rec + i), MaxRec);
	}	// sanity check, check max and sum

	verbosefile << "REC \u24FC ***  max =" << MaxRec << "  Sum =" << SumRec << endl;
	unsigned char *i_reconstruction = (unsigned char *) calloc(TA.reconstruction_size, sizeof(unsigned char)); // on host
	// write reconstruction image to disk /////////////////////////////////
	for (int i = 0; i < TA.reconstruction_size; i++)
		i_reconstruction[i] = 255.0 * original_rec[i] / MaxRec;			// image value

	verbosefile << "REC \u24FC Path to reconstruction original " << reconstructionImagefile << " .....\n";
	sdkSavePGM(reconstructionImagefile, i_reconstruction, TA.Nb_Cols_reconstruction,
			TA.Nb_Rows_reconstruction);
	free(i_reconstruction);
//	free(double_rec);
}

bool Recvalidate_host(void) {
	bool testrec;
	float MaXTile = 0.0f, Sum3rec = 0.0f, max3rec = 0.0f;

	// write rec in memory and validate
	unsigned char *i_rec = (unsigned char *) calloc(TA.reconstruction_size, sizeof(unsigned char)); // on host
	hipMallocManaged(&val_rec, TA.reconstruction_size * sizeof(float));

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the rec kernel
	Recvalidate_device<<<dimGrid, dimBlock, 0>>>(TA.Nb_Rows_reconstruction, TA.Nb_Cols_reconstruction);
	hipDeviceSynchronize();

	for (int row = 0; row < TA.Nb_Rows_reconstruction; row++)
		for (int col = 0; col < TA.Nb_Cols_reconstruction; col++) {
			int tempr = row * TA.Nb_Cols_reconstruction + col;
			Sum3rec += *(val_rec + tempr);
			max3rec = max(max3rec, *(val_rec + tempr));
		}
	verbosefile << endl << "on host: Sum3rec  " << Sum3rec << " max3rec %f   " << max3rec << endl;

	// write rec image validation to disk
	/////////////////////////////////
	MaXTile = 0.0f;

	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
		for (int i = 0; i < TA.reconstruction_size; i++) {
			MaXTile = max(MaXTile, val_rec[i]); // sanity check, check max
		}
	verbosefile << "max device =" << MaXTile << "\n";
	for (int i = 0; i < TA.reconstruction_size; i++) {
		i_rec[i] = 255.0 * val_rec[i] / MaXTile;			// Validation image value
		if (VERBOSE)
			if (i_rec[i] > 1)
				printf(
						"REC VALIDATION \u24FC position %d, size, %d column width %d xy position (x*y) (%d*%d) "
						"value %g normalized %d\n",
						i, TA.reconstruction_size, TA.Nb_Cols_reconstruction, i % TA.Nb_Cols_reconstruction,
						i / TA.Nb_Cols_reconstruction, val_rec[i], i_rec[i]);
	}
	verbosefile << "REC \u24FC Path to rec validation " << recValImagefile << endl;
	if (VERBOSE)
		printf("---------------------------------------\n");
	sdkSavePGM(recValImagefile, i_rec, TA.Nb_Cols_reconstruction, TA.Nb_Rows_reconstruction);

	verbosefile << "REC \u24FC Comparing files ... ";
	testrec = compareData(val_rec, original_rec, TA.reconstruction_size, MAX_EPSILON_ERROR, 0.15f);

	for (int jrec = 0; jrec < TA.reconstruction_size; jrec++) {
		Sumdel[1] += fabsf(*(val_rec + jrec) - *(original_rec + jrec));
	}
	verbosefile << "Sumdel[1] " << Sumdel[1] << " ... " << endl;
	verbosefile << "testrec = " << testrec << "\n";

	hipFree(val_rec);
	return (testrec);
}

void Scratchprepare(void) {
	float *tile_rec = (float*) std::calloc(ATile * tile.NbTileXY, sizeof(float)); 					// on host
	unsigned char *i_tilerec = (unsigned char *) calloc(ATile * tile.NbTileXY, sizeof(unsigned char)); // on host
	hipMallocManaged(&scratchpad_matrix, tile.NbTileXY * ASCRATCH * sizeof(float));
	unsigned char *i_scratchpad = (unsigned char *) calloc(tile.NbTileXY * XSCRATCH * YSCRATCH,
			sizeof(unsigned char)); // on host

	/***********************************TILE RECONSTRUCTION ************************/
	verbosefile << "TILE \u24FC Path to tile reconstruction  " << rectilereconstructionfile << endl;

	int deltilex = tile.NbTilex * XTile - TA.Nb_Cols_reconstruction;
	int deltiley = tile.NbTiley * YTile - TA.Nb_Rows_reconstruction;
	verbosefile << " offset = - del /2 !! x " << deltilex / 2 << "  y  " << deltiley / 2 << endl;

	for (int arg = 0; arg < TA.reconstruction_size; arg++) {
		maxTile = max(maxTile, *(original_rec + arg));
		SumTile += *(original_rec + arg);
	}
	Maxscratch = maxTile;
	Sumscratch = SumTile;
	verbosefile << "TILE \u24FC maxTile " << maxTile << " SumTile " << SumTile << endl;

	for (int row = 0; row < TA.Nb_Rows_reconstruction; row++)
		for (int col = 0; col < TA.Nb_Cols_reconstruction; col++) {
			int itemp = col + deltilex / 2 + (row + deltiley / 2) * tile.NbTilex * XTile;
			int itemp2 = col + row * TA.Nb_Cols_reconstruction;
			*(tile_rec + itemp) = *(original_rec + itemp2);
			i_tilerec[itemp] = 255. * *(tile_rec + itemp) / maxTile;
			if (VERBOSE)
				if (i_tilerec[itemp] > 1)
					if (VERBOSE)
						printf(
								"SCRATCHPAD VALIDATION \u24FC position %d, size, %d column width %d xy position (x*y) (%d*%d) "
								"value %g normalized %d\n",
								itemp, TA.reconstruction_size, TA.Nb_Cols_reconstruction, itemp % TA.Nb_Cols_reconstruction,
								itemp / TA.Nb_Cols_reconstruction, tile_rec[itemp], i_tilerec[itemp]);
		}

	if (VERBOSE)
		printf("---------------------------------------\n");
	sdkSavePGM(rectilereconstructionfile, i_tilerec, XTile * tile.NbTilex, YTile * tile.NbTiley);

	verbosefile << "SCRATCHPAD \u24FC Image of scratchpad matrix " << scratchpadImagefile << " .....\n";
	verbosefile << "SCRATCHPAD \u24FC : Max Scratchpad " << Maxscratch << " Sum scratchpad " << Sumscratch
			<< endl;
	verbosefile << "SCRATCHPAD \u24FC : " << XSCRATCH * YSCRATCH << " of full SCRATCHPAD 2D "
			<< XSCRATCH * YSCRATCH * tile.NbTileXY << endl;
	// write scratchpad matrix to disk
	scratchreaddisplay(tile_rec, scratchpad_matrix, scratchpadImagefile, TRUE);

	free(i_scratchpad);

}

bool Scratchvalidate_host(void) {
	bool testScratchpad;
	float Sum3Scratchpad = 0.0f, max3Scratchpad = 0.0f;
	float * dummy = { 0 };

	// write Scratchpad in memory and validate
	unsigned char *i_scratchpad = (unsigned char *) calloc(tile.NbTileXY * XSCRATCH * YSCRATCH,
			sizeof(unsigned char)); // on host
	hipMallocManaged(&val_scratchpad, tile.NbTileXY * ASCRATCH * sizeof(float));
	hipMallocManaged(&val2_scratchpad, tile.NbTileXY * ASCRATCH * Ndistrib * sizeof(float));

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(1, 1, 1);
	// Execute the Scratchpad kernel
	Scratchvalidate_device<<<dimGrid, dimBlock, 0>>>(tile.NbTilex,tile.NbTiley, lostpixels);
	hipDeviceSynchronize();

	for (int arg = 0; arg < ASCRATCH * tile.NbTileXY; arg++) {
		Sum3Scratchpad += *(val_scratchpad + arg);
		max3Scratchpad = max(max3Scratchpad, *(val_scratchpad + arg));
	}
	printf("SCRATCHPAD \u24FC Sum3Scratchpad  %f max3Scratchpad %f   \n", Sum3Scratchpad, max3Scratchpad);

	// write Scratchpad image validation to disk
	/////////////////////////////////
	Maxscratch = 0.0f;

	for (int idistrib = 0; idistrib < Ndistrib; idistrib++)
		for (int i = 0; i < ASCRATCH * tile.NbTileXY; i++) {
			if (val_scratchpad[i] > 1. && VERBOSE)
				printf("SCRATCHPAD \u24FC i %d x position in scratch %d y position %d val %f\n", i,
						i % XSCRATCH, i / XSCRATCH, val_scratchpad[i]);
			Maxscratch = max(Maxscratch, val_scratchpad[i]); // sanity check, check max
		}
	verbosefile << "max device =" << Maxscratch << "\n";

	scratchreaddisplay(dummy, val_scratchpad, ScratchpadValImagefile, FALSE);
	verbosefile << "SCRATCHPAD \u24FC Path to Scratchpad validation " << ScratchpadValImagefile << " .....\n";
	verbosefile << "SCRATCHPAD \u24FC Comparing files ... " << endl;
	testScratchpad = compareData(val_scratchpad, scratchpad_matrix, ASCRATCH * tile.NbTileXY,
	MAX_EPSILON_ERROR, 0.15f);

	for (int jScratchpad = 0; jScratchpad < ASCRATCH * tile.NbTileXY; jScratchpad++) {
		Sumdel[8] += fabsf(*(val_scratchpad + jScratchpad) - *(scratchpad_matrix + jScratchpad));
	}
	verbosefile << "Sumdel[8] " << Sumdel[8] << endl;
	verbosefile << "testScratchpad = " << testScratchpad << "\n";
	hipFree(val_scratchpad);
	return (testScratchpad);
}

