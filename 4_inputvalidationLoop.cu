#include "hip/hip_runtime.h"
#include "0_Mainparameters.h"

using namespace tinyxml2;
using namespace std;

#define TEST 1
#define VERBOSEINITLOOP 1

const char * Laserfile = "results/LaserDevice.txt";


//////////////pPSF parameters
//__device__ float PSF_array[pPSF*pZOOM*pPSF*pZOOM]; // pPSF in constant memory

__managed__ float *PSF_valid = NULL, *PSFARRAY = NULL, *original_PSF=NULL, *test2_psf = NULL;
__managed__ float *distrib = NULL;			 // on device original and for validation
__managed__ float *MicroImages = NULL;				 // on device original and for validation
__managed__ float *LaserPositions = NULL, *v_LaserPositions = NULL;		 // on device original and for validation
__managed__ float *PosLaserx = NULL, *PosLasery = NULL;
__managed__ float *d_PosLaserx, *d_PosLasery;
__managed__ int *PosxScratch = NULL, *PosyScratch = NULL;
__managed__ int *posxREC, *posyREC, *offsetFULL;
__managed__ int *d_PosxScratch = NULL, *d_PosyScratch = NULL;
__managed__ int *d_posxREC, *d_posyREC, *d_offsetFULL;

__managed__ float *original_distrib, *val_distrib, *test_distrib, *test2_distrib;
__managed__ double *double_distrib;
__managed__ float *original_microimages, *valmicroimages, *zoomed_microimages;
__managed__ float *original_rec, *val_rec;
__managed__ float *scratchpad_matrix, *val_scratchpad, *val2_scratchpad, *scratch1D;

__managed__ float *Sumdevmicroimages, *Maxdevmicroimages, *Sumdevzoommicroimages, *Maxdevzoommicroimages;

__global__ void PSFvalidateondevice(int Nb_Rows_PSF, int Nb_Cols_PSF) {
	double SumPSF = 0, maxPSF = 0, Sum2PSF = 0, max2PSF = 0; // in pPSF the Sum of all pixels is normalize to 1.00000
	float tempv;
	time_start = clock64();
	time_init = clock64();
	printf(" PSF \u2776 device: Rows: %d cols: %d ...", Nb_Rows_PSF, Nb_Cols_PSF);
// calculate pPSF on device Sum and max
	for (int row = 0; row < Nb_Rows_PSF; row++)
		for (int col = 0; col < Nb_Cols_PSF; col++) {
			int tempp = row * Nb_Cols_PSF + col;
			tempv = *(original_PSF + tempp);
			PSFARRAY[tempp] = *(original_PSF + tempp);
			*(PSF_valid + tempp) = tempv;
			Sum2PSF += *(PSF_valid + tempp);
			SumPSF += PSFARRAY[tempp];
			if (maxPSF < PSFARRAY[tempp])
				maxPSF = PSFARRAY[tempp];
			if (max2PSF < *(PSF_valid + row * Nb_Cols_PSF + col))
				max2PSF = *(PSF_valid + row * Nb_Cols_PSF + col);
		}
	if ((threadIdx.x == 0) && (threadIdx.y == 0))
		printf("  SumPSF %f Sum2PSF %f maxPSF %f max2PSF %f ...  \n\n", SumPSF, Sum2PSF, maxPSF,
				max2PSF);
	timer = clock64();
}

__global__ void validate_distrib(int Nb_Rows_distrib, int Nb_Cols_distrib, int Nb_Distrib) {
	double Sumdistrib = 0, maxdistrib = 0, Sum2distrib = 0, max2distrib = 0; // in distrib the Sum of all pixels is normalize to 1.00000
	float tempv;
	int tempp;
	time_start = clock64();
	printf("DISTRIBUTIONS \u2777  device: Nb_Row %d, Nb_col %d, Nb_distrib %d ...", Nb_Rows_distrib, Nb_Cols_distrib,
			Nb_Distrib);
// calculate distrib Sum and max
	for (int idistrib = 0; idistrib < Nb_Distrib; idistrib++)
		for (int row = 0; row < Nb_Rows_distrib; row++)
			for (int col = 0; col < Nb_Cols_distrib; col++) {
				tempp = (idistrib * Nb_Rows_distrib + row) * Nb_Cols_distrib + col;
				tempv = *(original_distrib + tempp);
				*(val_distrib + tempp) = tempv;
				Sum2distrib += *(val_distrib + row * Nb_Cols_distrib + col);
				Sumdistrib += *(original_distrib + row * Nb_Cols_distrib + col);
				if (maxdistrib < *(original_distrib + row * Nb_Cols_distrib + col))
					maxdistrib = *(original_distrib + row * Nb_Cols_distrib + col);
				if (max2distrib < *(val_distrib + row * Nb_Cols_distrib + col))
					max2distrib = *(val_distrib + row * Nb_Cols_distrib + col);
			}
	printf(" Sum distrib %f Sum2distrib %f ..."
			"  max distrib %f max2distrib %f ...  \n", Sumdistrib, Sum2distrib,
			maxdistrib, max2distrib);
	timer = clock64();

}

__global__ void validateLaserPositions_device(int Nb_LaserPositions) {
	double maxLaserPositionx = 0, maxLaserPositiony = 0; // in LaserPosition the max in x is xmax
	double minLaserPositionx = 1E6, minLaserPositiony = 1E6; // in LaserPosition the max in x is xmax
	int xREC, yREC, xREC1, temp, tilex, tiley;
// calculate LaserPositions Sum and max
	time_start = clock64();

	for (int ipos = 0; ipos < Nb_LaserPositions; ipos++) {
		d_PosLaserx[ipos] = PosLaserx[ipos];
		d_PosLasery[ipos] = PosLasery[ipos];
		// Laser positions in y zoomed in integer
		tilex = pZOOM * *(PosLaserx + ipos) / XTile; xREC1 = pZOOM * *(PosLaserx + ipos); temp = tilex * XTile; xREC = xREC1 - temp;
		*(PosxScratch + ipos) = xREC + dxSCR / 2; // Laser positions in x zoomed integer in the scratchpad of tile
		tiley = pZOOM * *(PosLasery + ipos) / YTile; yREC = pZOOM * *(PosLasery + ipos) - tiley * YTile;
		*(PosyScratch + ipos) = yREC + dySCR / 2; // Laser positions in x zoomed integer in the scratchpad of tile
		*(offsetFULL + ipos) = *(PosxScratch + ipos) + *(PosyScratch + ipos) * XSCRATCH + lostpixels - (XSCRATCH + 1) * PSFZoomo2;
						// adding lostpixels and difference between center and edge of PSF region in the scratchpad

		if(NOVERBOSE){
			printf(" Laser \u2778 DEVICE 1: laser position n° %d original position x:%f , y: %f ....\n", ipos,
					PosLaserx[ipos], PosLasery[ipos]);
			printf(" Laser \u2778 DEVICE 2:  ....xREC1 %d temp %d xREC %d yREC %d tilex %d tiley %d Scratchx %d Scratchy %d \n", xREC1, temp, xREC,
					yREC, tilex, tiley, PosxScratch[ipos], PosyScratch[ipos]);
			printf(" Laser \u2778 DEVICE 3: copy position: %f y: %f\n", d_PosLaserx[ipos], d_PosLasery[ipos]);
			printf("Laser \u2778 DEVICE laser position n°%d offset %d \n",ipos, *(offsetFULL + ipos));
			printf("__________________________\n");
		}

		minLaserPositionx = min(minLaserPositionx, PosLaserx[ipos]);
		minLaserPositiony = min(minLaserPositiony, PosLasery[ipos]);
		maxLaserPositionx = max(maxLaserPositionx, PosLaserx[ipos]);
		maxLaserPositiony = max(maxLaserPositiony, PosLasery[ipos]);
	}
	printf("Laser \u2778  DEVICE: MAX & MIN: LaserPosition x max %f min %f ... LaserPositiony max %f min %f \n",
			maxLaserPositionx, minLaserPositionx, maxLaserPositiony, minLaserPositiony);
	timer = clock64();

}

__managed__ float * SumMI;

__global__ void validate_microimages(int Nb_LaserPositions) {
	float tempv;
	int tempp, tempz;
	int iprint = threadIdx.x + threadIdx.y;
	int iblock = blockIdx.x + blockIdx.y;
	if (!(iprint + iblock))
		time_start = clock64();
	__syncthreads();

	for (int ilaser = 0; ilaser < Nb_LaserPositions; ilaser++)
		for (int row = 0; row < Npixel; row++)
			for (int col = 0; col < Npixel; col++) {
				tempp = (ilaser * Npixel + row) * Npixel + col;
				tempv = *(original_microimages + tempp);
				*(valmicroimages + tempp) = tempv;

				for (int yz = 0; yz < pZOOM; yz++)
					for (int xz = 0; xz < pZOOM; xz++) {
						tempz = ilaser * PixZoomSquare + (row * pZOOM + yz) * PixZoom
								+ pZOOM * col + xz;
				*(zoomed_microimages + tempz) = tempv;
					}
				__syncthreads();
			}
	if (!(iprint + iblock))
		timer = clock64();

}

__global__ void microimages_intiles(int Nb_tiles, int nbLintile) {
}
__global__ void Recvalidate_device(int Nb_Rows_reconstruction, int Nb_Cols_reconstruction) {
	double Sumreconstruction = 0.0f;
	double maxreconstruction = 1.0f;

	float tempv;
	int tempp;
// calculate reconstruction Sum and max
	time_start = clock64();
	if(VERBOSE)
		printf("REC \u277D DEVICE ----------------------------------------------------------------------------------------------------\n");
	for (int row = 0; row < Nb_Rows_reconstruction; row++)
		for (int col = 0; col < Nb_Cols_reconstruction; col++) {
			tempp = row * Nb_Cols_reconstruction + col;
			tempv = *(original_rec + tempp);
			*(val_rec + tempp) = tempv;
			if ((tempv != 0.0f) && (TEST) && VERBOSE) {
				printf("REC \u277D DEVICE position %d position x: %d y: %d value %f\n", tempp,
						tempp % Nb_Cols_reconstruction, tempp / Nb_Cols_reconstruction, tempv);
			}
			Sumreconstruction += *(original_rec + row * Nb_Cols_reconstruction + col);
			if (maxreconstruction < *(original_rec + row * Nb_Cols_reconstruction + col))
				maxreconstruction = *(original_rec + row * Nb_Cols_reconstruction + col);
		}
	if(VERBOSE)
		printf(
			"REC \u277D DEVICE ----------------------------------------------------------------------------------------------------\n\n");
	printf("REC \u277D DEVICE:  Sum reconstruction %f max reconstruction %f ...  \n\n", Sumreconstruction,
			maxreconstruction);
	__syncthreads();
	if ((threadIdx.x == 0) && (threadIdx.y == 0))
		timer = clock64();
	__syncthreads();

}

__global__ void Scratchvalidate_device(int NbTilex, int NbTiley, int dels) {
	float Sumscratchpad = 0.0f, maxscratchpad = 0.0f;
	float tempv;
	int NbTileXY = NbTilex * NbTiley; // local copy ??
// calculate scratchpad Sum and max
	time_start = clock64();
	for (int iscratch = 0; iscratch < ASCRATCH * NbTileXY; iscratch++) {
		tempv = *(scratchpad_matrix + iscratch);
		*(val_scratchpad + iscratch) = tempv;
		Sumscratchpad += *(val_scratchpad + iscratch);
		maxscratchpad = max(maxscratchpad, *(val_scratchpad + iscratch));


		if ((*(val_scratchpad + iscratch) != 0.0f) && (TEST)) {
			int positionx = (iscratch - dels) % (XSCRATCH * NbTilex);
			int positiony = (iscratch - dels) / (XSCRATCH * NbTilex);
			if (VERBOSE)
				printf(
						"SCRATCHPAD \u277E position %d, size XY (x*y), (%d*%d) xy position (x*y) (%d*%d) value %g\n",
						iscratch, XSCRATCH*NbTilex, YSCRATCH*NbTiley, positionx,
						positiony, val_scratchpad[iscratch]);
			Maxscratch = max(Maxscratch, val_scratchpad[iscratch]); // sanity check, check max
		}

	}
	printf("SCRATCHPAD \u277E DEVICE:  Sum scratchpad %f max scratchpad %f ... \n", Sumscratchpad,maxscratchpad);
	if(VERBOSE)
		printf("SCRATCHPAD \u277E DEVICE ----------------------------------------------------------------------------------------------------\n\n");

	timer = clock64();
	__syncthreads();

}

