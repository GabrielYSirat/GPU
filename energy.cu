#include "hip/hip_runtime.h"
/*
 * energy.cu
 *
 *  Created on: Sep 19, 2017
 *      Author: gabriel
 */

#include "NewLoop.h"
	__managed__ double Energy = 0.0f, absdiff = 0.0f;
float EnergyCal(void) {
	int it;
	for (int iimage =0; iimage < onhost.Nb_LaserPositions; iimage++)
		for (int ipix = 0; ipix < NThreads; ipix++) {
			it = ipix + iimage * NThreads;
			Energy += new_simus[it] - Data[it]*log(new_simus[it]+onhost.Bconstant);
			absdiff += abs(new_simus[it] - Data[it]);
			Rfactor[it] = 1 - Data[it] / (new_simus[it] + onhost.Bconstant);
		}
	printf("Energy %8.6f absdiff %8.6f\n\n", Energy, absdiff);
	return (Energy);
}

