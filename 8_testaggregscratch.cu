
#include <hip/hip_runtime.h>
#ifdef TESTAGG
	DD.step++;
	if(!itc && VERBOSELOOP) printf(" \u2463******DEVICE:  AGGREGATES x %d y %d *****\n", aggregx, aggregy);
	if (!iprint) printf("end \u2463***********************DEVICE:  AGGREGATES & TILES*****************\n");
	if (!iprint) printf("DEVICE:\u2463 AGGREGATES \nDEVICE:\u2463 TILES tilex %d, tiley %d tile %d MemoryOffsetscratch %d\n",
			tilex, tiley, tileXY, MemoryOffsetscratch);
	if (!iprint) printf("end \u2463***********************DEVICE:  AGGREGATES & TILES*****************\n");

#endif

#ifdef TESTSCRATCH
	if(!ithreads)
		for (int jscratch = 0; jscratch < ASCRATCH; jscratch ++){
		Sumscratch += *(Scratchpad + jscratch);
		maxscratch = max(Scratchpad[jscratch], maxscratch);
		val2_scratchpad[jscratch + MemoryOffsetscratch] = Scratchpad[jscratch];  // scratchpad image validation
		if(*(Scratchpad + jscratch) != 0.0f && !ithreads) printf("ithreads %d itb %d position in scratchpad %d value %f Sum %f max %f\n",
				ithreads, itc, jscratch, *(Scratchpad + jscratch), Sumscratch, maxscratch);
	}

	if (!iprint) printf("DEVICE: \u2464 SUM SCRATCHPAD: Sum of scratchpad %5.1f Max of Scratchpad %5.1f \n", Sumscratch, maxscratch);
	if (!iprint)
		printf( "DEVICE: \u23f1**DEVICE:  step %d   TIMING (msec) ** processing  %f from start  %g  total %g \n",
			DD.step, (float) (timer - time_start) / DD.clockRate,
			(float) (  time_start - time_init) / DD.clockRate,
			(float) (timer - time_init) / DD.clockRate);
	if (!iprint) printf("end \u2464*******************************DEVICE:  SCRATCHPAD ********************\n\n");
			__syncthreads();

#endif
